#include "hip/hip_runtime.h"
#include "Common.h"
#include "Common_cuda.cuh"
#include "hip/hip_runtime.h"

extern "C"
{
#include "Buddy_System.h"
}
using namespace std;
Mem_Mgr oMem_Mgr_GPU{};

void SB_Common_Cuda()
{//���е�ģ�庯��ʵ����
	Disp_Part_GPU<char>(NULL, 0, 0, 0, 0, 0);
    Disp_Part_GPU<unsigned char>(NULL, 0, 0, 0, 0, 0);
	Disp_Part_GPU<int>(NULL, 0, 0, 0, 0, 0);
	Disp_Part_GPU<unsigned int>(NULL, 0, 0, 0, 0, 0);
	Disp_Part_GPU<float>(NULL, 0, 0, 0, 0, 0);
	Disp_Part_GPU<double>(NULL, 0, 0, 0, 0, 0);
	Disp_Part_GPU<short>(NULL, 0, 0, 0, 0, 0);
	Disp_Part_GPU<unsigned short>(NULL, 0, 0, 0, 0, 0);

	Disp_GPU<char>(NULL, 0, 0);
	Disp_GPU<unsigned char>(NULL, 0, 0);
	Disp_GPU<int>(NULL, 0, 0);
	Disp_GPU<unsigned int>(NULL, 0, 0);
	Disp_GPU<float>(NULL, 0, 0);
	Disp_GPU<double>(NULL, 0, 0);
	Disp_GPU<short>(NULL, 0, 0);
	Disp_GPU<unsigned short > (NULL, 0, 0);

	Disp_Sum_GPU<double> (NULL, 0);
	Disp_Sum_GPU<float>(NULL, 0);
	Disp_Sum_GPU<int>(NULL, 0);
	Disp_Sum_GPU<unsigned int>(NULL, 0);
	Disp_Sum_GPU<char>(NULL, 0);
	Disp_Sum_GPU<unsigned char>(NULL, 0);
	Disp_Sum_GPU<short>(NULL, 0);
	Disp_Sum_GPU<unsigned short>(NULL, 0);
}

/*****************һ���Դ������**********************/
void Init_Env()
{//��ʼ��������������Щ�ڴ湩һ�к�����ʱʹ��
	//�����Ŀ��ͳһ�ڴ�
	unsigned long long iSize = 1000000000;
	const int iBlock_Size = 2048;
	void* pBuffer = NULL;
	hipError_t cudaStatus=hipHostMalloc(&pBuffer, iSize);
	if (cudaStatus != hipSuccess)
	{
		printf("Fail to allocate memory in Init_Env");
		return;
	}
	Init_Mem_Mgr_GPU(&oMem_Mgr, iSize, iBlock_Size, 997, pBuffer);
	return;
}

void Init_Env_GPU()
{//��ʼ��������������Щ�ڴ湩һ�к�����ʱʹ��
	//��ʼ��GPU���ڴ����
	unsigned long long iSize = 2000000000;
	const int iBlock_Size = 2048;
	void* pBuffer = NULL;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc(&pBuffer, iSize + iBlock_Size);
	if (cudaStatus != hipSuccess)
	{
		printf("Fail to allocate memory in Init_Env");
		return;
	}
	Init_Mem_Mgr_GPU(&oMem_Mgr_GPU, iSize, iBlock_Size, 997, pBuffer);
	return;
}

void Free_Env()
{
	printf("CPU Memory\n");
	if (oMem_Mgr.m_iPiece_Count)
		Disp_Mem(&oMem_Mgr, 0);

	if (oMem_Mgr.m_pBuffer)
	{
		hipFree(oMem_Mgr.m_pOrg_Buffer);
		oMem_Mgr.m_pOrg_Buffer = NULL;
		Free_Mem_Mgr(&oMem_Mgr);
	}
	oMem_Mgr = {};
}

void Free_Env_GPU()
{
	printf("GPU Memory\n");
	//�ͷ��̴�������
	if (oMem_Mgr_GPU.m_iPiece_Count)
		Disp_Mem(&oMem_Mgr_GPU, 0);

	if (oMem_Mgr_GPU.m_pBuffer)
	{
		hipFree(oMem_Mgr_GPU.m_pOrg_Buffer);
		oMem_Mgr_GPU.m_pOrg_Buffer = NULL;
		Free_Mem_Mgr(&oMem_Mgr_GPU);
	}
	oMem_Mgr_GPU = {};
}
void* pMalloc_GPU(unsigned int iSize)
{//ԭ����pMalloc���˸�����̫�鷳���ɴ��һ�����
	return pMalloc(&oMem_Mgr_GPU, iSize);
}
void Free_GPU(void* p)
{
	Free(&oMem_Mgr_GPU, p);
}

void Init_Env_All()
{
	Init_Env();
	Init_Env_GPU();
}
void Free_Env_All()
{
	Free_Env();
	Free_Env_GPU();
}
void Disp_Mem_GPU()
{
	Disp_Mem(&oMem_Mgr_GPU, 0);
}
void Shrink_GPU(void* p, unsigned int iSize)
{
	Shrink(&oMem_Mgr_GPU, p, iSize);
	return;
}
/*****************һ���Դ������**********************/
void Disp_Cuda_Error()
{
	hipDeviceSynchronize();
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		printf("%s\n", hipGetErrorString(cudaStatus));
}

template<typename _T>__global__ void _Disp_Part_GPU(_T* M, int iStride, int x, int y, int w, int h)
{
	int i, j;
	for (i = 0; i < h; i++)
	{
		for (j = 0; j < w; j++)
		{
			int iPos = (y + i) * iStride + (x + j);
			if (std::is_same_v<_T, float> || 
				std::is_same_v<_T, double>)
			{
				if(M[iPos] - (int)(M[iPos]))
					printf("%.8f\t", (float)M[iPos]);
				else
					printf("%d ", (int)M[iPos]);
			}
			else if (std::is_same_v<_T, int> || std::is_same_v<_T, unsigned int> ||
				std::is_same_v<_T, short> || std::is_same_v<_T, unsigned short> ||
				std::is_same_v<_T, char> || std::is_same_v<_T, unsigned char>)
				printf("%d,", (int)M[iPos]);

			//printf("Pos:%d\n", iPos);
		}
		printf("\n");
	}
}

template<typename _T>void Disp_Part_GPU(_T* M, int iStride, int x, int y, int w, int h, const char* pcCaption)
{
	if (w + x > iStride)
	{
		printf("x position exceeds width\n");
		return;
	}
	if (pcCaption)
		printf("%s\n", (char*)pcCaption);
	Disp_Cuda_Error();
	_Disp_Part_GPU << <1, 1 >> > (M, iStride, x, y, w, h);
	Disp_Cuda_Error();
}

template<typename _T>__global__ void _Disp_GPU(_T* M, int iHeight, int iWidth)
{
	for (int i = 0; i < iHeight; i++)
	{
		for (int j = 0; j < iWidth; j++)
		{
			if (std::is_same_v<_T, float>)
			{
				if (M[i * iWidth + j] - (int)M[i * iWidth + j])
					//printf("%.10ef, ", (double)M[i * iWidth + j]);
					printf("%f, ", (double)M[i * iWidth + j]);
				else
					printf("%d, ", (int)M[i * iWidth + j]);
			}
			else if (std::is_same_v<_T, double>)
			{
				if (M[i * iWidth + j] - (int)M[i * iWidth + j])
					printf("%f, ", (double)M[i * iWidth + j]);
				//printf("%.10ef, ", (double)M[i * iWidth + j]);
				else
					printf("%d, ", (int)M[i * iWidth + j]);
				//printf("%f,", (double)M[i * iWidth + j]);
			}
			else if ( std::is_same_v<_T, unsigned int> ||
				std::is_same_v<_T, int> ||
				std::is_same_v<_T, short> ||
				std::is_same_v<_T, unsigned short> ||
				std::is_same_v<_T, char> ||
				std::is_same_v<_T, unsigned char>)
				printf("%d   ", (int)(M[i * iWidth + j]));
		}
		printf("\n");
	}
	return;
}

template<typename _T>void Disp_GPU(_T* M, int iHeight, int iWidth, const char* pcCaption)
{
	if (pcCaption)
		printf("%s\n", (char*)pcCaption);
	Disp_Cuda_Error();
	_Disp_GPU << <1, 1 >> > (M, iHeight,iWidth);
	Disp_Cuda_Error();
}
template<typename _T>__global__ void _Disp_Sum_GPU(_T M[], int iSize)
{
	double fTotal = 0;
	for (int i = 0; i < iSize; i++)
		fTotal += M[i];
	printf("Sum: %lf\n", fTotal);
}
template<typename _T>void Disp_Sum_GPU(_T M[], int iSize)
{
	_Disp_Sum_GPU << <1, 1 >> > (M, iSize);
	Disp_Cuda_Error();
}
int bSave_Raw_Data_GPU(const char* pcFile, unsigned char* pBuffer, int iSize)
{
	unsigned char* pBuffer_1 = (unsigned char*)pMalloc(iSize);
	hipMemcpy(pBuffer_1, pBuffer, iSize, hipMemcpyDeviceToHost);
	Disp_Cuda_Error();
	int bResult=bSave_Raw_Data(pcFile, pBuffer_1, iSize);
	Free(pBuffer_1);
	return bResult;
}

int bLoad_Raw_Data_GPU(const char* pcFile, unsigned char**ppBuffer, int* piSize)
{
	unsigned char* pBuffer = NULL, * pBuffer_GPU = *ppBuffer;
	if (!bLoad_Raw_Data(pcFile, &pBuffer, piSize))
		return 0;
	
	if (!pBuffer_GPU)
		pBuffer_GPU= (unsigned char*)pMalloc(*piSize);
	hipMemcpy(pBuffer_GPU, pBuffer, *piSize, hipMemcpyHostToDevice);
	Disp_Cuda_Error();
	return 0;
}