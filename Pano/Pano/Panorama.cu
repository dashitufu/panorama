#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include "Common.h"
#include "Common_cuda.cuh"
#include "Matrix.h"
#include "image_cuda.h"
#include "Panorama.h"

void SB_Pano()
{
    Init_Stitch<double>(NULL, 0, 0);
    Free_Stitch<double>(NULL);
    Resize_Seam_Image<double>(NULL);
    Warp_2<double>({}, {}, NULL, NULL, 0, NULL, NULL, INTER_LINEAR, INTER_NEAREST, NULL);
    Warp_3<double>({}, NULL, NULL, 0, NULL, INTER_AREA, INTER_AREA, NULL, BORDER_CONSTANT, BORDER_CONSTANT);

    Feed<double>(NULL);
    Find<double>(NULL);
    Re_Map_3_GPU<double>({}, {}, 0, 0, NULL, INTER_LINEAR, INTER_LINEAR, BORDER_CONSTANT, BORDER_CONSTANT);
    Block_Compensate<double>(NULL, NULL, NULL);
    Resize_Bitwise_And<double>(NULL, NULL, NULL);

}

__device__ static int iGet_Border_y_GPU(int y, int iHeight, Border_Type iBorder_Type, int iThread_ID = 0)
{
    if (y < 0)
    {
        switch (iBorder_Type)
        {
        case Border_Type::BORDER_REFLECT:
            return -y - 1;
        case Border_Type::BORDER_REFLECT_101:
            return -y;
        case Border_Type::BORDER_CONSTANT:
        case Border_Type::BORDER_REPLICATE:
            return 0;
        }
    }
    else if (y >= iHeight)
    {
        /*if (iThread_ID == 77803)
            printf("Here");*/
        switch (iBorder_Type)
        {

        case Border_Type::BORDER_REFLECT:
            return iHeight - (y - iHeight + 1);
        case Border_Type::BORDER_REFLECT_101:
            return iHeight - (y - iHeight + 2);
        case Border_Type::BORDER_CONSTANT:
        case Border_Type::BORDER_REPLICATE:
            return iHeight - 1;
        }
    }
    return y;
}

__device__ static int iGet_Border_x_GPU(int x, int iWidth, Border_Type iBorder_Type)
{
    if (x < 0)
    {
        switch (iBorder_Type)
        {
        case Border_Type::BORDER_REFLECT:
            return -x - 1;
        case Border_Type::BORDER_REFLECT_101:
            return -x;
        case Border_Type::BORDER_CONSTANT:
        case Border_Type::BORDER_REPLICATE:
            return 0;
        }
    }
    else if (x >= iWidth)
    {
        switch (iBorder_Type)
        {
        case Border_Type::BORDER_REFLECT:
            return iWidth - (x - iWidth + 1);
        case Border_Type::BORDER_REFLECT_101:
            return iWidth - (x - iWidth + 2);
        case Border_Type::BORDER_CONSTANT:
        case Border_Type::BORDER_REPLICATE:
            return iWidth - 1;
        }
    }
    return x;
}

template<typename _T>void Free_Stitch(Stitch<_T>* poStitch)
{
    int i;
    if (poStitch->m_pCamera)
    {
        Free(poStitch->m_pCamera);
        poStitch->m_pCamera = NULL;
    }

    if (poStitch->m_pImage_Source)
    {
        //�Ѿ��Ľ���һ��ָ��㶨
        for(int i=0;i<poStitch->m_iImage_Count;i++)
            Free_Image_GPU(&poStitch->m_pImage_Source[0]);
        Free(poStitch->m_pImage_Source);
        poStitch->m_pImage_Source = NULL;
    }
    if (poStitch->m_pSeam_Est)
    {
        //for (i = 0; i < poStitch->m_iImage_Count; i++)
        Free_Image_GPU(&poStitch->m_pSeam_Est[0]);
        //Free_GPU(poStitch->m_pSeam_Est);
        poStitch->m_pSeam_Est = NULL;
    }
    /*if (poStitch->m_pMask)
    {
        for (i = 0; i < poStitch->m_iImage_Count; i++)
            Free_Image(&poStitch->m_pMask[i]);
        Free(poStitch->m_pMask);
        poStitch->m_pMask = NULL;
    }*/
    if (poStitch->m_pImage_Match)
    {
        for (i = 0; i < poStitch->m_iImage_Count; i++)
        {
            for (int j = 0; j < poStitch->m_iImage_Count; j++)
            {
                if (poStitch->m_pImage_Match[i * poStitch->m_iImage_Count + j].m_iMatch_Count)
                    Free(poStitch->m_pImage_Match[i & poStitch->m_iImage_Count].m_pPoint_1);
            }
        }
        Free(poStitch->m_pImage_Match);
        poStitch->m_pImage_Match = NULL;
    }

    if (poStitch->m_pImage_Warp)
    {
        for (i = 0; i < poStitch->m_iImage_Count; i++)
            Free_Image_GPU(&poStitch->m_pImage_Warp[i]);
        //Free(poStitch->m_pImage_Warped);
        poStitch->m_pImage_Warp = NULL;
    }

    if (poStitch->m_pMasks_Warped)
    {
        for (i = 0; i < poStitch->m_iImage_Count; i++)
            Free_Image_GPU(&poStitch->m_pMasks_Warped[i]);
        //Free(poStitch->m_pMasks_Warped);
        //poStitch->m_pMasks_Warped = NULL;
    }

    //if (poStitch->m_pBlock_Images_GPU)//ÿһ��Image�ֳ����ٸ�Title
    //{
    //    for (i = 0; i < poStitch->m_iBlock_Count; i++)
    //        Free_Image(&poStitch->m_pBlock_images[i]);
    //    Free(poStitch->m_pBlock_images);
    //    poStitch->m_pBlock_images = NULL;
    //}

    if (poStitch->m_pBlock_Masks)//ÿ��Image�ֳ����ٸ�mask
    {
        for (i = 0; i < poStitch->m_iBlock_Count; i++)
            Free_Image(&poStitch->m_pBlock_Masks[i]);
        Free(poStitch->m_pBlock_Masks);
        poStitch->m_pBlock_Masks = NULL;
    }

    /*if (poStitch->m_pBlock_Corners)
    {
        Free(poStitch->m_pBlock_corners);
        poStitch->m_pBlock_corners = NULL;
    }*/

    /*if (poStitch->m_pBlock_per_imgs)
    {
        Free(poStitch->m_pBlock_per_imgs);
        poStitch->m_pBlock_per_imgs = NULL;
    }*/
    /*if (poStitch->m_pCorner)
    {
        Free(poStitch->m_pCorner);
        poStitch->m_pCorner = NULL;
    }*/
    if (poStitch->m_pImage_Source_Header_GPU)
        Free_GPU(poStitch->m_pImage_Source_Header_GPU);
    if (poStitch->m_pBlock_Corner)
        Free(poStitch->m_pBlock_Corner);
    if (poStitch->m_pBlock_Corner_GPU)
        Free_GPU(poStitch->m_pBlock_Corner_GPU);
    if (poStitch->m_pBlock_Image)
        Free(poStitch->m_pBlock_Image);
    if (poStitch->m_pBlock_Image_Header_GPU)
        Free_GPU(poStitch->m_pBlock_Image_Header_GPU);
    if (poStitch->m_pBlock_Image_Data_GPU)
        Free_GPU(poStitch->m_pBlock_Image_Data_GPU);
    if (poStitch->m_oComp.m_pGain)
        Free_GPU(poStitch->m_oComp.m_pGain);

    return;
}

template<typename _T>void Set_Scale(Stitch<_T>* poStitch)
{//ƴ�������д���scale���ܹ�һ����һ��

    //������׼��
    poStitch->registr_resol_ = 0.3;

    //��workscale����registr_resol_����
    //work_scale_ = std::min(1.0, std::sqrt(registr_resol_ * 1e6 / full_img_sizes_[i].area()));
    int iArea = poStitch->m_Source_Size[0] * poStitch->m_Source_Size[1];
    _T fValue = poStitch->registr_resol_ * 1e6 / (_T)iArea;
    fValue = sqrt(fValue);  
    poStitch->work_scale = Min(1.0, fValue);
    poStitch->m_bWork_Scale_Set = 1;

    //��seam_scale_/ seam_work_aspect_
    //seam_scale_ = std::min(1.0, std::sqrt(seam_est_resol_ * 1e6 / full_img_sizes_[i].area()));
    poStitch->seam_est_resol_ = 0.1;    //���Դ�ֵ
    fValue = poStitch->seam_est_resol_ * 1e6 / (_T)iArea;
    fValue = sqrt(fValue);
    poStitch->seam_scale = Min(1.0, fValue);
    poStitch->seam_work_aspect = poStitch->seam_scale / poStitch->work_scale;
    poStitch->m_bSeam_Scale_set = 1;
    poStitch->m_Seam_Size[0] = (short)(poStitch->m_Source_Size[0] * poStitch->seam_scale + 0.5f);
    poStitch->m_Seam_Size[1] = (short)(poStitch->m_Source_Size[1] * poStitch->seam_scale + 0.5f);

    //����warped_image_scale
    //�ɼ���warped_image_scale����һ����������ȵ�scale, ���Ǹ�������Ľ��������ֵ
        
    //�˴��и���ֵ��㷨���ݲ�֪���⣬ԭ���Ĳ���Բ���cv������֤
    //_T fTotal = 0;
    //for (int i = 0; i < poStitch->m_iImage_Count; i++)
    //    fTotal += poStitch->m_pCamera[i].K[0];
    //poStitch->warped_image_scale = fTotal/poStitch->m_iImage_Count;
    if (poStitch->m_iImage_Count & 1)
    {//ż��ʱ��ȡ�м�֡�Ľ���Ϊwarped_image_scale
        poStitch->warped_image_scale = poStitch->m_pCamera[poStitch->m_iImage_Count >> 1].K[0];
    }else
    {
        int iHalf = poStitch->m_iImage_Count >> 1;
        _T fTotal = poStitch->m_pCamera[iHalf - 1].K[0] + poStitch->m_pCamera[iHalf].K[0];
        poStitch->warped_image_scale = fTotal / 2;
    }

    return;
}
template<typename _T>void Init_Stitch(Stitch<_T>* poStitch, int iWidth, int iHeight)
{//iWidth,iHeightΪԭͼ�ķֱ���
    int i, iSize;
    Stitch<_T> oSt = *poStitch;
    Light_Ptr oPtr_GPU;
    unsigned char* p;

    //ͳһ���ø���scale
    oSt.m_Source_Size[0] = iWidth, oSt.m_Source_Size[1] = iHeight;
    Set_Scale(&oSt);
   
    //�ȸ�CPU
    //���Ƿ����Դ��ͼ��ͷ
    int iSize_Header = oSt.m_iImage_Count * sizeof(Image) * (1 +        //Source
        1 +     //Seam_est
        1 +     //Mask
        1 +     //Image_Warped
        1) +       //Mask_Warped
        oSt.m_iImage_Count * sizeof(Image_Size_In_Block) +    //Block_Per_Image
        oSt.m_iImage_Count * 2 * 2 * sizeof(int) + //Corner
        oSt.m_iImage_Count * 2 * sizeof(int) ;      //Size;

    p = (unsigned char*)pMalloc(iSize_Header);
    memset(p, 0, iSize_Header);
    
    oSt.m_pImage_Source = (Image*)p;
    oSt.m_pSeam_Est = oSt.m_pImage_Source + oSt.m_iImage_Count;
    oSt.m_pMask = oSt.m_pSeam_Est + oSt.m_iImage_Count;
    oSt.m_pImage_Warp = oSt.m_pMask + oSt.m_iImage_Count;
    oSt.m_pMasks_Warped = oSt.m_pImage_Warp + oSt.m_iImage_Count;
    oSt.m_pBlock_Per_Image = (Image_Size_In_Block*)(oSt.m_pMasks_Warped + oSt.m_iImage_Count);
    oSt.m_pCorner = (int(*)[2][2])(oSt.m_pBlock_Per_Image + oSt.m_iImage_Count);
    oSt.m_pSize = (int(*)[2])(oSt.m_pCorner + oSt.m_iImage_Count);

    //�ɷ�������
    oSt.m_pKer_GPU = (_T*)(oSt.m_pCorner + oSt.m_iImage_Count);
    _T Ker[] = { 0.25, 0.5, 0.25,0,0 };
    hipMemcpy(oSt.m_pKer_GPU, Ker, 3 * sizeof(_T), hipMemcpyHostToDevice);

    iSize = iSize_Header +
        oSt.m_iImage_Count * (iWidth * iHeight * 4 + 128);                //Source Image

    Attach_Light_Ptr(oPtr_GPU, (unsigned char*)pMalloc_GPU(iSize), iSize, 0);
    p = (unsigned char*)pMalloc_GPU(iSize_Header);
    oSt.m_pImage_Source_Header_GPU = (Image*)p;
    oSt.m_pSeam_Est_Header_GPU = oSt.m_pImage_Source_Header_GPU + oSt.m_iImage_Count;
    oSt.m_pMask_Header_GPU = oSt.m_pSeam_Est_Header_GPU + oSt.m_iImage_Count;
    oSt.m_pImage_Warp_Header_GPU = oSt.m_pMask_Header_GPU + oSt.m_iImage_Count;
    oSt.m_pMask_Warp_Header_GPU = oSt.m_pImage_Warp_Header_GPU + oSt.m_iImage_Count;
    oSt.m_pBlock_Per_Image_GPU = (Image_Size_In_Block*)(oSt.m_pMask_Warp_Header_GPU + oSt.m_iImage_Count);
    oSt.m_pCorner_GPU =(int (*)[2][2]) oSt.m_pBlock_Per_Image_GPU + oSt.m_iImage_Count;
    for (i = 0; i < poStitch->m_iImage_Count; i++)
    {
        Init_Image_GPU(&oSt.m_pImage_Source[i], iWidth, iHeight, Image::IMAGE_TYPE_BMP, 32, &oPtr_GPU);
        hipMemset(oSt.m_pImage_Source[i].m_pChannel[3], 255, iWidth * iHeight * 3);
    }

    //��Щ�ں�����ͷţ�������һ���Դ� Seam_est + Seam_Mask + Image_Warp
    iSize = oSt.m_iImage_Count * (oSt.m_Seam_Size[0] * oSt.m_Seam_Size[1] * 3 + 128) +      //Seam_est
        oSt.m_iImage_Count * (oSt.m_Seam_Size[0] * oSt.m_Seam_Size[1] + 128) ;               //Seam_Mask
    Attach_Light_Ptr(oPtr_GPU, (unsigned char*)pMalloc_GPU(iSize), iSize, 0);
   
    //Seam_est + Seam_Mask + Image_Warp
    for (i = 0; i < poStitch->m_iImage_Count; i++)
        Init_Image_GPU(&oSt.m_pSeam_Est[i], oSt.m_Seam_Size[0], oSt.m_Seam_Size[1], Image::IMAGE_TYPE_BMP, 24, &oPtr_GPU);
    for (i = 0; i < poStitch->m_iImage_Count; i++)
    {
        Init_Image_GPU(&oSt.m_pMask[i], oSt.m_Seam_Size[0], oSt.m_Seam_Size[1], Image::IMAGE_TYPE_BMP, 8, &oPtr_GPU);
        Set_Color_GPU(oSt.m_pMask[i],255);
        //Disp_Cuda_Error();
    }

    //��������������ͨ��scale�������Image���ٿռ�
    hipMemcpy(p, oSt.m_pImage_Source, iSize_Header,hipMemcpyHostToDevice);
        

    *poStitch = oSt;
    //Disp_Cuda_Error();
    return;
}
template<typename _T>void Map_Forward(float x, float y, float* pu, float* pv, Sphere_Projector<_T> oProjector, Point_Cloud<float>* poPC = NULL)
{
    static int iCount = 0;
    //������(x,y,1),�󾭹�ͶӰ�������
    //(x_,y_,z_)' = R * K(-1) * (x,y,1)'

    //ͶӰ��
    float x_ = (float)(oProjector.r_kinv[0] * x + oProjector.r_kinv[1] * y + oProjector.r_kinv[2]);
    float y_ = (float)(oProjector.r_kinv[3] * x + oProjector.r_kinv[4] * y + oProjector.r_kinv[5]);
    float z_ = (float)(oProjector.r_kinv[6] * x + oProjector.r_kinv[7] * y + oProjector.r_kinv[8]);

    if (poPC)
    {
        if (iCount < 1278)
            Draw_Point(poPC, x_, y_, z_, 255, 0, 0);
        else
            Draw_Point(poPC, x_, y_, z_, 0, 255, 0);
    }

    *pu = (float)(oProjector.scale * atan2f(x_, z_));
    float w = y_ / sqrtf(x_ * x_ + y_ * y_ + z_ * z_);
    *pv = (float)(oProjector.scale * (PI - acosf(w == w ? w : 0)));

    iCount++;
    return;
}
template<typename _T>void Detect_Result_Roi_By_Border(int w, int h, int Dest_tl[2], int Dest_br[2],
    Sphere_Projector<_T> oProjector, Point_Cloud<float>* poPC = NULL)
{//tl: Top left     br: Bottom Right
    static int iCount = 0;
    float tl_uf = (std::numeric_limits<float>::max)();
    float tl_vf = (std::numeric_limits<float>::max)();
    float br_uf = -(std::numeric_limits<float>::max)();
    float br_vf = -(std::numeric_limits<float>::max)();
    float u, v;

    //�˴�Ҳ����ɨ��һ�λ��������������ģ����������ĵ���˼
    for (int x = 0; x < w; x++)
    {
        Map_Forward((float)x, 0, &u, &v, oProjector, poPC);
        tl_uf = Min(tl_uf, u);
        tl_vf = Min(tl_vf, v);
        br_uf = Max(br_uf, u);
        br_vf = Max(br_vf, v);

        Map_Forward((float)x, (float)(h - 1), &u, &v, oProjector, poPC);
        tl_uf = Min(tl_uf, u);
        tl_vf = Min(tl_vf, v);
        br_uf = Max(br_uf, u);
        br_vf = Max(br_vf, v);
        //printf("x:%f %f %f %f %f\n", (float)x, tl_uf, tl_vf, br_uf, br_vf);
    }
    for (int y = 0; y < h; y++)
    {
        Map_Forward(0, (float)y, &u, &v, oProjector, poPC);
        tl_uf = Min(tl_uf, u); tl_vf = Min(tl_vf, v);
        br_uf = Max(br_uf, u); br_vf = Max(br_vf, v);

        Map_Forward((float)(w - 1), (float)y, &u, &v, oProjector, poPC);
        tl_uf = Min(tl_uf, u); tl_vf = Min(tl_vf, v);
        br_uf = Max(br_uf, u); br_vf = Max(br_vf, v);
    }
    //bSave_PLY<float>("c:\\tmp\\1.ply", oPC);

    Dest_tl[0] = (int)tl_uf;
    Dest_tl[1] = (int)tl_vf;
    Dest_br[0] = (int)br_uf;
    Dest_br[1] = (int)br_vf;
    iCount++;
    return;
}
template<typename _T>void Detect_Result_Roi(int w, int h, int Dest_tl[2], int Dest_br[2],
    Sphere_Projector<_T> oProjector, Point_Cloud<float>* poPC = NULL)
{//tl: Top left     br: Bottom Right
    static int iCount = 0;

    Detect_Result_Roi_By_Border(w, h, Dest_tl, Dest_br, oProjector, poPC);
    float tl_uf = (float)Dest_tl[0];
    float tl_vf = (float)Dest_tl[1];
    float br_uf = (float)Dest_br[0];
    float br_vf = (float)Dest_br[1];

    float x = (float)oProjector.rinv[1];
    float y = (float)oProjector.rinv[4];
    float z = (float)oProjector.rinv[7];
    if (y > 0.f)
    {
        float x_ = (float)((oProjector.K[0] * x + oProjector.K[1] * y) / z + oProjector.K[2]);
        float y_ = (float)(oProjector.K[4] * y / z + oProjector.K[5]);

        if (x_ > 0.f && x_ < w && y_ > 0.f && y_ < h)
        {
            tl_uf = Min(tl_uf, 0.f); tl_vf = std::min(tl_vf, float(PI * oProjector.scale));
            br_uf = Max(br_uf, 0.f); br_vf = std::max(br_vf, float(PI * oProjector.scale));
        }
    }

    x = (float)oProjector.rinv[1];
    y = -(float)oProjector.rinv[4];
    z = (float)oProjector.rinv[7];

    if (y > 0.f)
    {
        float x_ = (float)((oProjector.K[0] * x + oProjector.K[1] * y) / z + oProjector.K[2]);
        float y_ = (float)(oProjector.K[4] * y / z + oProjector.K[5]);
        if (x_ > 0.f && x_ < w && y_ > 0.f && y_ < h)
        {
            tl_uf = Min(tl_uf, 0.f); tl_vf = Min(tl_vf, 0);
            br_uf = Max(br_uf, 0.f); br_vf = Max(br_vf, 0);
        }
    }
    Dest_tl[0] = (int)tl_uf;
    Dest_tl[1] = (int)tl_vf;
    Dest_br[0] = (int)br_uf;
    Dest_br[1] = (int)br_vf;
    iCount++;
    return;
}

template<typename _T>void Set_Camera_Params(Sphere_Projector<_T>* poProjector, _T _K[3 * 3], _T _R[3 * 3], _T t[3])
{
    static int iCount = 0;
    memcpy(poProjector->K, _K, 3 * 3 * sizeof(_T));
    //Disp(_K, 3, 3, "K");
    //Disp(_R, 3, 3, "R");*/
     //Disp(t, 1, 3, "t");

     //��R��һ��ת��
    Matrix_Transpose(_R, 3, 3, poProjector->rinv);
    //R_Kinv = R * K.inv();
    _T K_inv[3 * 3];
    int iResult = 0;
    Get_Inv_Matrix_Row_Op(_K, K_inv, 3, &iResult);
    /*if (iCount == 4)
        Disp(poProjector->rinv, 3, 3, "rinv");*/

    Matrix_Multiply_3x3(_R, K_inv, poProjector->r_kinv);
    /*if (iCount == 4)
        Disp(_K, 3, 3, "K");*/

        //K_Rinv = K * Rinv;
    Matrix_Multiply_3x3(_K, poProjector->rinv, poProjector->k_rinv);

    if (t)
        memcpy(poProjector->t, t, 3 * sizeof(_T));
    else
        memset(poProjector->t, 0, 3 * sizeof(_T));
    iCount++;
}

template<typename _T> void Build_Map(int w, int h, _T K[], _T R[], _T fScale, 
    /*_T** ppx_Map, _T** ppy_Map,*/
    int Dest_roi[2][2], Sphere_Projector<_T>* poProjector, Point_Cloud<float>* poPC)
{//Dest_roi: [0] Ϊ���Ͻǣ� [1]Ϊ w,h
    static int iCount = 0;

    Sphere_Projector<_T> oProjector = {};

    //ÿ�ε���ǰ���뽫scale����
    oProjector.scale = fScale;  // 530.474915;   //529.692383;  //529.69240946121784;
    Set_Camera_Params(&oProjector, K, R, (_T*)NULL);

    int Dest_tl[2], Dest_br[2];
    Detect_Result_Roi(w, h, Dest_tl, Dest_br, oProjector, poPC);

    Dest_roi[0][0] = Dest_tl[0];
    Dest_roi[0][1] = Dest_tl[1];

    Dest_roi[1][0] = Dest_br[0] - Dest_tl[0];
    Dest_roi[1][1] = Dest_br[1] - Dest_tl[1];

    *poProjector = oProjector;
    iCount++;
    return;
}

__device__ static void Pix_Inter_2_GPU(Image::Part_1 oImage, float x, float y, unsigned char Pix[4],
    Border_Type iImage_Border_Type, Border_Type iMask_Border_Type, int iThread_ID = 0)
{//Bi_Linear + Nearest ����һ
    short x1 = floor(x),
        y1 = floor(y);
    //h_1 = oImage.m_iHeight - 1,
    //w_1 = oImage.m_iWidth - 1;
    unsigned char A, B, C, D;
    int iCur_Line_Pos, iNext_Line_Pos;
    short xl_Pos, xr_Pos;

    //�ȸ�Image_Warp�Ĳ�ֵ
    if (iImage_Border_Type == BORDER_CONSTANT)
    {//���������
        iCur_Line_Pos = iNext_Line_Pos = -1;
        short y2 = y1;
        if (y2 >= 0 && y2 < oImage.m_iHeight)
            iCur_Line_Pos = y2 * oImage.m_iWidth;
        y2++;
        if (y2 >= 0 && y2 < oImage.m_iHeight)
            iNext_Line_Pos = y2 * oImage.m_iWidth;

        xl_Pos = xr_Pos = -1;
        short x2 = x1;
        if (x2 >= 0 && x2 < oImage.m_iWidth)
            xl_Pos = x2;
        x2++;
        if (x2 >= 0 && x2 < oImage.m_iWidth)
            xr_Pos = x2;
    }
    else
    {
        iCur_Line_Pos = iGet_Border_y_GPU(y1, oImage.m_iHeight, iImage_Border_Type) * oImage.m_iWidth;
        iNext_Line_Pos = iGet_Border_y_GPU(y1 + 1, oImage.m_iHeight, iImage_Border_Type) * oImage.m_iWidth;
        xl_Pos = iGet_Border_x_GPU(x1, oImage.m_iWidth, iImage_Border_Type);
        xr_Pos = iGet_Border_x_GPU(x1 + 1, oImage.m_iWidth, iImage_Border_Type);
    }

    for (int i = 0; i < 3; i++)
    {
        if (oImage.m_pChannel[i])
        {
            if (iCur_Line_Pos >= 0)
            {
                A = xl_Pos >= 0 ? oImage.m_pChannel[i][iCur_Line_Pos + xl_Pos] : 0;
                B = xr_Pos >= 0 ? oImage.m_pChannel[i][iCur_Line_Pos + xr_Pos] : 0;
            }
            else
                A = B = 0;
            if (iNext_Line_Pos >= 0)
            {
                C = xl_Pos >= 0 ? oImage.m_pChannel[i][iNext_Line_Pos + xl_Pos] : 0;
                D = xr_Pos >= 0 ? oImage.m_pChannel[i][iNext_Line_Pos + xr_Pos] : 0;
            }
            else
                C = D = 0;

            float fValue_0, fValue_1;
            {
                float w1 = x - x1, w0 = 1.f - w1;
                fValue_0 = w0 * A + w1 * B;
                fValue_1 = w0 * C + w1 * D;
                //if (iThread_ID == 2466)
                    //printf("%d %d %d %d\n", iCur_Line_Pos, iNext_Line_Pos, xl_Pos, xr_Pos);
            }

            {
                float w3 = y - y1, w2 = 1.f - w3;
                Pix[i] = (unsigned char)(w2 * fValue_0 + w3 * fValue_1 + 0.5f);
            }
        }
    }

    //�ٸ�Mask�Ĳ�ֵ
    if (iMask_Border_Type == BORDER_CONSTANT)
    {//���������
        iCur_Line_Pos = iNext_Line_Pos = -1;
        short y2 = y1;
        if (y2 >= 0 && y2 < oImage.m_iHeight)
            iCur_Line_Pos = y2 * oImage.m_iWidth;
        y2++;
        if (y2 >= 0 && y2 < oImage.m_iHeight)
            iNext_Line_Pos = y2 * oImage.m_iWidth;

        xl_Pos = xr_Pos = -1;
        short x2 = x1;
        if (x2 >= 0 && x2 < oImage.m_iWidth)
            xl_Pos = x2;
        x2++;
        if (x2 >= 0 && x2 < oImage.m_iWidth)
            xr_Pos = x2;
    }
    else
    {
        iCur_Line_Pos = iGet_Border_y_GPU(y1, oImage.m_iHeight, iMask_Border_Type) * oImage.m_iWidth;
        iNext_Line_Pos = iGet_Border_y_GPU(y1 + 1, oImage.m_iHeight, iMask_Border_Type) * oImage.m_iWidth;
        xl_Pos = iGet_Border_x_GPU(x1, oImage.m_iWidth, iMask_Border_Type);
        xr_Pos = iGet_Border_x_GPU(x1 + 1, oImage.m_iWidth, iMask_Border_Type);
    }

    //�ж�ȡ�Ǹ�λ�ø���
    int iLine_Pos;
    short x_Pos;
    if (y - y1 <= 0.5f)
        iLine_Pos = iCur_Line_Pos;
    else
        iLine_Pos = iNext_Line_Pos;

    if (x - x1 <= 0.5f)
        x_Pos = xl_Pos;
    else
        x_Pos = xr_Pos;

    if (iLine_Pos >= 0 && x_Pos >= 0)
        Pix[3] = oImage.m_pChannel[3][iLine_Pos + x_Pos];
    else
        Pix[3] = 0;
}

template<typename _T> __global__ void _Re_Map_2_GPU(Image::Part_1 oImage, Image::Part_1 oImage_Warped,
    short roi_x, short roi_y, float fScale, K_Rinv<_T> oK_Rinv,
    Border_Type iImage_Border_Type, Border_Type iMask_Border_Type = BORDER_CONSTANT)
{
    int iThread_ID = GET_THREAD_ID();
    if (iThread_ID >= oImage_Warped.m_iWidth * oImage_Warped.m_iHeight)
        return;
    float x, y;
    {
        //����u,v�Ǵ�Dest��x,y �ָ��������������
        float u = fScale * (int)(iThread_ID % oImage_Warped.m_iWidth + roi_x),
            v = fScale * (int)(iThread_ID / oImage_Warped.m_iWidth + roi_y);

        float sinv = sinf((float)(PI - v));
        float x_ = sinv * sinf(u);
        float y_ = (float)cosf((float)(PI - v));
        float z_ = sinv * cosf(u);

        float z;
        x = (float)(oK_Rinv.M[0] * x_ + oK_Rinv.M[1] * y_ + oK_Rinv.M[2] * z_);
        y = (float)(oK_Rinv.M[3] * x_ + oK_Rinv.M[4] * y_ + oK_Rinv.M[5] * z_);
        z = (float)(oK_Rinv.M[6] * x_ + oK_Rinv.M[7] * y_ + oK_Rinv.M[8] * z_);

        if (z > 0)
            x /= z, y /= z;
        else
            x = y = -1;
    }

    unsigned char Pix[4];
    Pix_Inter_2_GPU(oImage, x, y, Pix, iImage_Border_Type, iMask_Border_Type, iThread_ID);
    oImage_Warped.m_pChannel[0][iThread_ID] = Pix[0];
    oImage_Warped.m_pChannel[1][iThread_ID] = Pix[1];
    oImage_Warped.m_pChannel[2][iThread_ID] = Pix[2];
    oImage_Warped.m_pChannel[3][iThread_ID] = Pix[3];

    /*if (iThread_ID == 100 * oImage_Warped.m_iWidth + 100)
        printf("%d\n", Pix[3]);*/

}

template<typename _T>void Re_Map_3_GPU(Image oImage, Image oImage_Warped,
    int roi_x, int roi_y, Sphere_Projector<_T>* poProjector,
    Interpolation_Flag iImage_Inter_Type, Interpolation_Flag iMask_Inter_Type,
    Border_Type iImage_Border_Type, Border_Type iMask_Border_Type)
{//Image, Maskһ�����
    static int iCount = 0;
    int iThread_Per_Block = 512;
    dim3 oGrid;
    oGrid.x = (oImage_Warped.m_iWidth * oImage_Warped.m_iHeight + iThread_Per_Block - 1) / iThread_Per_Block;
    oGrid.y = 1;    //oDest.m_iChannel_Count;
    oGrid.z = 1;

    _Re_Map_2_GPU<_T> << <oGrid, iThread_Per_Block >> > (oImage.m_oPart_1, oImage_Warped.m_oPart_1, roi_x, roi_y,
        (float)(1.f / poProjector->scale), poProjector->oK_Rinv, iImage_Border_Type, iMask_Border_Type);

    iCount++;
    return;
}

template<typename _T>void Re_Map_2_GPU(Image oImage, Image oMask,
    Image oImage_Warped, Image oMask_Warped,
    int roi_x, int roi_y, Sphere_Projector<_T>* poProjector,
    Interpolation_Flag iImage_Inter_Type, Interpolation_Flag iMask_Inter_Type,
    Border_Type iImage_Border_Type, Border_Type iMask_Border_Type)
{//Image, Maskһ�����
    static int iCount = 0;
    int iThread_Per_Block = 512;
    dim3 oGrid;
    oGrid.x = (oImage_Warped.m_iWidth * oImage_Warped.m_iHeight + iThread_Per_Block - 1) / iThread_Per_Block;
    oGrid.y = 1;    //oDest.m_iChannel_Count;
    oGrid.z = 1;

    //С���ɣ���Mask��ΪImage��Alphaͨ��
    oImage.m_pChannel[3] = oMask.m_pChannel[0];
    oImage_Warped.m_pChannel[3] = oMask_Warped.m_pChannel[0];
    _Re_Map_2_GPU<_T> << <oGrid, iThread_Per_Block >> > (oImage.m_oPart_1, oImage_Warped.m_oPart_1, roi_x, roi_y,
        (float)(1.f / poProjector->scale), poProjector->oK_Rinv, iImage_Border_Type, iMask_Border_Type);
    //oImage.m_pChannel[3] = oImage_Warped.m_pChannel[3] = NULL;
    //Disp_Cuda_Error();
    /*Disp_Part_GPU(oImage_Warped.m_pChannel[3], oImage_Warped.m_iWidth, 100, 100, 2, 2);
    Disp_Part_GPU(oMask_Warped.m_pChannel[0], oMask_Warped.m_iWidth, 100, 100, 2, 2);*/

    /*bSave_Image_GPU("c:\\tmp\\1.bmp", oImage_Warped);
    bSave_Image_GPU("c:\\tmp\\2.bmp", oMask_Warped);*/
    iCount++;
    return;
}

template<typename _T>void Warp_2(Image oImage, Image oMask, _T K[3 * 3], _T R[3 * 3], _T fScale,
    Image* poImage_Warped, Image* poMask_Warped,Interpolation_Flag iImage_Inter_Type, Interpolation_Flag iMask_Inter_Type,
    int Corner[2][2], Border_Type iImage_Border_Type, Border_Type iMask_Border_Type, Point_Cloud<float>* poPC)
{//��ͼ��Mask��ͶӰ�϶�Ϊһ��ʵ���ϣ����ֻ�����Ż��Ѿ������˳��������ԣ�
    //����û���κ����ü�ֵ�������룬Ψ���죡
    static int iCount = 0;
    //_T* puxmap, * puymap;    //��ʱδ֪
    int Dest_roi[2][2];   //[0][0-1]: x,y [1][0-1]: w,h
    Sphere_Projector<_T> oProjector = {};
    Build_Map<_T>(oImage.m_iWidth, oImage.m_iHeight, K, R, fScale, /*&puxmap, &puymap,*/ Dest_roi, &oProjector, poPC);
    Init_Image_GPU(poImage_Warped, Dest_roi[1][0] + 1, Dest_roi[1][1] + 1, Image::IMAGE_TYPE_BMP, oImage.m_iBit_Count);
    Init_Image_GPU(poMask_Warped, Dest_roi[1][0] + 1, Dest_roi[1][1] + 1, Image::IMAGE_TYPE_BMP, 8);
    Re_Map_2_GPU<_T>(oImage, oMask, *poImage_Warped, *poMask_Warped,
        Dest_roi[0][0], Dest_roi[0][1], &oProjector,
        iImage_Inter_Type, iMask_Inter_Type, iImage_Border_Type, iMask_Border_Type);

    poImage_Warped->m_pChannel[3] = poMask_Warped->m_pChannel[0];
    if (Corner)
    {
        Corner[0][0] = Dest_roi[0][0];
        Corner[0][1] = Dest_roi[0][1];
        Corner[1][0] = Dest_roi[0][0] + Dest_roi[1][0];
        Corner[1][1] = Dest_roi[0][1] + Dest_roi[1][1];
    }
    iCount++;
    return;
}

__device__ void Pix_Bi_Linear_GPU(Image::Part_1 oImage, float x, float y, unsigned char Pix[3], Border_Type iBorder_Type, int iThread_ID = 0)
{
    short x1 = floor(x),
        y1 = floor(y);
    //h_1 = oImage.m_iHeight - 1,
    //w_1 = oImage.m_iWidth - 1;
    unsigned char A, B, C, D;
    int iCur_Line_Pos, iNext_Line_Pos;
    short xl_Pos, xr_Pos;
    //int iSize = oImage.m_iWidth * oImage.m_iHeight;

    if (iBorder_Type == BORDER_CONSTANT)
    {//���������
        iCur_Line_Pos = iNext_Line_Pos = -1;
        short y2 = y1;
        if (y2 >= 0 && y2 < oImage.m_iHeight)
            iCur_Line_Pos = y2 * oImage.m_iWidth;
        y2++;
        if (y2 >= 0 && y2 < oImage.m_iHeight)
            iNext_Line_Pos = y2 * oImage.m_iWidth;

        xl_Pos = xr_Pos = -1;
        short x2 = x1;
        if (x2 >= 0 && x2 < oImage.m_iWidth)
            xl_Pos = x2;
        x2++;
        if (x2 >= 0 && x2 < oImage.m_iWidth)
            xr_Pos = x2;
    }
    else
    {
        iCur_Line_Pos = iGet_Border_y_GPU(y1, oImage.m_iHeight, iBorder_Type) * oImage.m_iWidth;
        iNext_Line_Pos = iGet_Border_y_GPU(y1 + 1, oImage.m_iHeight, iBorder_Type) * oImage.m_iWidth;
        xl_Pos = iGet_Border_x_GPU(x1, oImage.m_iWidth, iBorder_Type);
        xr_Pos = iGet_Border_x_GPU(x1 + 1, oImage.m_iWidth, iBorder_Type);
    }

    for (int i = 0; i < 3; i++)
    {
        if (oImage.m_pChannel[i])
        {
            if (iCur_Line_Pos >= 0)
            {
                A = xl_Pos >= 0 ? oImage.m_pChannel[i][iCur_Line_Pos + xl_Pos] : 0;
                B = xr_Pos >= 0 ? oImage.m_pChannel[i][iCur_Line_Pos + xr_Pos] : 0;
            }
            else
                A = B = 0;
            if (iNext_Line_Pos >= 0)
            {
                C = xl_Pos >= 0 ? oImage.m_pChannel[i][iNext_Line_Pos + xl_Pos] : 0;
                D = xr_Pos >= 0 ? oImage.m_pChannel[i][iNext_Line_Pos + xr_Pos] : 0;
            }
            else
                C = D = 0;

            float fValue_0, fValue_1;
            {
                float w1 = x - x1, w0 = 1.f - w1;
                fValue_0 = w0 * A + w1 * B;
                fValue_1 = w0 * C + w1 * D;
                //if (iThread_ID == 2466)
                    //printf("%d %d %d %d\n", iCur_Line_Pos, iNext_Line_Pos, xl_Pos, xr_Pos);
            }

            {
                float w3 = y - y1, w2 = 1.f - w3;
                Pix[i] = (unsigned char)(w2 * fValue_0 + w3 * fValue_1 + 0.5f);
            }
        }
    }
    return;
}

template<typename _T> __global__ void _Re_Map_Bi_Binear_GPU(Image::Part_1 oSource, Image::Part_1 oDest, short roi_x,
    short roi_y, float fScale, K_Rinv<_T> oK_Rinv, Border_Type iBorder_Type = BORDER_REFLECT)
{
    int iThread_ID = GET_THREAD_ID();
    if (iThread_ID >= oDest.m_iWidth * oDest.m_iHeight)
        return;
    float x, y;
    {
        //����u,v�Ǵ�Dest��x,y �ָ��������������
        float u = fScale * (int)(iThread_ID % oDest.m_iWidth + roi_x),
            v = fScale * (int)(iThread_ID / oDest.m_iWidth + roi_y);

        float sinv = sinf((float)(PI - v));
        float x_ = sinv * sinf(u);
        float y_ = (float)cosf((float)(PI - v));
        float z_ = sinv * cosf(u);

        float z;
        x = (float)(oK_Rinv.M[0] * x_ + oK_Rinv.M[1] * y_ + oK_Rinv.M[2] * z_);
        y = (float)(oK_Rinv.M[3] * x_ + oK_Rinv.M[4] * y_ + oK_Rinv.M[5] * z_);
        z = (float)(oK_Rinv.M[6] * x_ + oK_Rinv.M[7] * y_ + oK_Rinv.M[8] * z_);

        if (z > 0)
            x /= z, y /= z;
        else
            x = y = -1;
    }

    unsigned char Pix[3];
    Pix_Bi_Linear_GPU(oSource, x, y, Pix, iBorder_Type, iThread_ID);
    oDest.m_pChannel[0][iThread_ID] = Pix[0];
    oDest.m_pChannel[1][iThread_ID] = Pix[1];
    oDest.m_pChannel[2][iThread_ID] = Pix[2];

    //oDest.m_pChannel[blockIdx.y][iThread_ID] = (unsigned char)(w2 * fValue_0 + w3 * fValue_1 + 0.5);
    //if (iThread_ID == 6 * oDest.m_iWidth + 312)
    //{
    //    //printf("x:%f y:%f\n", x, y);
    //    //printf("%d %d %d %d\n", Pix[0],Pix[1],Pix[2],iThread_ID);
    //    //printf("%f %f %d\n", fValue_0, fValue_1, oDest.m_pChannel[blockIdx.y][iThread_ID]);
    //}

    return;
}

__device__ void Pix_Nearest_GPU(Image::Part_1 oImage, float x, float y, unsigned char* Pix, Border_Type iBorder_Type, int iThread_ID = 0)
{
    short x1 = floor(x),
        y1 = floor(y);

    int iCur_Line_Pos, iNext_Line_Pos;
    short xl_Pos, xr_Pos;

    if (iBorder_Type == BORDER_CONSTANT)
    {//���������
        iCur_Line_Pos = iNext_Line_Pos = -1;
        short y2 = y1;
        if (y2 >= 0 && y2 < oImage.m_iHeight)
            iCur_Line_Pos = y2 * oImage.m_iWidth;
        y2++;
        if (y2 >= 0 && y2 < oImage.m_iHeight)
            iNext_Line_Pos = y2 * oImage.m_iWidth;

        xl_Pos = xr_Pos = -1;
        short x2 = x1;
        if (x2 >= 0 && x2 < oImage.m_iWidth)
            xl_Pos = x2;
        x2++;
        if (x2 >= 0 && x2 < oImage.m_iWidth)
            xr_Pos = x2;
    }
    else
    {
        iCur_Line_Pos = iGet_Border_y_GPU(y1, oImage.m_iHeight, iBorder_Type) * oImage.m_iWidth;
        iNext_Line_Pos = iGet_Border_y_GPU(y1 + 1, oImage.m_iHeight, iBorder_Type) * oImage.m_iWidth;
        xl_Pos = iGet_Border_x_GPU(x1, oImage.m_iWidth, iBorder_Type);
        xr_Pos = iGet_Border_x_GPU(x1 + 1, oImage.m_iWidth, iBorder_Type);
    }

    //�ж�ȡ�Ǹ�λ�ø���
    int iLine_Pos;
    short x_Pos;
    if (y - y1 <= 0.5f)
        iLine_Pos = iCur_Line_Pos;
    else
        iLine_Pos = iNext_Line_Pos;

    /*if (iThread_ID == 35300)
    {
        printf("%d %d %f\n", iLine_Pos, x_Pos, y - y1);
    }*/

    if (x - x1 <= 0.5f)
        x_Pos = xl_Pos;
    else
        x_Pos = xr_Pos;

    if (iLine_Pos >= 0 && x_Pos >= 0)
    {
        int iPos = iLine_Pos + x_Pos;
        for (int i = 0; i < 3; i++)
            if (oImage.m_pChannel[i])
                Pix[i] = oImage.m_pChannel[i][iPos];
    }
    else
        Pix[0] = Pix[1] = Pix[2] = 0;

}
template<typename _T> __global__ void _Re_Map_Nearest_GPU(Image::Part_1 oSource, Image::Part_1 oDest, short roi_x,
    short roi_y, float fScale, K_Rinv<_T> oK_Rinv, Border_Type iBorder_Type = BORDER_REFLECT)
{
    int iThread_ID = GET_THREAD_ID();
    if (iThread_ID >= oDest.m_iWidth * oDest.m_iHeight)
        return;
    float x, y;
    {
        //����u,v�Ǵ�Dest��x,y �ָ��������������
        float u = fScale * (int)(iThread_ID % oDest.m_iWidth + roi_x),
            v = fScale * (int)(iThread_ID / oDest.m_iWidth + roi_y);

        float sinv = sinf((float)(PI - v));
        float x_ = sinv * sinf(u);
        float y_ = (float)cosf((float)(PI - v));
        float z_ = sinv * cosf(u);

        float z;
        x = (float)(oK_Rinv.M[0] * x_ + oK_Rinv.M[1] * y_ + oK_Rinv.M[2] * z_);
        y = (float)(oK_Rinv.M[3] * x_ + oK_Rinv.M[4] * y_ + oK_Rinv.M[5] * z_);
        z = (float)(oK_Rinv.M[6] * x_ + oK_Rinv.M[7] * y_ + oK_Rinv.M[8] * z_);

        if (z > 0)
            x /= z, y /= z;
        else
            x = y = -1;
    }
    unsigned char Pix[3];
    Pix_Nearest_GPU(oSource, x, y, Pix, iBorder_Type, iThread_ID);
    for (int i = 0; i < 3; i++)
        if (oDest.m_pChannel[i])
            oDest.m_pChannel[i][iThread_ID] = Pix[i];

    /*if (iThread_ID == 98 * oDest.m_iWidth + 118)
    {
        printf("%f %f thread:%d\n", x, y,iThread_ID);
    }*/

    return;
}

template<typename _T>void Re_Map_GPU(Image oSource, Image oDest, int roi_x, int roi_y,
    Sphere_Projector<_T>* poProjector, Interpolation_Flag iInter_Type, Border_Type iBorder_Type)
{//������Map_Backwardһ����
    static int iCount = 0;
    int iThread_Per_Block = 512;
    dim3 oGrid;
    oGrid.x = (oDest.m_iWidth * oDest.m_iHeight + iThread_Per_Block - 1) / iThread_Per_Block;
    oGrid.y = 1;    //oDest.m_iChannel_Count;
    oGrid.z = 1;
    //unsigned long long tStart = iGet_Tick_Count();

    //for(int i=0;i<10000;i++)
    if (iInter_Type == INTER_LINEAR)
        _Re_Map_Bi_Binear_GPU<_T> << <oGrid, iThread_Per_Block >> > (oSource.m_oPart_1, oDest.m_oPart_1, roi_x, roi_y,
            (float)(1.f / poProjector->scale), poProjector->oK_Rinv, iBorder_Type);
    else
        _Re_Map_Nearest_GPU<_T> << <oGrid, iThread_Per_Block >> > (oSource.m_oPart_1, oDest.m_oPart_1, roi_x, roi_y,
            (float)(1.f / poProjector->scale), poProjector->oK_Rinv, iBorder_Type);

    ////if (iCount == 1)
    //{
    //    Disp_Cuda_Error();
    //    //Disp_Part_GPU(oSource.m_pChannel[2], oSource.m_iWidth, 121, 99, 2, 2);
    //    //bSave_Image_GPU("c:\\tmp\\3.bmp", oDest);
    //    Compare_Image("c:\\tmp\\2.bmp", "c:\\tmp\\3.bmp",0);
    //}

    iCount++;
    return;
}

//template<typename _T>void Warp(Image oImage, _T K[3 * 3], _T R[3 * 3], _T fScale, Image* poImages_Warped,
//    Interpolation_Flag iInter_Type, int Corner[2][2], Point_Cloud<float>* poPC = NULL, Border_Type iBorder_Type = BORDER_REFLECT)
//{//ͶӰ��������ͼ����ԭ����K,T���»�ԭ����һ��ƽ�棬��ͶӰ����һ������Щ��ƽ����
//    //�������ѧ�Ƶ�����������һ��
//    static int iCount = 0;
//    _T* puxmap, * puymap;    //��ʱδ֪
//    int Dest_roi[2][2];   //[0][0-1]: x,y [1][0-1]: w,h
//    Sphere_Projector<_T> oProjector = {};
//    //unsigned long long tStart;
//
//    //����������K����ԭͼoImage��������һ���ռ���
//    //Build_Map<_T>(oImage.m_iWidth, oImage.m_iHeight, K, R, fScale, &puxmap, &puymap, Dest_roi, &oProjector, poPC);
//    //Init_Image_GPU(poImages_Warped, Dest_roi[1][0] + 1, Dest_roi[1][1] + 1, Image::IMAGE_TYPE_BMP, oImage.m_iBit_Count);
//    //Re_Map_GPU(oImage, *poImages_Warped, Dest_roi[0][0], Dest_roi[0][1], &oProjector, iInter_Type, iBorder_Type);
//
//    iCount++;
//    return;
//}

template<typename _T>void Resize_Seam_Image(Stitch<_T>* poStitch, Light_Ptr oPtr)
{
    Stitch<_T> oSt = *poStitch;
    //bLoad_Image_GPU("c:\\tmp\\1.bmp", &oSt.m_pImage_Source[0]);

    //BORDER_REFLECT
    //unsigned long long tStart = iGet_Tick_Count();

    /*Bi_Linear_cv_GPU(oSt.m_pImage_Source, oSt.m_pSeam_Est,oSt.m_iImage_Count,
        (float)oSt.seam_scale, (float)oSt.seam_scale, oPtr, BORDER_REFLECT);*/
    //for(int i=0;i<10000;i++)
    Bi_Linear_cv_GPU(oSt.m_pImage_Source_Header_GPU,
        oSt.m_pSeam_Est_Header_GPU, oSt.m_iImage_Count,
        oSt.m_Source_Size[0], oSt.m_Source_Size[1],
        oSt.m_Seam_Size[0], oSt.m_Seam_Size[1], 3,
        (float)oSt.seam_scale, (float)oSt.seam_scale, BORDER_REFLECT);
    /*Bi_Linear_cv_GPU(oSt.m_pImage_Source,
            oSt.m_pSeam_Est, oSt.m_iImage_Count,
            oSt.m_Source_Size[0], oSt.m_Source_Size[1],
            oSt.m_Seam_Size[0], oSt.m_Seam_Size[1], 3,
            (float)oSt.seam_scale, (float)oSt.seam_scale, BORDER_REFLECT);*/
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    //for(int i=0;i<4;i++)
    //    bSave_Image_GPU("c:\\tmp\\1.bmp", oSt.m_pSeam_Est[i]);
    //Compare_Image("c:\\tmp\\2.bmp", "c:\\tmp\\3.bmp");
    return;
}

__global__ void Feed_Split_Image(Image Warp[],
    Image_Size_In_Block Block_Per_Image[],      // unsigned char Block_Per_Image[][2],
    /*int bw,int bh,*/
    Image Block_Image[], unsigned char Space[],
    int Corner[][2][2], int Block_Corner[][2])
{
    int iThread_ID = threadIdx.y * blockDim.x + threadIdx.x;

    //Ҫ�ر�������Щ�������ڹ����ڴ棬��Щ��˽�б�������������׳����ڴ��ȡ����
    __shared__ Image::Part_1 oImage;
    __shared__ Image oBlock;
    __shared__ int block_tl[2], block_br[2];
    __shared__ unsigned char Block_Size[2];
    __shared__ int iBlock_ID;
    if (iThread_ID == 0)
    {
        Image_Size_In_Block oBlock_Per_Image = Block_Per_Image[blockIdx.z];
        Block_Size[0] = oBlock_Per_Image.m_Block_Per_Image[0];
        Block_Size[1] = oBlock_Per_Image.m_Block_Per_Image[1];

        if (blockIdx.x < Block_Size[0] && blockIdx.y < Block_Size[1])
        {
            //int iBlock_ID = blockIdx.z * bw * bh + blockIdx.y * bw + blockIdx.x;
            iBlock_ID = oBlock_Per_Image.m_iBlock_Start + blockIdx.y * Block_Size[0] + blockIdx.x;
            oImage = Warp[blockIdx.z].m_oPart_1;
            unsigned char block_width = (oImage.m_iWidth + Block_Size[0] - 1) / Block_Size[0];
            unsigned char block_height = (oImage.m_iHeight + Block_Size[1] - 1) / Block_Size[1];
            block_tl[0] = blockIdx.x * block_width;
            block_tl[1] = blockIdx.y * block_height;
            block_br[0] = min(block_tl[0] + block_width, oImage.m_iWidth);
            block_br[1] = min(block_tl[1] + block_height, oImage.m_iHeight);

            Block_Corner[iBlock_ID][0] = Corner[blockIdx.z][0][0] + block_tl[0];
            Block_Corner[iBlock_ID][1] = Corner[blockIdx.z][0][1] + block_tl[1];

            oBlock.m_iWidth = block_br[0] - block_tl[0];
            oBlock.m_iHeight = block_br[1] - block_tl[1];
            oBlock.m_iBit_Count = 32;
            oBlock.m_iImage_Type = Image::IMAGE_TYPE_BMP;
            oBlock.m_iChannel_Count = oBlock.m_iBit_Count >> 3;

            short iBlock_Size = oBlock.m_iWidth * oBlock.m_iHeight;
            oBlock.m_pChannel[0] = &Space[iBlock_ID * 32 * 32 * 4];
            oBlock.m_pChannel[1] = oBlock.m_pChannel[0] + iBlock_Size;
            oBlock.m_pChannel[2] = oBlock.m_pChannel[1] + iBlock_Size;
            oBlock.m_pChannel[3] = oBlock.m_pChannel[2] + iBlock_Size;

            Block_Image[iBlock_ID] = oBlock;
        }
    }
    __syncthreads();

    if (blockIdx.x >= Block_Size[0] || blockIdx.y >= Block_Size[1] ||
        threadIdx.x >= oBlock.m_iWidth || threadIdx.y >= oBlock.m_iHeight)
        return;

    int iSource_Pos = (block_tl[1] + threadIdx.y) * oImage.m_iWidth + block_tl[0] + threadIdx.x;
    int iDest_Pos = threadIdx.y * oBlock.m_iWidth + threadIdx.x;

    oBlock.m_pChannel[0][iDest_Pos] = oImage.m_pChannel[0][iSource_Pos];
    oBlock.m_pChannel[1][iDest_Pos] = oImage.m_pChannel[1][iSource_Pos];
    oBlock.m_pChannel[2][iDest_Pos] = oImage.m_pChannel[2][iSource_Pos];
    oBlock.m_pChannel[3][iDest_Pos] = oImage.m_pChannel[3][iSource_Pos];
    //if (oImage.m_pChannel[0][iSource_Pos] == 123)
    //{
    //}
}
template<typename _T>__device__ _T _Sep_Filter_2D_GPU(_T A[], short w, short h, short x, short y, _T Kernel[], int iKernel_Size)
{
    float fTotal;
    int x1, i;  //,iPadding = iKernel_Size >> 1;
    _T* pLine = &A[y * w];

    for (fTotal = 0, i = 0, x1 = x - (iKernel_Size >> 1); i < iKernel_Size; x1++, i++)
    {
        float fValue;
        if (x1 < 0)
        {
            if (-x1 >= w)
                fValue = pLine[w - 1];
            else
                fValue = pLine[-x1];
        }
        else if (x1 >= w)
        {
            if (x - (x1 - x) < 0)
                fValue = pLine[0];
            else
                fValue = pLine[x - (x1 - x)];
        }
        else
            fValue = pLine[x1];
        /*if (blockIdx.y == 0 && y == 0 && x == 9)
            printf("x1:%d fValue: %f\n", x1, fValue);*/
        fTotal += fValue * Kernel[i];
    }
    return fTotal;  // / iKernel_Size;
}
template<typename _T>__global__ void Batch_Sep_Filter_2D(_T Gain[], Image_Size_In_Block Block_Per_Image[],
    _T Kernel[], int iKernel_Size, int iImage_Count, short* pN_2_A_Map, int iIter_Count)
{
    int iThread_ID = GET_THREAD_ID();
    __shared__ Image_Size_In_Block oBlock_Per_Image;
    __shared__ _T Share_Mid[32 * 32];
    __shared__ _T Share_Gain[32 * 32];

    if (iThread_ID == 0)
        oBlock_Per_Image = Block_Per_Image[blockIdx.y];
    __syncthreads();

    if (iThread_ID >= oBlock_Per_Image.m_Block_Per_Image[0] * oBlock_Per_Image.m_Block_Per_Image[1])
        return;

    unsigned short /*w = oBlock_Per_Image.m_Block_Per_Image[0],*/
        /*h = oBlock_Per_Image.m_Block_Per_Image[1],*/
        y = iThread_ID / oBlock_Per_Image.m_Block_Per_Image[0],
        x = iThread_ID % oBlock_Per_Image.m_Block_Per_Image[0];

    //�����ݵ�Buffer��
    Share_Gain[iThread_ID] = Gain[oBlock_Per_Image.m_iBlock_Start + iThread_ID];
    __syncthreads();

    for (short i = 0; i < iIter_Count; i++)
    {
        Share_Mid[x * oBlock_Per_Image.m_Block_Per_Image[1] + y] =
            _Sep_Filter_2D_GPU(Share_Gain, oBlock_Per_Image.m_Block_Per_Image[0],
                oBlock_Per_Image.m_Block_Per_Image[1], x, y, Kernel, iKernel_Size);
        __syncthreads();

        Share_Gain[y * oBlock_Per_Image.m_Block_Per_Image[0] + x] =
            _Sep_Filter_2D_GPU(Share_Mid, oBlock_Per_Image.m_Block_Per_Image[1],
                oBlock_Per_Image.m_Block_Per_Image[0], y, x, Kernel, iKernel_Size);
        __syncthreads();
    }
    Gain[oBlock_Per_Image.m_iBlock_Start + iThread_ID] = Share_Gain[iThread_ID];
}

__device__ static int bGet_Matrix_xy_GPU(int iPos, int n, int* px, int* py)
{//����һ������Ľ�n,��һ��˳��λ�ã������ھ���������
    //��󲻵�Ϊ  (n + n-y-1)*y/2 <=N
    //������h, ��h-1�еĸ���Ϊ n-(h-1)
    //����Ϊ [n + n-(h-1)]*h/2 = (2n+1 -h)*h/2 <=N
    // (2n+1)*h - h^2 <=2N
    //h^2 - (2n+1)h >=-2N
    //h^2 - (2n+1)h + 2N >=0

    //���߿�������
    int/* a = 1,*/
        b = -(1 + 2 * n), c = 2 * iPos;
    float fDelta = b * b - 4 * c;
    float sqrt_b_sqr_4ac = sqrt(fDelta);
    if (fDelta < 0)
    {
        printf("��������");
        return 0;
    }

    float h[2] = { (-b + sqrt_b_sqr_4ac) / 2.f,	//������
        (-b - sqrt_b_sqr_4ac) / 2.f };

    //��������������y1[0]Ϊ�����y1[1]ΪС��
    //��������
    int h1 = floor(h[1]);
    if (h1 > n || h1 < 0)
    {
        h1 = std::floor(h[1]);
        if (h1 > n || h1 < 0)
            return 0;
    }

    //yΪʲô���ڸߣ���Ϊ����һ�п�ʼ
    int x, x_Start = h1, y = h1;
    int iLine_Start = (n + (n - y + 1)) * y / 2;
    if (iLine_Start == iPos)
        x = y;
    else
        x = iPos - iLine_Start + x_Start;
    *px = x, * py = y;
    return 1;
}

__device__ int bIs_Overlap_GPU(int tl1[2], int tl2[2], int w1, int h1, int w2, int h2, short roi[2][2])
{
    int x_tl = max(tl1[0], tl2[0]);
    int y_tl = max(tl1[1], tl2[1]);
    int x_br = min(tl1[0] + w1, tl2[0] + w2);
    int y_br = min(tl1[1] + h1, tl2[1] + h2);
    if (x_tl < x_br && y_tl < y_br)
    {
        roi[0][0] = x_tl, roi[0][1] = y_tl;
        roi[1][0] = x_br - x_tl, roi[1][1] = y_br - y_tl;
        return 1;
    }
    return 0;
}

__global__ void _Get_Overlap(Image Block_Image[], int Block_Corner[][2],
    int iBlock_Count, Overlap_Pair Pair[], int* piCount)
{
    __shared__ Overlap_Pair Pair_Buffer[1024];
    __shared__ int iPair_Count, iPos_1;

    int iThread_ID = GET_THREAD_ID();
    if (iThread_ID >= (iBlock_Count + 1) * iBlock_Count / 2)
        return;
    if (threadIdx.x == 0)
        iPair_Count = 0;
    __syncthreads();

    int x, y;
    if (!bGet_Matrix_xy_GPU(iThread_ID, iBlock_Count, &x, &y))
    {
        printf("err");
        return;
    }
    /*if(x>=iBlock_Count || y>=iBlock_Count || x<y)
        printf("%d %d\n",x,y);*/
    Overlap_Pair oPair;
    if (bIs_Overlap_GPU(Block_Corner[x], Block_Corner[y],
        Block_Image[x].m_iWidth, Block_Image[x].m_iHeight,
        Block_Image[y].m_iWidth, Block_Image[y].m_iHeight,
        oPair.roi))
    {
        oPair.m_Pair[0] = x, oPair.m_Pair[1] = y;
        int iPos = atomicAdd(&iPair_Count, 1);
        Pair_Buffer[iPos] = oPair;
    }
    __syncthreads();

    //�����Լ��Ժ󣬼ӵ��ڴ���

    if (threadIdx.x == 0)
        iPos_1 = atomicAdd(piCount, iPair_Count);
    __syncthreads();
    for (int i = threadIdx.x; i < iPair_Count; i++)
        Pair[iPos_1 + i] = Pair_Buffer[i];

    return;
}

void Single_Feed_Get_Overlap(Image Block_Image[], int Block_Corner[][2], int iBlock_Count,
    Overlap_Pair** ppPair, int* piCount)
{//����һЩBlock������λ�ã������Ƿ��н�
    int iPair_Count, iSize = iBlock_Count * 10;
    int* piCount_GPU = (int*)pMalloc_GPU(iBlock_Count * 10 * sizeof(Overlap_Pair));
    hipMemset(piCount_GPU, 0, sizeof(int));
    Overlap_Pair* pPair = (Overlap_Pair*)(piCount_GPU + 1);

    //�������ǵĴ�С
    iSize = (iBlock_Count + 1) * iBlock_Count / 2;
    int iThread_Per_Block = 1024;
    int iBlock_Count_1 = (iSize + iThread_Per_Block - 1) / iThread_Per_Block;

    unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    {
        _Get_Overlap << <iBlock_Count_1, iThread_Per_Block >> > (Block_Image,
            Block_Corner, iBlock_Count, pPair, piCount_GPU);
    }

    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    hipMemcpy(&iPair_Count, piCount_GPU, 4, hipMemcpyDeviceToHost);
    if (iPair_Count > iBlock_Count * 10)
    {
        printf("iPair_Count exceeds max count\n");
        exit(0);
    }

    iSize = sizeof(int) + iPair_Count * sizeof(Overlap_Pair);
    Shrink_GPU(piCount_GPU, iSize);
    *piCount = iPair_Count;
    *ppPair = pPair;
    return;
}

__device__ static void Get_Overlap_GPU(short roi[2][2], int Corner[2], short Overlap[2][2])
{
    Overlap[0][0] = roi[0][0] - Corner[0];
    Overlap[0][1] = roi[0][1] - Corner[1];
    //roi.br() - corners[i]
    Overlap[1][0] = roi[0][0] + roi[1][0] - Corner[0] - Overlap[0][0];
    Overlap[1][1] = roi[0][1] + roi[1][1] - Corner[1] - Overlap[0][1];
}

template<typename _T> __device__ float fGet_Mod_3(_T x, _T y, _T z)
{//�������ģ
    return (_T)sqrt(x * x + y * y + z * z);
}

__global__ void Get_Intersect_1(Image Block_Image[], int iBlock_Count,
    Overlap_Pair Pair[], int iPair_Count, int Block_Corner[][2],
    int N[], double I[], unsigned char Skip[])
{//�ú����ڴ����������⣬��������
    __shared__ unsigned int iShare_Intersect_Count;     //������� 1200ms
    __shared__ unsigned short iSub_Image_Width, iSub_Image_Height,            //Crop�����Ŀ鶼��һ����С
        iSub_Image_Size,                                //��ͼ�Ĵ�С
        iChannel_Count;                                 //��ɫͨ����������alpha

    __shared__ short Overlap_1[2][2], Overlap_2[2][2];
    //�ص����ڴλ������У�����plannar����rgba    
    __shared__ unsigned char Sub_Image_1[32 * 32][4], Sub_Image_2[32 * 32][4], Intersect[32 * 32];
    __shared__  Overlap_Pair oPair;
    __shared__ Image::Part_1  oBlock_Image_1, oBlock_Image_2;

    //__shared__ int Share_Intersect_Count[32];
    __shared__ float Share_Sum[2][32];  //��Isum1, Isum2

    if (threadIdx.x == 0)
    {//��ȻҪ��ԭ������ˬ
        iShare_Intersect_Count = 0;     //�����������ֵ0
        oPair = Pair[blockIdx.x];       //ע�⣬����һ��Block_Image, ����[1]Ϊy��, [0]Ϊx��

        //����roi��Block_Corner�����ص������γ�Sub_Image
        //�˴���Ҫ������ѧԭ��
        Get_Overlap_GPU(oPair.roi, Block_Corner[oPair.m_Pair[1]], Overlap_1);
        iSub_Image_Width = Overlap_1[1][0];
        iSub_Image_Height = Overlap_1[1][1];
        iSub_Image_Size = iSub_Image_Width * iSub_Image_Height;

        Get_Overlap_GPU(oPair.roi, Block_Corner[oPair.m_Pair[0]], Overlap_2);
        iChannel_Count = Block_Image[oPair.m_Pair[0]].m_iChannel_Count;
        oBlock_Image_1 = Block_Image[oPair.m_Pair[1]].m_oPart_1;
        oBlock_Image_2 = Block_Image[oPair.m_Pair[0]].m_oPart_1;
    }
    for (int i = threadIdx.x; i < 32; i += blockDim.x)
        Share_Sum[0][i] = Share_Sum[1][i] = 0;
    __syncthreads();
    if (threadIdx.x >= iSub_Image_Size) //���ǳ���Sub_Image��С���̶߳������˳���
        return;
    //Share_Intersect_Count[i] = 0;

    unsigned short Intersect_Count = 0;
    for (unsigned int iPos_d = threadIdx.x; iPos_d < iSub_Image_Size; iPos_d += blockDim.x)
    {
        //������㷽���������߳���(x,y)
        unsigned short x_d = iPos_d % iSub_Image_Width;
        unsigned short y_d = iPos_d / iSub_Image_Width;
        //��ԭͼBlock_Image��λ��
        unsigned short iPos_s = (Overlap_1[0][1] + y_d) * oBlock_Image_1.m_iWidth +
            Overlap_1[0][0] + x_d;
        //���س���Sub_Image_1�У��첻�쿴����
        /*Pixel_4 oPixel = { oBlock_Image_1.m_pChannel[0][iPos_s],
         oBlock_Image_1.m_pChannel[0][iPos_s],
         oBlock_Image_1.m_pChannel[0][iPos_s] ,
            oBlock_Image_1.m_pChannel[0][iPos_s] };*/

        *(Pixel_4*)Sub_Image_1[iPos_d] =
        { oBlock_Image_1.m_pChannel[0][iPos_s],
         oBlock_Image_1.m_pChannel[1][iPos_s],
         oBlock_Image_1.m_pChannel[2][iPos_s] ,
            oBlock_Image_1.m_pChannel[3][iPos_s] };

        /*Sub_Image_1[iPos_d][0] = oBlock_Image_1.m_pChannel[0][iPos_s];
        Sub_Image_1[iPos_d][1] = oBlock_Image_1.m_pChannel[1][iPos_s];
        Sub_Image_1[iPos_d][2] = oBlock_Image_1.m_pChannel[2][iPos_s];
        Sub_Image_1[iPos_d][3] = oBlock_Image_1.m_pChannel[3][iPos_s];*/

        iPos_s = (Overlap_2[0][1] + y_d) * oBlock_Image_2.m_iWidth +
            Overlap_2[0][0] + x_d;
        *(Pixel_4*)Sub_Image_2[iPos_d] = { oBlock_Image_2.m_pChannel[0][iPos_s],
            oBlock_Image_2.m_pChannel[1][iPos_s],
            oBlock_Image_2.m_pChannel[2][iPos_s],
            oBlock_Image_2.m_pChannel[3][iPos_s] };
        /*Sub_Image_2[iPos_d][0] = oBlock_Image_2.m_pChannel[0][iPos_s];
        Sub_Image_2[iPos_d][1] = oBlock_Image_2.m_pChannel[1][iPos_s];
        Sub_Image_2[iPos_d][2] = oBlock_Image_2.m_pChannel[2][iPos_s];
        Sub_Image_2[iPos_d][3] = oBlock_Image_2.m_pChannel[3][iPos_s];*/

        if (Sub_Image_1[iPos_d][3] == 255 && Sub_Image_2[iPos_d][3] == 255)
        {//�˴����ǽ������ж����ݣ�����MaskͬΪ255��Ϊ����
            Intersect[iPos_d] = 255;
            Intersect_Count++;
        }
        else
            Intersect[iPos_d] = 0;
    }
    atomicAdd(&iShare_Intersect_Count, Intersect_Count);
    __syncthreads();
    /*if (oPair.m_Pair[0] == 0 && oPair.m_Pair[1] == 0)
        printf("%d\n", iShare_Intersect_Count);*/

        //����N�����ֵ������Skip��Ӧ��ֵ
    if (threadIdx.x == 0)
    {
        //�ɼ���N�������װ���Լ�������ͼƬ�����Ĵ�С
        N[oPair.m_Pair[0] * iBlock_Count + oPair.m_Pair[1]] =
            N[oPair.m_Pair[1] * iBlock_Count + oPair.m_Pair[0]] =
            Max(1, iShare_Intersect_Count);

        //����Լ����������н�������������
        if (oPair.m_Pair[0] != oPair.m_Pair[1])
            Skip[oPair.m_Pair[0]] = Skip[oPair.m_Pair[1]] = 0;    //��ʾi,j Ԫ���ж���
        //Isum1 = Isum2 = 0;
    }
    __syncthreads();

    //����I����
    for (unsigned short iPos_d = threadIdx.x; iPos_d < iSub_Image_Size; iPos_d += blockDim.x)
    {
        //unsigned short x_d = iPos_d % iSub_Image_Width;
        //unsigned short y_d = iPos_d / iSub_Image_Width;
        if (iChannel_Count >= 3)
        {
            if (Intersect[iPos_d])
            {
                atomicAdd(&Share_Sum[0][iPos_d % 32], fGet_Mod_3<float>(Sub_Image_1[iPos_d][0],
                    Sub_Image_1[iPos_d][1],
                    Sub_Image_1[iPos_d][2]));

                atomicAdd(&Share_Sum[1][iPos_d % 32], fGet_Mod_3<float>(Sub_Image_2[iPos_d][0],
                    Sub_Image_2[iPos_d][1],
                    Sub_Image_2[iPos_d][2]));
            }
        }
        else if (iChannel_Count == 1)
        {
            printf("Not implemented yet\n");
            /*atomicAdd(&Isum1, oSub_Image_1.m_pChannel[3][iPos_d]);
            atomicAdd(&Isum2, oSub_Image_2.m_pChannel[3][iPos_d]);*/
        }
        else
        {
            printf("Not implemented yet\n");
        }
    }
    __syncthreads();

    if (threadIdx.x == 0)
    {
        for (int i = 1; i < 32; i++)
        {//ֱ�����������Isum1, Isum2
            Share_Sum[0][0] += Share_Sum[0][i];
            Share_Sum[1][0] += Share_Sum[1][i];
            /*if (oPair.m_Pair[1] == 2 && oPair.m_Pair[0] == 91)
                printf("%f %f\n", Share_Sum[0][i], Share_Sum[0][0]);*/
        }

        int iPos_1 = oPair.m_Pair[1] * iBlock_Count + oPair.m_Pair[0];
        int iPos_2 = oPair.m_Pair[0] * iBlock_Count + oPair.m_Pair[1];
        I[iPos_1] = Share_Sum[0][0] / N[iPos_1];
        I[iPos_2] = Share_Sum[1][0] / N[iPos_2];
        /*if (oPair.m_Pair[1] == 2 && oPair.m_Pair[0] == 91)
            printf("I[2,9]:%f N:%d Isam1:%f\n",
                I[iPos_1],N[iPos_1], Share_Sum[0][0]);*/
    }
    return;
}

__global__ void Get_num_eq(unsigned char Skip[], int iBlock_Count, int* piNum_eq,
    short A_2_N_Map[], short N_2_A_Map[])
{//һ��㶨   
    int iNum_eq = 0;
    for (int i = 0; i < iBlock_Count; i++)
    {
        if (Skip[i] == 0)
        {
            A_2_N_Map[iNum_eq] = i;
            N_2_A_Map[i] = iNum_eq;
            iNum_eq++;
        }
    }
    *piNum_eq = iNum_eq;
    /*for(int i=0;i<iNum_eq;i++)
        printf("%d\n", A_2_N_Map[i]);*/
}

__global__ void Gen_Eq(double A[], int iOrder, /*double x[], */double b[],
    short pA_2_N_Map[], int N[], double I[], int iBlock_Count,
    double alpha, double beta)
{
    int iThread_ID = GET_THREAD_ID();
    int x1, y1;

    if (iThread_ID >= iOrder * iOrder)
        return;
    y1 = iThread_ID / iOrder;
    x1 = iThread_ID % iOrder;

    /*if (iThread_ID >= (iOrder + 1) * iOrder / 2)
        return;

    if (!bGet_Matrix_xy_GPU(iThread_ID, iOrder, &x1, &y1))
    {
        printf("error");
        return;
    }*/

    int iPos_N = pA_2_N_Map[y1] * iBlock_Count + pA_2_N_Map[x1];
    //��Ȼ���ǶԽ���Ԫ��
    if (N[iPos_N])
    {
        atomicAdd(&b[y1], beta * N[iPos_N]);
        __threadfence();
        atomicAdd(&A[y1 * iOrder + y1], beta * N[iPos_N]);
        //if (y1 == 0 /*&& x1 == 0*/)
            //printf("%f %f\n", beta * N[iPos_N], A[y1 * iOrder + y1]);
        __threadfence();
        if (x1 != y1)
        {//�ǶԽ���Ԫ��
            atomicAdd(&A[y1 * iOrder + y1], 2 * alpha * I[iPos_N] * I[iPos_N] * N[iPos_N]);
            //if (y1 == 0)
                //printf("y_s:%d x_s:%d %f %f\n", pA_2_N_Map[y1], pA_2_N_Map[x1], I[iPos_N], A[y1 * iOrder + y1]);
            __threadfence();
            A[y1 * iOrder + x1] -= 2 * alpha * I[iPos_N] * I[pA_2_N_Map[x1] * iBlock_Count + pA_2_N_Map[y1]] * N[iPos_N];
            //if (y1 == 0 /*&& x1 == 64*/)
                //printf("%d, %d, %lf\n", y1, x1, I[pA_2_N_Map[x1] * iBlock_Count + pA_2_N_Map[y1]]);
        }
    }
    return;
}

__global__ void Set_Gain(unsigned char Skip[], double x[], int iBlock_Count,
    double Gain[], short N_2_A_Map[])
{
    int iThread_ID = GET_THREAD_ID();
    if (iThread_ID >= iBlock_Count)
        return;
    Gain[iThread_ID] = !Skip[iThread_ID] ? x[N_2_A_Map[iThread_ID]] : 1;
}

template<typename _T>void Single_Feed(Stitch<_T>* poStitch,
    Compensator* poComp)
{//�����е�Block�󽻣���������һ��Ȩֵ���㣬�������
//�ڴ棬���˲���pGainһ���⣬����ȫ�������ͷ�
    Overlap_Pair* pPair;
    int iSize, iPair_Count, iBlock_Count = poStitch->m_iBlock_Count;
    Single_Feed_Get_Overlap(poStitch->m_pBlock_Image_Header_GPU,
        poStitch->m_pBlock_Corner_GPU, poStitch->m_iBlock_Count, &pPair, &iPair_Count);

    //�Ž��������ظ���,pN�Ǹ��Գƾ���
    //int* piOverlap_Count = (int*)pMalloc_GPU(4);
    int* pN = (int*)pMalloc_GPU(iBlock_Count * iBlock_Count * sizeof(int));
    //I[i] = Isum/pN[i]  Isum1 ���ۼ�����ֵ֮�ͣ��ٳ������ظ�������������ֵƽ��ֵ
    //�������������ʲô��ɫ
    double* pI = (double*)pMalloc_GPU(iBlock_Count * iBlock_Count * sizeof(double));
    hipMemset(pN, 0, iBlock_Count * iBlock_Count * sizeof(int));
    hipMemset(pI, 0, iBlock_Count * iBlock_Count * sizeof(double));

    iSize = iPair_Count * 2 * sizeof(Image) +
        iPair_Count * 2 * 32 * 32 * 4;
    //Image* pSub_Image_1_GPU, * pSub_Image_2_GPU;
    //unsigned char* pSpace;
    //pSub_Image_1_GPU = (Image*)pMalloc_GPU(iSize);
    //pSub_Image_2_GPU = pSub_Image_1_GPU + iPair_Count;
    //pSpace = (unsigned char*)(pSub_Image_2_GPU + iPair_Count);

    //���еĿ鶼�п���������γɽ��������ͱ����γɲ��˽���
    //�����������
    unsigned char* pSkip = (unsigned char*)pMalloc_GPU(iBlock_Count);
    hipMemset(pSkip, 1, iBlock_Count); //ȱʡ�£����еĿ鶼����н���

    dim3 oThread, oGrid;
    //oThread.x = oThread.y = 32;
    oThread.x = 512;
    oGrid.x = iPair_Count;
    Disp_Cuda_Error();
    //bSave_Image_GPU("c:\\tmp\\2.bmp", &poStitch->m_pBlock_Image_Header_GPU[2]);
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    Get_Intersect_1 << <oGrid, oThread >> >
        (poStitch->m_pBlock_Image_Header_GPU, iBlock_Count, pPair, iPair_Count, poStitch->m_pBlock_Corner_GPU,
            /* pSub_Image_1_GPU, pSub_Image_2_GPU, pSpace,*/ pN, pI, pSkip);
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);;

    //unsigned short* pN_2_A_Map = (unsigned short*)pMalloc_GPU(iBlock_Count);
    int* piNum_eq = (int*)pMalloc(4);  //ϵ������A�Ĵ�С
    short* pA_2_N_Map = (short*)pMalloc_GPU(iBlock_Count * 2 * sizeof(short));
    short* pN_2_A_Map = pA_2_N_Map + iBlock_Count;

    hipMemset(pA_2_N_Map, -1, iBlock_Count * 2 * sizeof(short));
    /* tStart = iGet_Tick_Count();
     for(int i=0;i<10000;i++)*/
    Get_num_eq << <1, 1 >> > (pSkip, iBlock_Count, piNum_eq, pA_2_N_Map, pN_2_A_Map);
    Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    //������Ҫ���� ����
    int num_eq = *piNum_eq;
    iSize = num_eq * num_eq * sizeof(double) + 128 +
        num_eq * sizeof(double) + 128;
    Light_Ptr oPtr;
    Attach_Light_Ptr(oPtr, (unsigned char*)pMalloc_GPU(iSize), iSize, 0);
    double* pA_GPU, * pb_GPU, * pA, * pb;
    unsigned char* p;
    Malloc(oPtr, num_eq * num_eq * sizeof(double), p);
    pA_GPU = (double*)p;
    Malloc(oPtr, num_eq * sizeof(double), p);
    pb_GPU = (double*)p;

    Attach_Light_Ptr(oPtr, (unsigned char*)pMalloc(iSize), iSize, 0);
    Malloc(oPtr, num_eq * num_eq * sizeof(double), p);
    pA = (double*)p;
    Malloc(oPtr, num_eq * sizeof(double), p);
    pb = (double*)p;

    double* px = (double*)pMalloc(num_eq * sizeof(double));
    hipMemset(pA_GPU, 0, num_eq * num_eq * sizeof(double));
    hipMemset(pb_GPU, 0, num_eq * sizeof(double));
    const double beta = 100, alpha = 0.01;

    int iThread_Count = num_eq * num_eq;    //(num_eq + 1) * num_eq / 2;
    oThread.x = 1024;
    oGrid.x = (iThread_Count + oThread.x - 1) / oThread.x;
    Disp_Cuda_Error();

    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    {
        Gen_Eq << <oGrid, oThread >> > (pA_GPU, num_eq, /*px,*/ pb_GPU, pA_2_N_Map, pN, pI,
            iBlock_Count, alpha, beta);
        hipMemcpy(pA, pA_GPU, iSize, hipMemcpyDeviceToHost);
    }
    Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    //�ⷽ��
    int iResult;


    //tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    //CPU�� 1400ms, ��������
    Solve_Linear_Gause_AAt(pA, num_eq, pb, px, &iResult);   //|px| = 16.933621

    //printf("%lld\n", iGet_Tick_Count() - tStart);
    printf("Sum of x:%f\n", fGet_Mod(px, num_eq));

    double* pGain = (double*)pMalloc_GPU(iBlock_Count * sizeof(double));
    oThread.x = 512;
    oGrid.x = (iBlock_Count + oThread.x - 1) / oThread.x;

    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    Set_Gain << <oGrid, oThread >> > (pSkip, px, iBlock_Count, pGain, pN_2_A_Map);    //78ms, û���Ż���Ҫ
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    //Disp_GPU(pGain, 1, iBlock_Count);
    poComp->m_pGain = pGain;

    //Ȼ���е�û��ȫ�ͷ�
    Free_GPU(((unsigned int*)pPair) - 1);   //�˴���Щ��֣��Ժ�����ʰ��
    Free_GPU(pN);
    Free_GPU(pI);
    Free_GPU(pSkip);
    Free(piNum_eq);
    Free_GPU(pA_2_N_Map);
    Free_GPU(pA_GPU);
    Free(pA);
    Free(px);
    return;
}

template<typename _T>void Feed(Stitch<_T>* poStitch, Compensator* poComp)
{
    short* pN_2_A_Map = NULL;
    for (int n = 0; n < poComp->nr_feed; n++)
    {
        Single_Feed(poStitch, poComp);
    }
    //poComp->m_pGain_Map = (double**)pMalloc(poStitch->m_iImage_Count * sizeof(double*));
    //_T* pKer = poStitch->m_pKer_GPU;
    int iMax_Size = 0;
    for (int i = 0; i < poStitch->m_iImage_Count; i++)
    {
        int iSize = poStitch->m_pBlock_Per_Image[i].m_Block_Per_Image[0] * poStitch->m_pBlock_Per_Image[i].m_Block_Per_Image[1];
        if (iSize > iMax_Size)
            iMax_Size = iSize;
    }
    if (iMax_Size >= 32 * 32)
    {
        printf("Exceed max size in Feed\n");
        exit(0); //�޿����
    }
    //�˴���һ�����ӵĿɷ�����
    //����Mid�Ĵ�С
    //_T* pMid = (_T*)pMalloc_GPU(poStitch->m_iBlock_Count * sizeof(_T));
    dim3 oThread, oGrid;
    oThread.x = Min(iMax_Size, 512);
    oGrid.x = (iMax_Size + oThread.x - 1) / oThread.x;
    oGrid.y = poStitch->m_iImage_Count;

    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    Batch_Sep_Filter_2D<_T> << <oGrid, oThread >> > (poComp->m_pGain, poStitch->m_pBlock_Per_Image_GPU,
        poStitch->m_pKer_GPU, 3, /*pMid,*/ poStitch->m_iImage_Count, pN_2_A_Map, 2);
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    ////����ûʲô�ã�ֻ�ܲ�Ҫ
    //for (int i = 0; i < poStitch->m_iImage_Count; i++)
    //{
    //    Image_Block oBlock = poStitch->m_pBlock_Per_Image[i];
    //    poComp->m_pGain_Map[i] = &poComp->m_pGain[oBlock.m_iBlock_Start];
    //}
    return;
}

template<typename _T>__device__ static _T Pix_Inter_2_GPU(_T Image[], short w, short h, float x, float y,
    Border_Type iImage_Border_Type, int iThread_ID = 0)
{
    short x1 = floor(x),
        y1 = floor(y);
    //h_1 = oImage.m_iHeight - 1,
    //w_1 = oImage.m_iWidth - 1;
    _T A, B, C, D;
    int iCur_Line_Pos, iNext_Line_Pos;
    short xl_Pos, xr_Pos;

    //�ȸ�Image_Warp�Ĳ�ֵ
    if (iImage_Border_Type == BORDER_CONSTANT)
    {//���������
        iCur_Line_Pos = iNext_Line_Pos = -1;
        short y2 = y1;
        if (y2 >= 0 && y2 < h)
            iCur_Line_Pos = y2 * w;
        y2++;
        if (y2 >= 0 && y2 < h)
            iNext_Line_Pos = y2 * w;

        xl_Pos = xr_Pos = -1;
        short x2 = x1;
        if (x2 >= 0 && x2 < w)
            xl_Pos = x2;
        x2++;
        if (x2 >= 0 && x2 < w)
            xr_Pos = x2;
    }
    else
    {
        iCur_Line_Pos = iGet_Border_y_GPU(y1, h, iImage_Border_Type) * w;
        iNext_Line_Pos = iGet_Border_y_GPU(y1 + 1, h, iImage_Border_Type) * w;
        xl_Pos = iGet_Border_x_GPU(x1, w, iImage_Border_Type);
        xr_Pos = iGet_Border_x_GPU(x1 + 1, w, iImage_Border_Type);
    }

    if (iCur_Line_Pos >= 0)
    {
        A = xl_Pos >= 0 ? Image[iCur_Line_Pos + xl_Pos] : 0;
        B = xr_Pos >= 0 ? Image[iCur_Line_Pos + xr_Pos] : 0;
    }
    else
        A = B = 0;
    if (iNext_Line_Pos >= 0)
    {
        C = xl_Pos >= 0 ? Image[iNext_Line_Pos + xl_Pos] : 0;
        D = xr_Pos >= 0 ? Image[iNext_Line_Pos + xr_Pos] : 0;
    }
    else
        C = D = 0;


    float fValue_0, fValue_1;
    {
        float w1 = x - x1, w0 = 1.f - w1;
        fValue_0 = w0 * A + w1 * B;
        fValue_1 = w0 * C + w1 * D;
    }

    {
        float w3 = y - y1, w2 = 1.f - w3;
        return w2 * fValue_0 + w3 * fValue_1;   //���ڲ������룬�ʴ˲��ؼ���0.5
    }
}

template<typename _T>__global__ void _Block_Compensate(Image Warp[], _T Gain[],
    Image_Size_In_Block Block_Per_Image[], _T Aux[] = NULL)
{//��������Ѿ���ȫ����opencv�����˸���������Ϊǰ��ĸ�����������΢�Ĳ������
    //����ǰ��������������⣬����������������������
    int iThread_ID = GET_THREAD_ID();
    __shared__ Image_Size_In_Block oBlock_Per_Image;
    __shared__ Image::Part_1 oImage;
    __shared__ _T Share_Gain[32 * 32];

    if (threadIdx.x == 0)
    {
        oImage = Warp[blockIdx.y].m_oPart_1;
        oBlock_Per_Image = Block_Per_Image[blockIdx.y];
    }
    __syncthreads();

    if (iThread_ID >= oImage.m_iHeight * oImage.m_iWidth)
        return;
    //printf("Thread:%d %d\n", iThread_ID,oImage.m_iWidth*oImage.m_iHeight );
    
        
    short w = oBlock_Per_Image.m_Block_Per_Image[0],
        h = oBlock_Per_Image.m_Block_Per_Image[1],
        y = iThread_ID / oImage.m_iWidth,
        x = iThread_ID % oImage.m_iWidth;

    //Gain�㹻С��װ�빲���ڴ�����
    int iSize = w * h;

    for (int i = threadIdx.x; i < iSize; i += blockDim.x)
        Share_Gain[i] = Gain[oBlock_Per_Image.m_iBlock_Start + i];
    __syncthreads();

    float f_y = (float)h / oImage.m_iHeight,
        f_x = (float)w / oImage.m_iWidth;

    //opencv����
    //y_s_f = (y + 0.5f) * f_y - 0.5f;
    //x_s_f = (x + 0.5f) * f_x - 0.5f;
    _T fPix = Pix_Inter_2_GPU(Share_Gain, w, h, (x + 0.5f) * f_x - 0.5f, (y + 0.5f) * f_y - 0.5f, BORDER_REFLECT, iThread_ID);
    /*unsigned*/ short iPix;
    iPix = oImage.m_pChannel[0][iThread_ID] * fPix + 0.5f;
    oImage.m_pChannel[0][iThread_ID] = Clip(iPix);

    iPix = oImage.m_pChannel[1][iThread_ID] * fPix + 0.5f;
    oImage.m_pChannel[1][iThread_ID] = Clip(iPix);

    iPix = oImage.m_pChannel[2][iThread_ID] * fPix + 0.5f;
    oImage.m_pChannel[2][iThread_ID] = Clip(iPix);
    /*if (Aux && blockIdx.y == 0 && y == 1089 && x == 257)
    {
        printf("B:%d\n",iPix);
    }*/
    return;
}

template<typename _T>void Block_Compensate(Stitch<_T>* poStitch, 
    Image Image_Warp[],Image Image_Warp_Header_GPU[])
{//��ԭ����Warp ͶӰͼ���в��⣿
    //�˴���������ΪWarp Image���ף�ֱ����
    static int iCount = 0;
    int iMax_Size = 0;
    for (int i = 0; i < poStitch->m_iImage_Count; i++)
    {
        /*int iSize = poStitch->m_pImage_Warp[i].m_iWidth *
            poStitch->m_pImage_Warp[i].m_iHeight;*/
        int iSize = Image_Warp[i].m_iWidth * Image_Warp[i].m_iHeight;
        if (iSize > iMax_Size)
            iMax_Size = iSize;
        /*if (iCount == 1)
            printf("%d\n", iSize);*/
    }
    dim3 oThread, oGrid;

    oThread.x = 512;
    oGrid.x = (iMax_Size + oThread.x - 1) / oThread.x;
    oGrid.y = poStitch->m_iImage_Count;
    //Disp_Part_GPU<unsigned char>(poStitch->m_pImage_Warp[0].m_pChannel[1], poStitch->m_pImage_Warp[0].m_iWidth, 129, 0, 3, 2);
    //double* pGain_Resize = (double*)pMalloc(iMax_Size * sizeof(double*));
    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    //_Block_Compensate << <oGrid, oThread >> > (poStitch->m_pImage_Warp_Header_GPU,
    //    poStitch->m_oComp.m_pGain, poStitch->m_pBlock_Per_Image_GPU/*, pGain_Resize*/);

    /*_T* pAux = NULL;
    if (iCount == 1)
        pAux = (_T*)pMalloc(10);*/

    _Block_Compensate << <oGrid, oThread >> > (Image_Warp_Header_GPU,
        poStitch->m_oComp.m_pGain, poStitch->m_pBlock_Per_Image_GPU);
    //Disp_Cuda_Error();
    //if (iCount == 1)
    //{
    //    //bSave_Image_GPU("c:\\tmp\\1.bmp", &Image_Warp_Header_GPU[0]);
    //    //Compare_Image("c:\\tmp\\Warp_Comp_0.bmp", "c:\\tmp\\1.bmp",1);
    //    //printf("Here");
    //}
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    //Image oImage = poStitch->m_pImage_Warp[0];
    //Disp_Part<double>(pGain_Resize, oImage.m_iWidth, 0, 0, oImage.m_iWidth, 1);
    //Disp_Part<double>(pGain_Resize, oImage.m_iWidth, 129,0, 3, 2);
    //Disp_Part_GPU<unsigned char>(oImage.m_pChannel[1], oImage.m_iWidth, 130, 0, 1, 1);

    /*bSave_Image_GPU("c:\\tmp\\2.bmp", &poStitch->m_pImage_Warp_Header_GPU[3]);
    Compare_Image("c:\\tmp\\1.bmp", "c:\\tmp\\2.bmp");*/
    iCount++;
    return;
}

template<typename _T>void Feed(Stitch<_T>* poStitch)
{//����Ƿ����ձ��ԣ�����˵������У�������ͨ�ýӿ�
    //�ڴο���Ҳ�޷�
    int (*Corner)[2][2] = poStitch->m_pCorner;
    Image* Images_Warped = poStitch->m_pImage_Warp;
    //*Masks_Warped = poStitch->m_pMasks_Warped;
    int iBlock_Count = 0;

    //��ͳ��������
    for (int iImage_Index = 0; iImage_Index < poStitch->m_iImage_Count; iImage_Index++)
    {
        int block_per_img[2] = { (Images_Warped[iImage_Index].m_iWidth + 32 - 1) / 32,
                                (Images_Warped[iImage_Index].m_iHeight + 32 - 1) / 32 };

        poStitch->m_pBlock_Per_Image[iImage_Index].m_Block_Per_Image[0] = block_per_img[0];
        poStitch->m_pBlock_Per_Image[iImage_Index].m_Block_Per_Image[1] = block_per_img[1];
        if (iImage_Index == 0)
            poStitch->m_pBlock_Per_Image[iImage_Index].m_iBlock_Start = 0;
        else
        {
            poStitch->m_pBlock_Per_Image[iImage_Index].m_iBlock_Start =
                poStitch->m_pBlock_Per_Image[iImage_Index - 1].m_iBlock_Start +
                poStitch->m_pBlock_Per_Image[iImage_Index - 1].m_Block_Per_Image[0] *
                poStitch->m_pBlock_Per_Image[iImage_Index - 1].m_Block_Per_Image[1];
        }
        iBlock_Count += block_per_img[0] * block_per_img[1];
    }
    poStitch->m_iBlock_Count = iBlock_Count;
    hipMemcpy(poStitch->m_pBlock_Per_Image_GPU, poStitch->m_pBlock_Per_Image,
        poStitch->m_iImage_Count * sizeof(Image_Size_In_Block), hipMemcpyHostToDevice);

    /********************��ͼ�ֿ�*********************************/
    dim3 oBlock, oGrid;
    int i, iMax_bx = 0, iMax_by = 0;
    for (i = 0; i < poStitch->m_iImage_Count; i++)
    {
        if (poStitch->m_pBlock_Per_Image[i].m_Block_Per_Image[0] > iMax_bx)
            iMax_bx = poStitch->m_pBlock_Per_Image[i].m_Block_Per_Image[0];
        if (poStitch->m_pBlock_Per_Image[i].m_Block_Per_Image[1] > iMax_by)
            iMax_by = poStitch->m_pBlock_Per_Image[i].m_Block_Per_Image[1];
    }

    //�˴��˷ѵ��ڴ�����
    oBlock.x = 32;
    oBlock.y = 32;

    oGrid.x = iMax_bx;
    oGrid.y = iMax_by;
    oGrid.z = 4;    // poStitch->m_iImage_Count;

    poStitch->m_pBlock_Image = (Image*)pMalloc(poStitch->m_iImage_Count * iMax_bx * iMax_by * sizeof(Image));
    poStitch->m_pBlock_Image_Header_GPU = (Image*)pMalloc_GPU(poStitch->m_iImage_Count * iMax_bx * iMax_by * sizeof(Image));
    poStitch->m_pBlock_Image_Data_GPU = (unsigned char*)pMalloc_GPU(poStitch->m_iImage_Count * iMax_bx * iMax_by * 32 * 32 * 4);
    //hipMemset(poStitch->m_pBlock_Image_Data_GPU, 0, poStitch->m_iImage_Count * iMax_bx * iMax_by * 32 * 32 * 4);

    poStitch->m_pBlock_Corner = (int(*)[2])pMalloc(poStitch->m_iImage_Count * iMax_bx * iMax_by * 2 * sizeof(int));
    poStitch->m_pBlock_Corner_GPU = (int(*)[2])pMalloc_GPU(poStitch->m_iImage_Count * iMax_bx * iMax_by * 2 * sizeof(int));

    //�˴��Ǹ�������Ӧ����Warp��ʱ���˳������
    hipMemcpy(poStitch->m_pImage_Warp_Header_GPU, poStitch->m_pImage_Warp,
        poStitch->m_iImage_Count * sizeof(Image), hipMemcpyHostToDevice);

    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    Feed_Split_Image << <oGrid, oBlock >> > (poStitch->m_pImage_Warp_Header_GPU,
        poStitch->m_pBlock_Per_Image,/*iMax_bx,iMax_by,*/
        poStitch->m_pBlock_Image_Header_GPU, poStitch->m_pBlock_Image_Data_GPU,
        Corner, poStitch->m_pBlock_Corner_GPU);

    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);
    /********************��ͼ�ֿ�*********************************/

    //for (i = 0; i < iBlock_Count; i++)
    //{
    //    char File_2[256];
    //    sprintf(File_2, "c:\\tmp\\2\\%04d.bmp", i);
    //    bSave_Image_GPU(File_2, &poStitch->m_pBlock_Image_Header_GPU[i]);
    //    //bSave_Comp_GPU(File_2, &poStitch->m_pBlock_Image_Header_GPU[i], 3);
    //    ////printf("%s\n", File_2);
    //    //char File_1[256];
    //    //sprintf(File_1, "c:\\tmp\\1\\%04d.bmp", i);
    //    //if (!Compare_Image(File_1, File_2))
    //    //    break;
    //}

    Compensator* poComp = &poStitch->m_oComp;
    Feed<_T>(poStitch, poComp);

    Block_Compensate(poStitch,poStitch->m_pImage_Warp,poStitch->m_pImage_Warp_Header_GPU);
    return;
}

int bIs_Overlap(int tl1[2], int tl2[2], int w1, int h1, int w2, int h2, short roi[2][2])
{
    int x_tl = max(tl1[0], tl2[0]);
    int y_tl = max(tl1[1], tl2[1]);
    int x_br = min(tl1[0] + w1, tl2[0] + w2);
    int y_br = min(tl1[1] + h1, tl2[1] + h2);
    if (x_tl < x_br && y_tl < y_br)
    {
        if (roi)
        {
            roi[0][0] = x_tl, roi[0][1] = y_tl;
            roi[1][0] = x_br - x_tl, roi[1][1] = y_br - y_tl;
        }
        return 1;
    }
    return 0;
}

__global__ void Find_Pair_Copy_Image_1(Image Warp[], int Corner[][2][2],
    int iGap, Image_Pair oPair)
{//ͬʱ����dx,dy, Pair����ͼ
    int iThread_ID = GET_THREAD_ID();
    __shared__ Image::Part_1 oImage_A, oImage_B;
    __shared__ int Share_Corner[2][2][2];

    if (threadIdx.x == 0)
    {
        oImage_A = Warp[oPair.m_iImage_A].m_oPart_1;
        oImage_B = Warp[oPair.m_iImage_B].m_oPart_1;
        Share_Corner[0][0][0] = Corner[oPair.m_iImage_A][0][0];
        Share_Corner[0][0][1] = Corner[oPair.m_iImage_A][0][1];
        Share_Corner[0][1][0] = Corner[oPair.m_iImage_A][1][0];
        Share_Corner[0][1][1] = Corner[oPair.m_iImage_A][1][1];
        Share_Corner[1][0][0] = Corner[oPair.m_iImage_B][0][0];
        Share_Corner[1][0][1] = Corner[oPair.m_iImage_B][0][1];
        Share_Corner[1][1][0] = Corner[oPair.m_iImage_B][1][0];
        Share_Corner[1][1][1] = Corner[oPair.m_iImage_B][1][1];
    }
    __syncthreads();

    if (iThread_ID >= oPair.m_oImage_A.m_iWidth * oPair.m_oImage_A.m_iHeight)
        return;

    short y = iThread_ID / oPair.m_oImage_A.m_iWidth;
    short x = iThread_ID % oPair.m_oImage_A.m_iWidth;
    //if (blockIdx.y == 0 && y == 100 /*&& x == 100*/)
    //    printf("Here");

    {
        //��ʾ����ԭͼ��λ��
        short y_A = oPair.roi[0][1] - Share_Corner[0][0][1] + y - iGap;
        short x_A = oPair.roi[0][0] - Share_Corner[0][0][0] + x - iGap;
        //int iPos = (y + iGap) * oPair.m_oImage_A.m_iWidth + x + iGap

        if (y_A >= 0 && x_A >= 0 && y_A < oImage_A.m_iHeight && x_A < oImage_A.m_iWidth)
        {
            int iPos_1 = y_A * oImage_A.m_iWidth + x_A;
            oPair.m_oImage_A.m_pChannel[0][iThread_ID] = oImage_A.m_pChannel[0][iPos_1];
            oPair.m_oImage_A.m_pChannel[1][iThread_ID] = oImage_A.m_pChannel[1][iPos_1];
            oPair.m_oImage_A.m_pChannel[2][iThread_ID] = oImage_A.m_pChannel[2][iPos_1];
            oPair.m_oImage_A.m_pChannel[3][iThread_ID] = oImage_A.m_pChannel[3][iPos_1];
        }
        else
        {
            oPair.m_oImage_A.m_pChannel[0][iThread_ID] =
                oPair.m_oImage_A.m_pChannel[1][iThread_ID] =
                oPair.m_oImage_A.m_pChannel[2][iThread_ID] =
                oPair.m_oImage_A.m_pChannel[3][iThread_ID] = 0;
        }
    }

    {
        //��ʾ����ԭͼ��λ��
        short y_B = oPair.roi[0][1] - Share_Corner[1][0][1] + y - iGap;
        short x_B = oPair.roi[0][0] - Share_Corner[1][0][0] + x - iGap;

        if (y_B >= 0 && x_B >= 0 && y_B < oImage_B.m_iHeight && x_B < oImage_A.m_iWidth)
        {
            int iPos_1 = y_B * oImage_B.m_iWidth + x_B;
            oPair.m_oImage_B.m_pChannel[0][iThread_ID] = oImage_B.m_pChannel[0][iPos_1];
            oPair.m_oImage_B.m_pChannel[1][iThread_ID] = oImage_B.m_pChannel[1][iPos_1];
            oPair.m_oImage_B.m_pChannel[2][iThread_ID] = oImage_B.m_pChannel[2][iPos_1];
            oPair.m_oImage_B.m_pChannel[3][iThread_ID] = oImage_B.m_pChannel[3][iPos_1];
        }
        else
        {
            oPair.m_oImage_B.m_pChannel[0][iThread_ID] =
                oPair.m_oImage_B.m_pChannel[1][iThread_ID] =
                oPair.m_oImage_B.m_pChannel[2][iThread_ID] =
                oPair.m_oImage_B.m_pChannel[3][iThread_ID] = 0;
        }
    }
    return;
}

__global__ void Find_Pair_Copy_Image(Image Warp[], int Corner[][2][2],
    int iGap, int* dx[], int* dy[], Image_Pair Pair[])
{//ͬʱ����dx,dy, Pair����ͼ
    int iThread_ID = GET_THREAD_ID();
    __shared__ Image_Pair oPair;
    __shared__ Image::Part_1 oImage_A, oImage_B;
    __shared__ int Share_Corner[2][2][2];

    if (threadIdx.x == 0)
    {
        oPair = Pair[blockIdx.y];
        oImage_A = Warp[oPair.m_iImage_A].m_oPart_1;
        oImage_B = Warp[oPair.m_iImage_B].m_oPart_1;
        Share_Corner[0][0][0] = Corner[oPair.m_iImage_A][0][0];
        Share_Corner[0][0][1] = Corner[oPair.m_iImage_A][0][1];
        Share_Corner[0][1][0] = Corner[oPair.m_iImage_A][1][0];
        Share_Corner[0][1][1] = Corner[oPair.m_iImage_A][1][1];
        Share_Corner[1][0][0] = Corner[oPair.m_iImage_B][0][0];
        Share_Corner[1][0][1] = Corner[oPair.m_iImage_B][0][1];
        Share_Corner[1][1][0] = Corner[oPair.m_iImage_B][1][0];
        Share_Corner[1][1][1] = Corner[oPair.m_iImage_B][1][1];
    }
    __syncthreads();

    if (iThread_ID >= oPair.m_oImage_A_GPU.m_iWidth * oPair.m_oImage_A_GPU.m_iHeight)
        return;

    short y = iThread_ID / oPair.m_oImage_A_GPU.m_iWidth;
    short x = iThread_ID % oPair.m_oImage_A_GPU.m_iWidth;
    {
        //��ʾ����ԭͼ��λ��
        short y_A = oPair.roi[0][1] - Share_Corner[0][0][1] + y - iGap;
        short x_A = oPair.roi[0][0] - Share_Corner[0][0][0] + x - iGap;
        //int iPos = (y + iGap) * oPair.m_oImage_A.m_iWidth + x + iGap

        if (y_A >= 0 && x_A >= 0 && y_A < oImage_A.m_iHeight && x_A < oImage_A.m_iWidth)
        {
            int iPos_1 = y_A * oImage_A.m_iWidth + x_A;
            oPair.m_oImage_A_GPU.m_pChannel[0][iThread_ID] = oImage_A.m_pChannel[0][iPos_1];
            oPair.m_oImage_A_GPU.m_pChannel[1][iThread_ID] = oImage_A.m_pChannel[1][iPos_1];
            oPair.m_oImage_A_GPU.m_pChannel[2][iThread_ID] = oImage_A.m_pChannel[2][iPos_1];
            oPair.m_oImage_A_GPU.m_pChannel[3][iThread_ID] = oImage_A.m_pChannel[3][iPos_1];
            oPair.m_pSub_dx_A[iThread_ID] = dx[oPair.m_iImage_A][iPos_1];
            oPair.m_pSub_dy_A[iThread_ID] = dy[oPair.m_iImage_A][iPos_1];
            /*if (blockIdx.y == 0 && y == 50 && x == 0)
                printf("%d Thread:%d \n", dx[oPair.m_iImage_A][iPos_1],iThread_ID);*/
        }
        else
        {
            oPair.m_oImage_A_GPU.m_pChannel[0][iThread_ID] =
                oPair.m_oImage_A_GPU.m_pChannel[1][iThread_ID] =
                oPair.m_oImage_A_GPU.m_pChannel[2][iThread_ID] =
                oPair.m_oImage_A_GPU.m_pChannel[3][iThread_ID] = 0;
            oPair.m_pSub_dx_A[iThread_ID] =
                oPair.m_pSub_dy_A[iThread_ID] = 0;
        }
    }

    {
        //��ʾ����ԭͼ��λ��
        short y_B = oPair.roi[0][1] - Share_Corner[1][0][1] + y - iGap;
        short x_B = oPair.roi[0][0] - Share_Corner[1][0][0] + x - iGap;

        if (y_B >= 0 && x_B >= 0 && y_B < oImage_B.m_iHeight && x_B < oImage_A.m_iWidth)
        {
            int iPos_1 = y_B * oImage_B.m_iWidth + x_B;
            oPair.m_oImage_B_GPU.m_pChannel[0][iThread_ID] = oImage_B.m_pChannel[0][iPos_1];
            oPair.m_oImage_B_GPU.m_pChannel[1][iThread_ID] = oImage_B.m_pChannel[1][iPos_1];
            oPair.m_oImage_B_GPU.m_pChannel[2][iThread_ID] = oImage_B.m_pChannel[2][iPos_1];
            oPair.m_oImage_B_GPU.m_pChannel[3][iThread_ID] = oImage_B.m_pChannel[3][iPos_1];
            oPair.m_pSub_dx_B[iThread_ID] = dx[oPair.m_iImage_B][iPos_1];
            oPair.m_pSub_dy_B[iThread_ID] = dy[oPair.m_iImage_B][iPos_1];
            /*if (blockIdx.y == 0 && y == 50 && x == oPair.m_oImage_B.m_iWidth - 1)
                printf("%d %d\n", y_B, x_B);*/
        }
        else
        {
            oPair.m_oImage_B_GPU.m_pChannel[0][iThread_ID] =
                oPair.m_oImage_B_GPU.m_pChannel[1][iThread_ID] =
                oPair.m_oImage_B_GPU.m_pChannel[2][iThread_ID] =
                oPair.m_oImage_B_GPU.m_pChannel[3][iThread_ID] = 0;
            oPair.m_pSub_dx_B[iThread_ID] =
                oPair.m_pSub_dy_B[iThread_ID] = 0;
        }
    }

    return;
}
template<typename _T>float fGet_Distance_Sqr(_T V_1[], _T V_2[], int n)
{//ȡŷ����þ����ƽ��
    float  fTotal = 0;
    for (int i = 0; i < n; i++)
        fTotal += ((float)V_1[i] - V_2[i]) * (V_1[i] - V_2[i]);

    return fTotal;
}
void Add_Edge(GCGraph* poGraph, int i, int j, float w, float revw)
{//�������ӱ�
    if (!poGraph->m_iCur_Edge)
    {
        poGraph->m_Edge[0] = poGraph->m_Edge[1] = {};
        poGraph->m_iCur_Edge = 2;
    }

    Graph_Edge fromI, toI;
    //���С�������ƽ������������Parent�ĵ�һ������
    fromI.dst = j;  //��ʾ�õ������ĸ��㣿
    fromI.next = poGraph->m_Vertex[i].first;
    fromI.weight = w;
    poGraph->m_Vertex[i].first = poGraph->m_iCur_Edge;
    poGraph->m_Edge[poGraph->m_iCur_Edge++] = fromI;
    /*if (poGraph->m_iCur_Edge >= 164775)
        printf("Here");*/

    toI.dst = i;
    toI.next = poGraph->m_Vertex[j].first;
    toI.weight = revw;
    poGraph->m_Vertex[j].first = poGraph->m_iCur_Edge;
    poGraph->m_Edge[poGraph->m_iCur_Edge++] = toI;

}
void Set_Graph_Weights_Color(Image oImage_A, Image oImage_B, GCGraph* poGraph)
{//û�б�Ҫ���Mask��ʵ����Alpha Channel����Mask
    int i, v, x, y, iSize = oImage_A.m_iWidth * oImage_B.m_iHeight;// ,
        //iCur_Edge = 0;
    *poGraph = {};
    Graph_Vertex* pVertex = (Graph_Vertex*)pMalloc(iSize * sizeof(Graph_Vertex));
    poGraph->m_iMax_Vertex_Count = iSize;
    memset(pVertex, 0, iSize * sizeof(Graph_Vertex));

    //�˴�Ӧ�ÿ�������������治��Shrink
    //��ʱ�ȸ�opencv���㷨�������������ܼ�
    poGraph->m_iMax_Edge_Count = (iSize * 2 - oImage_A.m_iWidth - oImage_A.m_iHeight) * 2 + 2; //��2��Ϊ��һ��Դ��һ����㣿

    Graph_Edge* pEdge = (Graph_Edge*)pMalloc(poGraph->m_iMax_Edge_Count * sizeof(Graph_Edge));
    const float terminal_cost_ = 10000;
    for (i = 0; i < iSize; i++)
    {
        Graph_Vertex* pV = &pVertex[i];
        float dw = pV->weight;
        //ע�⣬source��ʾԴ�㣬sink��ʾ���
        /*if (oMask_A.m_pChannel[0][i])
            printf("Here");*/
        float sourceW = oImage_A.m_pChannel[3][i] ? terminal_cost_ : 0;
        float sinkW = oImage_B.m_pChannel[3][i] ? terminal_cost_ : 0;
        if (dw > 0)
            sourceW += dw;
        else
            sinkW -= dw;

        poGraph->flow += (sourceW < sinkW) ? sourceW : sinkW;
        pV->weight = sourceW - sinkW;
        //printf("%f %f\n", poGraph->flow, pV->weight);
    }
    poGraph->m_iMax_Vertex_Count = iSize;
    poGraph->m_Edge = pEdge;
    poGraph->m_Vertex = pVertex;

    const float weight_eps = 1.f, bad_region_penalty_ = 1000;
    int w_1 = oImage_A.m_iWidth - 1,
        h_1 = oImage_A.m_iHeight - 1;
    i = 0;
    for (y = 0; y < oImage_A.m_iHeight; y++)
    {
        for (x = 0; x < oImage_A.m_iWidth; x++, i++)
        {
            //if (y == 271 && x == 136)
            /*if (y == 272 && x == 135)
                printf("Here");*/
            if (x < w_1)
            {
                float weight;
                {//ȡ��ͼ��Ӧ���ؼ�ľ���ƽ������ӳ��Ӧ���ؼ�ò���̶�
                    unsigned char Pix_3_A[] = { oImage_A.m_pChannel[0][i],oImage_A.m_pChannel[1][i],oImage_A.m_pChannel[2][i] },
                        Pix_3_B[] = { oImage_B.m_pChannel[0][i],oImage_B.m_pChannel[1][i],oImage_B.m_pChannel[2][i] };
                    weight = fGet_Distance_Sqr(Pix_3_A, Pix_3_B, 3);
                }

                {//ȡ��Mssk���Ӧλ�õĲ��ƽ��
                    v = i + 1;
                    unsigned char Pix_3_A[] = { oImage_A.m_pChannel[0][v],oImage_A.m_pChannel[1][v],oImage_A.m_pChannel[2][v] },
                        Pix_3_B[] = { oImage_B.m_pChannel[0][v],oImage_B.m_pChannel[1][v],oImage_B.m_pChannel[2][v] };
                    weight += fGet_Distance_Sqr(Pix_3_A, Pix_3_B, 3) + weight_eps;
                }
                if (!oImage_A.m_pChannel[3][i] || !oImage_A.m_pChannel[3][v] ||
                    !oImage_B.m_pChannel[3][i] || !oImage_B.m_pChannel[3][v])
                {
                    weight += bad_region_penalty_;
                }

                //(i,v)Ϊ�ߣ�����������Ȩֵ����֪�ǲ�������������������
                //pEdge[iCur_Edge++] = { i,v,weight,weight };
                Add_Edge(poGraph, i, v, weight, weight);
                //printf("(%d %d)\n", i, v);
            }

            if (y < h_1)
            {
                float weight;
                {
                    unsigned char Pix_3_A[] = { oImage_A.m_pChannel[0][i],oImage_A.m_pChannel[1][i],oImage_A.m_pChannel[2][i] },
                        Pix_3_B[] = { oImage_B.m_pChannel[0][i],oImage_B.m_pChannel[1][i],oImage_B.m_pChannel[2][i] };
                    weight = fGet_Distance_Sqr(Pix_3_A, Pix_3_B, 3);
                }

                {
                    v = i + oImage_A.m_iWidth;
                    unsigned char Pix_3_A[] = { oImage_A.m_pChannel[0][v],oImage_A.m_pChannel[1][v],oImage_A.m_pChannel[2][v] },
                        Pix_3_B[] = { oImage_B.m_pChannel[0][v],oImage_B.m_pChannel[1][v],oImage_B.m_pChannel[2][v] };
                    weight += fGet_Distance_Sqr(Pix_3_A, Pix_3_B, 3) + weight_eps;
                }
                if (!oImage_A.m_pChannel[3][i] || !oImage_A.m_pChannel[3][v] ||
                    !oImage_B.m_pChannel[3][i] || !oImage_B.m_pChannel[3][v])
                    weight += bad_region_penalty_;
                //�ӱ�
                Add_Edge(poGraph, i, v, weight, weight);
                //printf("(%d %d)\n", i, v);
                //pEdge[iCur_Edge++] = { i,v,weight,weight };
            }
        }
    }
    return;
}
void Max_Flow(GCGraph* poGraph)
{//�ȸ�һ�飬����֪�����ͼ��ƥ���й�
    const int TERMINAL = -1, ORPHAN = -2;
    GCGraph oG = *poGraph;
    Graph_Vertex stub = {}, * nilNode = &stub, * first = nilNode, * last = nilNode;
    stub.next = nilNode;

    //Vtx* vtxPtr = &vtcs[0];
    Graph_Edge* edgePtr = oG.m_Edge;
    Graph_Vertex* vtxPtr = oG.m_Vertex;

    int i, curr_ts = 0;
    for (i = 0; i < oG.m_iMax_Vertex_Count; i++)
    {
        Graph_Vertex* v = &oG.m_Vertex[i];
        v->ts = 0;
        if (v->weight != 0)
        {
            last = last->next = v;
            v->dist = 1;
            v->parent = TERMINAL;
            v->t = v->weight < 0;
        }
        else
            v->parent = 0;
    }

    first = first->next;
    last->next = nilNode;
    nilNode->next = 0;


    Graph_Vertex** orphans = (Graph_Vertex**)pMalloc(oG.m_iMax_Vertex_Count * sizeof(Graph_Vertex*));
    int iCur_Orphan = 0;
    int iCount = 0;

    for (;;)
    {
        Graph_Vertex* v, * u;
        int e0 = -1, ei = 0, ej = 0;
        float minWeight, weight;
        unsigned char vt;
        iCount++;
        while (first != nilNode)
        {
            v = first;
            if (v->parent)
            {
                vt = v->t;
                for (ei = v->first; ei != 0; ei = edgePtr[ei].next)
                {
                    if (edgePtr[ei ^ vt].weight == 0)
                        continue;
                    u = vtxPtr + edgePtr[ei].dst;
                    if (!u->parent)
                    {
                        u->t = vt;
                        u->parent = ei ^ 1;
                        u->ts = v->ts;
                        u->dist = v->dist + 1;
                        if (!u->next)
                        {
                            u->next = nilNode;
                            last = last->next = u;
                        }
                        continue;
                    }

                    if (u->t != vt)
                    {
                        e0 = ei ^ vt;
                        break;
                    }

                    if (u->dist > v->dist + 1 && u->ts <= v->ts)
                    {
                        // reassign the parent
                        u->parent = ei ^ 1;
                        u->ts = v->ts;
                        u->dist = v->dist + 1;
                    }
                }
                if (e0 > 0)
                    break;
            }

            // exclude the vertex from the active list
            first = first->next;
            v->next = 0;
        }//while

        if (e0 <= 0)
            break;

        // find the minimum edge weight along the path
        minWeight = edgePtr[e0].weight;

        // k = 1: source tree, k = 0: destination tree
        for (int k = 1; k >= 0; k--)
        {
            for (v = vtxPtr + edgePtr[e0 ^ k].dst;; v = vtxPtr + edgePtr[ei].dst)
            {
                if ((ei = v->parent) < 0)
                    break;
                weight = edgePtr[ei ^ k].weight;
                minWeight = Min(minWeight, weight);
            }
            weight = abs(v->weight);
            minWeight = Min(minWeight, weight);
        }

        // modify weights of the edges along the path and collect orphans
        edgePtr[e0].weight -= minWeight;
        edgePtr[e0 ^ 1].weight += minWeight;
        oG.flow += minWeight;

        // k = 1: source tree, k = 0: destination tree
        for (int k = 1; k >= 0; k--)
        {
            for (v = vtxPtr + edgePtr[e0 ^ k].dst;; v = vtxPtr + edgePtr[ei].dst)
            {
                if ((ei = v->parent) < 0)
                    break;
                edgePtr[ei ^ (k ^ 1)].weight += minWeight;
                if ((edgePtr[ei ^ k].weight -= minWeight) == 0)
                {
                    orphans[iCur_Orphan++] = v;
                    v->parent = ORPHAN;
                }
            }

            v->weight = v->weight + minWeight * (1 - k * 2);
            if (v->weight == 0)
            {
                orphans[iCur_Orphan++] = v;
                v->parent = ORPHAN;
            }
        }//for

        // restore the search trees by finding new parents for the orphans
        curr_ts++;
        while (iCur_Orphan)
        {
            Graph_Vertex* v2 = orphans[--iCur_Orphan];

            int d, minDist = INT_MAX;
            e0 = 0;
            vt = v2->t;

            for (ei = v2->first; ei != 0; ei = edgePtr[ei].next)
            {
                if (edgePtr[ei ^ (vt ^ 1)].weight == 0)
                    continue;
                u = vtxPtr + edgePtr[ei].dst;
                if (u->t != vt || u->parent == 0)
                    continue;
                // compute the distance to the tree root
                for (d = 0;; )
                {
                    if (u->ts == curr_ts)
                    {
                        d += u->dist;
                        break;
                    }
                    ej = u->parent;
                    d++;
                    if (ej < 0)
                    {
                        if (ej == ORPHAN)
                            d = INT_MAX - 1;
                        else
                        {
                            u->ts = curr_ts;
                            u->dist = 1;
                        }
                        break;
                    }
                    u = vtxPtr + edgePtr[ej].dst;
                }

                // update the distance
                if (++d < INT_MAX)
                {
                    if (d < minDist)
                    {
                        minDist = d;
                        e0 = ei;
                    }
                    for (u = vtxPtr + edgePtr[ei].dst; u->ts != curr_ts; u = vtxPtr + edgePtr[u->parent].dst)
                    {
                        u->ts = curr_ts;
                        u->dist = --d;
                    }
                }
            }

            if ((v2->parent = e0) > 0)
            {
                v2->ts = curr_ts;
                v2->dist = minDist;
                continue;
            }

            /* no parent is found */
            v2->ts = 0;
            for (ei = v2->first; ei != 0; ei = edgePtr[ei].next)
            {
                u = vtxPtr + edgePtr[ei].dst;
                ej = u->parent;
                if (u->t != vt || !ej)
                    continue;
                if (edgePtr[ei ^ (vt ^ 1)].weight && !u->next)
                {
                    u->next = nilNode;
                    last = last->next = u;
                }
                if (ej > 0 && vtxPtr + edgePtr[ej].dst == v2)
                {
                    orphans[iCur_Orphan++] = u;
                    u->parent = ORPHAN;
                }
            }

        }//while
    }
    Free(orphans);
    return;
}

int bIn_Source_Segment(GCGraph* poGraph, int i)
{
    return poGraph->m_Vertex[i].t == 0;
}

template<typename _T>void Find_Pair_1(Stitch<_T>* poStitch, int* dx[], int* dy[], Image_Pair** ppPair, Image_Pair** ppImage_Pair_GPU, int* piPair_Count)
{//���ڴ�㣬����ԴΪ Image_Warp ��ͼ���� Mask����
    int iSize, iMax_Size, iPair_Count = 0, iMax_Pair_Count = 10,
        iImage_Count = poStitch->m_iImage_Count;
    const int gap = 10;
    unsigned char* p;
    //����CPU���ҵ�������
    Light_Ptr oPtr;
    iSize = 100000000;
    Attach_Light_Ptr(oPtr, (unsigned char*)pMalloc(iSize), iSize,0);
    Image_Pair *pImage_Pair;

    iSize = iMax_Pair_Count * sizeof(Image_Pair);
    Malloc(oPtr, iSize, p);
    pImage_Pair = (Image_Pair*)p;

    //�Ƚ�Image_Warp��GPU  �������ڴ���
    iSize = iImage_Count * sizeof(Image);
    Malloc(oPtr, iSize, p);
    Image* pImage_Warp = (Image*)p;

    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\image_warp_f_0.bmp", &poStitch->m_pImage_Warp[0]);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\image_warp_f_1.bmp", &poStitch->m_pImage_Warp[1]);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\image_warp_f_2.bmp", &poStitch->m_pImage_Warp[2]);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\image_warp_f_3.bmp", &poStitch->m_pImage_Warp[3]);
    for (int i = 0; i < poStitch->m_iImage_Count; i++)
    {
        int w = poStitch->m_pImage_Warp[i].m_iWidth,
            h = poStitch->m_pImage_Warp[i].m_iHeight;
        Malloc(oPtr, w * h * 4, p);
        Attach_Buffer(&pImage_Warp[i], p, w, h, 4, Image::IMAGE_TYPE_BMP);
        //Copy_Image_To_CPU(poStitch->m_pImage_Warp[i],pImage_Warp[i]);
        //bSave_Image_GPU("c:\\tmp\\1.bmp", poStitch->m_pImage_Warp[0]);
        pImage_Warp[i].m_pChannel[0] = poStitch->m_pImage_Warp[i].m_pChannel[0];
        pImage_Warp[i].m_pChannel[1] = poStitch->m_pImage_Warp[i].m_pChannel[1];
        pImage_Warp[i].m_pChannel[2] = poStitch->m_pImage_Warp[i].m_pChannel[2];
        hipMemcpy(pImage_Warp[i].m_pChannel[3], poStitch->m_pImage_Warp[i].m_pChannel[3],
            pImage_Warp[i].m_iWidth * pImage_Warp[i].m_iHeight, hipMemcpyDeviceToHost);
        //Copy_Image_To_CPU(poStitch->m_pMasks_Warped[i], pImage_Mask[i]);
    }
    
    //�˴�����װ��Warp Image�Ա������
    
    iMax_Size = 0;
    for (int i = 0; i < poStitch->m_iImage_Count; i++)
    {
        for (int j = i + 1; j < poStitch->m_iImage_Count; j++)
        {
            Image_Pair oPair;
            if (bIs_Overlap(poStitch->m_pCorner[i][0], poStitch->m_pCorner[j][0],
                poStitch->m_pImage_Warp[i].m_iWidth, poStitch->m_pImage_Warp[i].m_iHeight,
                poStitch->m_pImage_Warp[j].m_iWidth, poStitch->m_pImage_Warp[j].m_iHeight, oPair.roi))
            {
                if (iPair_Count >= iMax_Pair_Count)
                {
                    printf("Pair count exceed:%d in Find_Pair\n", iMax_Pair_Count);
                    return;
                }

                oPair.m_iImage_A = i, oPair.m_iImage_B = j;
                //Init_Image_GPU(&oPair.m_oImage_A_GPU, oPair.roi[1][0] + 2 * gap, oPair.roi[1][1] + 2 * gap, Image::IMAGE_TYPE_BMP, 32, &oPtr_GPU);
                //Init_Image_GPU(&oPair.m_oImage_B_GPU, oPair.m_oImage_A_GPU.m_iWidth, oPair.m_oImage_A_GPU.m_iHeight, Image::IMAGE_TYPE_BMP, oPair.m_oImage_A_GPU.m_iBit_Count, &oPtr_GPU);
                int w = oPair.roi[1][0] + 2 * gap,
                    h = oPair.roi[1][1] + 2 * gap;
                Malloc(oPtr, w * h*4, p);
                Attach_Buffer(&oPair.m_oImage_A, p, w, h, 4, Image::IMAGE_TYPE_BMP);
                Malloc(oPtr, w * h*4, p);
                Attach_Buffer(&oPair.m_oImage_B, p, w, h, 4, Image::IMAGE_TYPE_BMP);

                if (oPair.m_oImage_A.m_iWidth * oPair.m_oImage_A.m_iHeight > iMax_Size)
                    iMax_Size = oPair.m_oImage_A.m_iWidth * oPair.m_oImage_A.m_iHeight;

                pImage_Pair[iPair_Count] = oPair;
                iPair_Count++;
            }
        }
    }
    //Disp_Cuda_Error();

    /*dim3 oThread, oGrid;
    oThread.x = 512;
    oGrid.x = (iMax_Size + oThread.x - 1) / oThread.x;
    oGrid.y = iPair_Count;*/

    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    //Find_Pair_Copy_Image_1 << <oGrid, oThread >> > (poStitch->m_pImage_Warp_Header_GPU, poStitch->m_pCorner_GPU, gap, pImage_Pair);

    //Disp_Cuda_Error();
    //Image_Pair oPair = pImage_Pair[0];
    //bSave_Image_GPU("c:\\tmp\\1.bmp", oPair.m_oImage_A);
    //bSave_Image_GPU("c:\\tmp\\2.bmp", oPair.m_oImage_B);

    for (int i = 0; i < iPair_Count; i++)
    {
        Image_Pair oPair = pImage_Pair[i];
        dim3 oThread, oGrid;
        iSize = oPair.m_oImage_A.m_iHeight * oPair.m_oImage_A.m_iWidth;
        oThread.x = Min(512, iSize);
        oGrid.x = (iMax_Size + oThread.x - 1) / oThread.x;

        Find_Pair_Copy_Image_1 << <oGrid, oThread >> > (pImage_Warp, poStitch->m_pCorner_GPU, gap, oPair);
        Disp_Cuda_Error();

        GCGraph oGraph;
        Set_Graph_Weights_Color(oPair.m_oImage_A, oPair.m_oImage_B, &oGraph);
        //unsigned long long tStart = iGet_Tick_Count();
        Max_Flow(&oGraph);
        //printf("%lld\n", iGet_Tick_Count() - tStart);
        //printf("flow:%f\n", oGraph.flow);

        int* tl_A = poStitch->m_pCorner[oPair.m_iImage_A][0],
            * tl_B = poStitch->m_pCorner[oPair.m_iImage_B][0];
        Image oImage_A = pImage_Warp[oPair.m_iImage_A],
            oImage_B = pImage_Warp[oPair.m_iImage_B];
        unsigned char* pMask_A = oImage_A.m_pChannel[3],
            * pMask_B = oImage_B.m_pChannel[3];
        
        for (int y = 0; y < oPair.roi[1][1]; y++)
        {
            for (int x = 0; x < oPair.roi[1][0]; x++)
            {
                if (bIn_Source_Segment(&oGraph, (y + gap) * (oPair.roi[1][0] + 2 * gap) + x + gap))
                {
                    if (pMask_A[(oPair.roi[0][1] - tl_A[1] + y) * oImage_A.m_iWidth + oPair.roi[0][0] - tl_A[0] + x])
                        pMask_B[(oPair.roi[0][1] - tl_B[1] + y) * oImage_B.m_iWidth + oPair.roi[0][0] - tl_B[0] + x] = 0;
                }
                else
                {
                    if (pMask_B[(oPair.roi[0][1] - tl_B[1] + y) * oImage_B.m_iWidth + oPair.roi[0][0] - tl_B[0] + x])
                        pMask_A[(oPair.roi[0][1] - tl_A[1] + y) * oImage_A.m_iWidth + oPair.roi[0][0] - tl_A[0] + x] = 0;
                }
            }
        }

        Free(oGraph.m_Edge);
        Free(oGraph.m_Vertex);
        //bSave_Comp("c:\\tmp\\1.bmp", pImage_Warp[1],3);
    }
    /*bSave_Comp("c:\\tmp\\1.bmp", pImage_Warp[0],3);
    bSave_Comp("c:\\tmp\\2.bmp", pImage_Warp[1],3);
    bSave_Comp("c:\\tmp\\3.bmp", pImage_Warp[2],3);
    bSave_Comp("c:\\tmp\\4.bmp", pImage_Warp[3],3);*/


    //�� Image_Warp��Mask
    for (int i = 0; i < iImage_Count; i++)
    {
        hipMemcpy(poStitch->m_pImage_Warp[i].m_pChannel[3], pImage_Warp[i].m_pChannel[3],
            pImage_Warp[i].m_iWidth * pImage_Warp[i].m_iHeight, hipMemcpyHostToDevice);
    }
    Free(oPtr.m_pBuffer);
    return;
}
template<typename _T>void Find_Pair(Stitch<_T>* poStitch, int* dx[], int* dy[], Image_Pair** ppPair, Image_Pair** ppImage_Pair_GPU, int* piPair_Count)
{
    int iSize, iMax_Size, iPair_Count = 0, iMax_Pair_Count = 10;
    const int gap = 10;
    unsigned char* p;

    //����CPU���ҵ�������
    Light_Ptr oPtr, oPtr_GPU;
    iSize = 100000000;
    Attach_Light_Ptr(oPtr_GPU, (unsigned char*)pMalloc_GPU(iSize), iSize,0);
    Image_Pair* pImage_Pair_GPU, * pImage_Pair = (Image_Pair*)pMalloc(iMax_Pair_Count * sizeof(Image_Pair));

    iMax_Size = 0;
    for (int i = 0; i < poStitch->m_iImage_Count; i++)
    {
        for (int j = i + 1; j < poStitch->m_iImage_Count; j++)
        {
            Image_Pair oPair;
            if (bIs_Overlap(poStitch->m_pCorner[i][0], poStitch->m_pCorner[j][0],
                poStitch->m_pImage_Warp[i].m_iWidth, poStitch->m_pImage_Warp[i].m_iHeight,
                poStitch->m_pImage_Warp[j].m_iWidth, poStitch->m_pImage_Warp[j].m_iHeight, oPair.roi))
            {
                if (iPair_Count >= iMax_Pair_Count)
                {
                    printf("Pair count exceed:%d in Find_Pair\n", iMax_Pair_Count);
                    return;
                }

                oPair.m_iImage_A = i, oPair.m_iImage_B = j;
                Init_Image_GPU(&oPair.m_oImage_A_GPU, oPair.roi[1][0] + 2 * gap, oPair.roi[1][1] + 2 * gap, Image::IMAGE_TYPE_BMP, 32, &oPtr_GPU);
                Init_Image_GPU(&oPair.m_oImage_B_GPU, oPair.m_oImage_A_GPU.m_iWidth, oPair.m_oImage_A_GPU.m_iHeight, Image::IMAGE_TYPE_BMP, oPair.m_oImage_A_GPU.m_iBit_Count, &oPtr_GPU);

                iSize = oPair.m_oImage_A_GPU.m_iWidth * oPair.m_oImage_A_GPU.m_iHeight * sizeof(int);
                Malloc(oPtr_GPU, iSize, p);
                oPair.m_pSub_dx_A = (int*)p;
                Malloc(oPtr_GPU, iSize, p);
                oPair.m_pSub_dy_A = (int*)p;
                Malloc(oPtr_GPU, iSize, p);
                oPair.m_pSub_dx_B = (int*)p;
                Malloc(oPtr_GPU, iSize, p);
                oPair.m_pSub_dy_B = (int*)p;

                if (oPair.m_oImage_A_GPU.m_iWidth * oPair.m_oImage_A_GPU.m_iHeight > iMax_Size)
                    iMax_Size = oPair.m_oImage_A_GPU.m_iWidth * oPair.m_oImage_A_GPU.m_iHeight;

                pImage_Pair[iPair_Count] = oPair;
                iPair_Count++;
                //��ӡһ�¿��Կ����Ƿ��ص����ȣ�һ��ͼͬʱ��ͬһ�����϶�ͼ�ص�
                //printf("%d %d\n", i, j);
            }
        }
    }

    Shrink(pImage_Pair, iPair_Count * sizeof(Image_Pair));

    iSize = iPair_Count * sizeof(Image_Pair);
    Malloc(oPtr_GPU, iSize, p);
    pImage_Pair_GPU = (Image_Pair*)p;
    hipMemcpy(pImage_Pair_GPU, pImage_Pair, iSize, hipMemcpyHostToDevice);

    dim3 oThread, oGrid;
    oThread.x = 512;
    oGrid.x = (iMax_Size + oThread.x - 1) / oThread.x;
    oGrid.y = iPair_Count;

    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    Find_Pair_Copy_Image << <oGrid, oThread >> > (poStitch->m_pImage_Warp_Header_GPU, poStitch->m_pCorner_GPU, gap,
        dx, dy, pImage_Pair_GPU);

    Attach_Light_Ptr(oPtr, (unsigned char*)pMalloc(oPtr_GPU.m_iCur), oPtr_GPU.m_iCur, 0);
    hipMemcpy(oPtr.m_pBuffer, oPtr_GPU.m_pBuffer, oPtr_GPU.m_iCur, hipMemcpyDeviceToHost);
    
    
    Image* pImage_Mask = (Image*)pMalloc(poStitch->m_iImage_Count * sizeof(Image));
    for (int i = 0; i < poStitch->m_iImage_Count; i++)
    {
        Init_Image(&pImage_Mask[i], poStitch->m_pMasks_Warped[i].m_iWidth,
            poStitch->m_pMasks_Warped[i].m_iHeight, Image::IMAGE_TYPE_BMP, 8);
        Copy_Image_To_CPU(poStitch->m_pMasks_Warped[i], pImage_Mask[i]);
    }
    Disp_Cuda_Error();
    //bSave_Image("c:\\tmp\\1.bmp", pImage_Mask[0]);

    //�˴�����������i������ʱû����ȥ�Ľ���ֻ�ܱ���Opencv�������
    for (int i = 0; i < iPair_Count; i++)
    {
        GCGraph oGraph;
        Image_Pair oPair = pImage_Pair[i];
        int w = oPair.m_oImage_A_GPU.m_iWidth,
            h = oPair.m_oImage_B_GPU.m_iHeight;
        iSize = w * h * 4;
        p = oPtr.m_pBuffer + (oPair.m_oImage_A_GPU.m_pChannel[0] - oPtr_GPU.m_pBuffer);
        Attach_Buffer(&oPair.m_oImage_A, p, w, h, 4, Image::IMAGE_TYPE_BMP);
        p = oPtr.m_pBuffer + (oPair.m_oImage_B_GPU.m_pChannel[0] - oPtr_GPU.m_pBuffer);
        Attach_Buffer(&oPair.m_oImage_B, p, w, h, 4, Image::IMAGE_TYPE_BMP);

        /*bSave_Image_GPU("c:\\tmp\\1.bmp", oPair.m_oImage_A_GPU);
        bSave_Image("c:\\tmp\\2.bmp", oPair.m_oImage_A);
        Compare_Image("c:\\tmp\\1.bmp", "c:\\tmp\\2.bmp");

        bSave_Image_GPU("c:\\tmp\\1.bmp", oPair.m_oImage_B_GPU);
        bSave_Image("c:\\tmp\\2.bmp", oPair.m_oImage_B);
        Compare_Image("c:\\tmp\\1.bmp", "c:\\tmp\\2.bmp");*/
                
        Set_Graph_Weights_Color(oPair.m_oImage_A, oPair.m_oImage_B, &oGraph);
        Max_Flow(&oGraph);
        printf("flow:%f\n", oGraph.flow);

        //Free_Image(&oPair.m_oImage_A);
        //Free_Image(&oPair.m_oImage_B);

        //Image oMask_A = poStitch->m_pMasks_Warped[i];

        //����������Image��Mask
        Image oMask_A = pImage_Mask[oPair.m_iImage_A],
            oMask_B = pImage_Mask[oPair.m_iImage_B];
        for (int y = 0; y < oPair.roi[1][1]; y++)
        {
            for (int x = 0; x < oPair.roi[1][0]; x++)
            {
                /*if (bIn_Source_Segment(&oGraph, (y + gap) * (roi[1][0] + 2 * gap) + x + gap))
                {
                    if (oMask_A.m_pChannel[0][(roi[0][1] - tl_A[1] + y) * oMask_A.m_iWidth + roi[0][0] - tl_A[0] + x])
                        oMask_B.m_pChannel[0][(roi[0][1] - tl_B[1] + y) * oMask_B.m_iWidth + roi[0][0] - tl_B[0] + x] = 0;
                }
                else
                {
                    if (oMask_B.m_pChannel[0][(roi[0][1] - tl_B[1] + y) * oMask_B.m_iWidth + roi[0][0] - tl_B[0] + x])
                        oMask_A.m_pChannel[0][(roi[0][1] - tl_A[1] + y) * oMask_A.m_iWidth + roi[0][0] - tl_A[0] + x] = 0;
                }*/
            }
        }

        Free(oGraph.m_Edge);
        Free(oGraph.m_Vertex);
    }

    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    //Disp_Cuda_Error();
    //Image_Pair oPair = pImage_Pair[0];
    //Disp_Part(oPair.m_pSub_dx_B, oPair.m_oImage_B.m_iWidth, 0, 50, oPair.m_oImage_B.m_iWidth, 1, "dx");

    //for(int i=0;i<iPair_Count;i++)
    //{
    //    Image_Pair oPair = pImage_Pair[i];
    //    Disp_Sum_GPU(oPair.m_oImage_A.m_pChannel[3], oPair.m_oImage_A.m_iWidth * oPair.m_oImage_A.m_iHeight);
    //    bSave_Comp_GPU("c:\\tmp\\2.bmp", oPair.m_oImage_A, 3);
    //    //Disp_Sum_GPU(oPair.m_pSub_dy_B, oPair.m_oImage_B.m_iWidth * oPair.m_oImage_B.m_iHeight);
    //}
    *ppPair = pImage_Pair;
    *piPair_Count = iPair_Count;
    /*Light_Ptr oPtr;
    Light_Ptr oPtr_GPU;
    int iSize = iMax_Pair_Count* sizeof(Image_Pair) +
        iMax_Pair_Count
    Attach_Light_Ptr(oPtr,(unsigned char*)pMalloc())*/
    //Image(*pImage_Pair)[2] = (Image(*)[2])pMalloc(iPair_Count * 2 * sizeof(Image));
    //Image(*pImage_Pair_GPU)[2] = (Image(*)[2])pMalloc_GPU(iPair_Count * 2 * sizeof(Image));

    return;
}

template<typename _T>__global__ void Set_Sobel_Mid(Image Warp[], int iImage_Count,
    _T* (*Sobel)[2][3], _T* (*Mid)[2][3], int** dx, int** dy, _T* p)
{
    int i, j, k;
    int* p1 = (int*)p;

    for (i = 0; i < iImage_Count; i++)
    {
        Image::Part_1 oImage = Warp[i].m_oPart_1;
        int iSize = oImage.m_iHeight * oImage.m_iWidth;
        dx[i] = p1;
        p1 += iSize;
        p1 = (int*)ALIGN_SIZE_128(p1);
        dy[i] = p1;
        p1 += iSize;
        p1 = (int*)ALIGN_SIZE_128(p1);
    }
    _T* p2 = (_T*)ALIGN_SIZE_128(p1);

    for (i = 0; i < iImage_Count; i++)
    {
        Image::Part_1 oImage = Warp[i].m_oPart_1;
        int iSize = oImage.m_iHeight * oImage.m_iWidth;
        for (j = 0; j < 2; j++)
            for (k = 0; k < 3; k++)
                Sobel[i][j][k] = p2 + (j * 3 + k) * iSize;
        p2 += iSize * 6;

        for (j = 0; j < 2; j++)
            for (k = 0; k < 3; k++)
                Mid[i][j][k] = p2 + (j * 3 + k) * iSize;
        p2 += iSize * 6;
    }
}

template<typename Dest_Type>__global__ void _Get_dx_dy(Image Warp[],
    Pixel_4 oKer_y, Pixel_4 oKer_x, int iKernel_Size, Dest_Type* Mid[][2][3], Dest_Type* B[][2][3],
    int** dx, int** dy, Border_Type iBorder_Type)
{
    int iThread_ID = GET_THREAD_ID();
    __shared__ Image::Part_1 oImage;
    if (threadIdx.x == 0)
        oImage = Warp[blockIdx.y].m_oPart_1;

    __syncthreads();
    if (iThread_ID >= oImage.m_iWidth * oImage.m_iHeight)
        return;         //�����߳��˳�
    short x = iThread_ID % oImage.m_iWidth,
        y = iThread_ID / oImage.m_iWidth;

    short r = iKernel_Size >> 1;
    Dest_Type Total[2][3] = {};
    int iPos;
    for (short i = -r; i <= r; i++)
    {//�˴�����Ҫ����Border_Type
        short y1 = iGet_Border_y_GPU(y + i, oImage.m_iHeight, iBorder_Type, iThread_ID);
        iPos = y1 * oImage.m_iWidth + x;
        Total[0][0] += Mid[blockIdx.y][0][0][iPos] * oKer_y.Data_c[i + r];
        Total[0][1] += Mid[blockIdx.y][0][1][iPos] * oKer_y.Data_c[i + r];
        Total[0][2] += Mid[blockIdx.y][0][2][iPos] * oKer_y.Data_c[i + r];

        Total[1][0] += Mid[blockIdx.y][1][0][iPos] * oKer_x.Data_c[i + r];
        Total[1][1] += Mid[blockIdx.y][1][1][iPos] * oKer_x.Data_c[i + r];
        Total[1][2] += Mid[blockIdx.y][1][2][iPos] * oKer_x.Data_c[i + r];

        /*if (blockIdx.y == 0 && y == 0 && x == 3)
            printf("y+i:%d y1:%d x:%d Mid:%d\n",y+i, y1, x, Mid[blockIdx.y][0][2][iPos]);*/
    }
    /*B[blockIdx.y][0][0][iThread_ID] = Total[0][0];
    B[blockIdx.y][0][1][iThread_ID] = Total[0][1];
    B[blockIdx.y][0][2][iThread_ID] = Total[0][2];*/

    //��һ��dx,dy����ֵ��Χ
    //Mid Ϊ[-1,0,1], [1,2,1] �ľ�������Max(Mid)Ϊ 255*(1+2+1) = 1020
    //Max(dx) = 1028*1020 * 3 = 3,121,200   ֻ����int��ʾ
    dx[blockIdx.y][iThread_ID] =
        Total[0][0] * Total[0][0] +
        Total[0][1] * Total[0][1] +
        Total[0][2] * Total[0][2];

    /*B[blockIdx.y][1][0][iThread_ID] = Total[1][0];
    B[blockIdx.y][1][1][iThread_ID] = Total[1][1];
    B[blockIdx.y][1][2][iThread_ID] = Total[1][2];*/

    dy[blockIdx.y][iThread_ID] =
        Total[1][0] * Total[1][0] +
        Total[1][1] * Total[1][1] +
        Total[1][2] * Total[1][2];

    return;
}

template<typename Dest_Type>__global__ void _Sep_Filter_row(Image Warp[],
    Pixel_4 oKer_x, Pixel_4 oKer_y, int iKernel_Size, Dest_Type* Mid[][2][3], Border_Type iBorder_Type)
{
    int iThread_ID = GET_THREAD_ID();
    __shared__ Image::Part_1 oImage;
    if (threadIdx.x == 0)
    {
        oImage = Warp[blockIdx.y].m_oPart_1;
    }
    __syncthreads();
    if (iThread_ID >= oImage.m_iWidth * oImage.m_iHeight)
        return;         //�����߳��˳�
    short x = iThread_ID % oImage.m_iWidth,
        y = iThread_ID / oImage.m_iWidth;

    /*if (blockIdx.y == 0 && y == oImage.m_iHeight - 1 && x == 1)
        printf("Here");*/

    short r = iKernel_Size >> 1;
    Dest_Type Total[2][3] = {};
    int iPos;
    for (short i = -r; i <= r; i++)
    {//�˴�����Ҫ����Border_Type
        short x1 = iGet_Border_x_GPU(x + i, oImage.m_iWidth, iBorder_Type);
        iPos = y * oImage.m_iWidth + x1;
        Total[0][0] += oImage.m_pChannel[0][iPos] * oKer_x.Data_c[i + r];
        Total[0][1] += oImage.m_pChannel[1][iPos] * oKer_x.Data_c[i + r];
        Total[0][2] += oImage.m_pChannel[2][iPos] * oKer_x.Data_c[i + r];

        Total[1][0] += oImage.m_pChannel[0][iPos] * oKer_y.Data_c[i + r];
        Total[1][1] += oImage.m_pChannel[1][iPos] * oKer_y.Data_c[i + r];
        Total[1][2] += oImage.m_pChannel[2][iPos] * oKer_y.Data_c[i + r];

        //if (blockIdx.y == 0 && y == 0 && x == 3)
            //printf("y:%d x:%d x1:%d Pix:%d Mid:%d\n", y, x, x1, oImage.m_pChannel[2][iPos], Total[0][2]);
    }

    Mid[blockIdx.y][0][0][iThread_ID] = Total[0][0];
    Mid[blockIdx.y][0][1][iThread_ID] = Total[0][1];
    Mid[blockIdx.y][0][2][iThread_ID] = Total[0][2];

    Mid[blockIdx.y][1][0][iThread_ID] = Total[1][0];
    Mid[blockIdx.y][1][1][iThread_ID] = Total[1][1];
    Mid[blockIdx.y][1][2][iThread_ID] = Total[1][2];

    /*if (Mid[0][0][2][3] != 7)
        printf("%d ", Mid[0][0][2][3]);*/

}

template<typename Dest_Type>static void Sep_Filter_2D_2(Image Warp[], int iImage_Count, int iMax_Size,
    Pixel_4 oKer_x, Pixel_4 oKer_y, int iKernel_Size, Dest_Type* B[][2][3], Dest_Type* Mid[][2][3], int** dx, int** dy)
{//�˴��Ѿ���ȫ��׼opencv�����ݣ����ಽ�϶�Ϊһ

    dim3 oThread, oGrid;
    oThread.x = 512;
    oGrid.x = (iMax_Size + oThread.x - 1) / oThread.x;
    oGrid.y = iImage_Count;

    _Sep_Filter_row << <oGrid, oThread >> > (Warp, oKer_x, oKer_y, iKernel_Size, Mid, BORDER_REFLECT101);
    _Get_dx_dy << <oGrid, oThread >> > (Warp, oKer_y, oKer_x, iKernel_Size, Mid, B, dx, dy, BORDER_REFLECT101);

    //Image oImage = Warp[0];
    ////Disp_Sum_GPU<short>(B[0][0][2], oImage.m_iWidth * oImage.m_iHeight);
    ////Disp_Part_GPU<short>(B[0][0][2], oImage.m_iWidth, 0, 0, oImage.m_iWidth, 1, "Sobel");
    //Disp_Part_GPU<short>(&Mid[0][0][2][0], oImage.m_iWidth, 0, 0, oImage.m_iWidth, 1, "Mid");
    //Disp_Part_GPU(oImage.m_pChannel[2], oImage.m_iWidth, 2, 0, 3, 2, "r");
    return;
}

//template<typename Source_Type, typename Dest_Type, typename Kernel_Type>static void Sep_Filter_2D_1
//(Source_Type A[], int w, int h, Kernel_Type Ker_x[], Kernel_Type Ker_y[], int iKernel_Size, Dest_Type B[])
//{
//    Dest_Type* pMid = (Dest_Type*)pMalloc(w * h * sizeof(Dest_Type));
//
//    Sep_Filter_2D_Line(A, w, h, Ker_x, iKernel_Size, pMid);
//    //Disp(pMid, w,h);
//
//    //Matrix_Transpose(pMid, w, h, pMid);
//    //Disp(pMid, 1,w);
//
//    //�����Ժ�pMid ��һ����Ϊh, ��Ϊw��ת�þ��󣬼������ƽ�����
//    Sep_Filter_2D_Line(pMid, h, w, Ker_y, iKernel_Size, B);
//    //Disp(B, 1, w);
//
//    Free(pMid);
//}

template<typename _T>void Find(Stitch<_T>* poStitch)
{//ע�⣬��ʵ��沢����Ҫdx,dy�������ù�����˳�����ˣ����Һ�ʱ����
    typedef short Sobel_Type;
    //��һ������Sobel �����ڴ�
    int i, iMax_Size = 0, iSize, iSize_All = 0;
    Sobel_Type* (*pSobel)[2][3];   //���Ƿ�Warpͼ�����ɷ�����֮���16λ����ֵ
    Sobel_Type* (*pMid)[2][3];    //�м���
    int** dx, ** dy;

    iSize = poStitch->m_iImage_Count * 3 * 2 * sizeof(Sobel_Type*) +    //Sobel
        poStitch->m_iImage_Count * 3 * 2 * sizeof(int*) +               //mid
        poStitch->m_iImage_Count * sizeof(int*);                        //dx,dy

    dx = (int**)pMalloc_GPU(iSize);
    dy = dx + poStitch->m_iImage_Count;
    pSobel = (Sobel_Type * (*)[2][3])(dy + poStitch->m_iImage_Count);
    pMid = (Sobel_Type * (*)[2][3])pSobel + poStitch->m_iImage_Count;

    for (i = 0; i < poStitch->m_iImage_Count; i++)
    {
        Image::Part_1 oImage = poStitch->m_pImage_Warp[i].m_oPart_1;
        iSize = oImage.m_iHeight * oImage.m_iWidth;
        if (iSize > iMax_Size)
            iMax_Size = iSize;
        iSize_All += iSize;
    }

    iSize = iSize_All * 3 * 2 * sizeof(Sobel_Type) +    //Soble
        iSize_All * 3 * 2 * sizeof(Sobel_Type) +        //Mid
        +iSize_All * 2 * sizeof(int) + 128 * 12;          //dx,dy

    Sobel_Type* p = (Sobel_Type*)pMalloc_GPU(iSize);
    poStitch->m_pdx_dy_GPU = (int*)p;
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    Set_Sobel_Mid << <1, 1 >> > (poStitch->m_pImage_Warp_Header_GPU,
        poStitch->m_iImage_Count, pSobel, pMid, dx, dy, p);
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);
    //exit(0);


    //ȫ��ͼһ����һ���ɷ�����
    Pixel_4 oKernel_x;  // = { -1, 0, 1 };
    Pixel_4     oKernel_y = { 1, 2, 1 };
    oKernel_x.Data_c[0] = -1, oKernel_x.Data_c[1] = 0, oKernel_x.Data_c[2] = 1;
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    Sep_Filter_2D_2<Sobel_Type>(poStitch->m_pImage_Warp_Header_GPU, poStitch->m_iImage_Count,
        iMax_Size, oKernel_x, oKernel_y, 3, pSobel, pMid, dx, dy);
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    //Disp_Mem_GPU();
    //Free_GPU(pSobel[0][0][0]);
    Shrink_GPU(poStitch->m_pdx_dy_GPU, iSize_All * 2 * sizeof(int) + 128 * 2);

    //Disp_Mem_GPU();
    //Disp_Cuda_Error();
    //int** dx_1, ** dy_1;
    //dx_1 = (int**)pMalloc(poStitch->m_iImage_Count * 2 * sizeof(int*));
    //dy_1 = dx_1 + poStitch->m_iImage_Count;
    //hipMemcpy(dx_1, dx, poStitch->m_iImage_Count * 2 * sizeof(int*),hipMemcpyDeviceToHost);
    //for(int i=0;i< poStitch->m_iImage_Count;i++)
    //{
    //    Image oImage;
    //    hipMemcpy(&oImage, &poStitch->m_pImage_Warp[i], sizeof(Image), hipMemcpyDeviceToHost);

    //    //for (int j = 0; j < 2; j++)
    //    //{
    //    //    /*for (int k = 0; k < 3; k++)
    //    //        Disp_Sum_GPU<short>(B[i][j][k], oImage.m_iWidth * oImage.m_iHeight);*/
    //    //    Disp_Sum_GPU<short>(B[i][j][0], oImage.m_iWidth * oImage.m_iHeight*3);
    //    //}
    //    Disp_Sum_GPU(dy_1[i], oImage.m_iWidth * oImage.m_iHeight);
    //    printf("\n");
    //}
    //bSave_Image_GPU("c:\\tmp\\1.bmp", poStitch->m_pImage_Warp[0]);
    //����Ҫfind_Pair
    Image_Pair* pPair, * pPair_GPU;
    int iPair_Count;    Disp_Cuda_Error();
    unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<100;i++)
    {
        Find_Pair_1<_T>(poStitch, dx, dy, &pPair, &pPair_GPU, &iPair_Count);
        //Find_Pair<_T>(poStitch, dx, dy, &pPair, &pPair_GPU, &iPair_Count);
        //Free(pPair);
        //Free(pPair[0].m_oImage_A.m_pBuffer);
    }
    Disp_Cuda_Error();
    printf("Max Pool: %lld ms\n", iGet_Tick_Count() - tStart);
    //Free_GPU(pPair[0].m_oImage_A_GPU.m_pBuffer);
    //Free(pPair);
    Free_GPU(poStitch->m_pdx_dy_GPU);
    poStitch->m_pdx_dy_GPU = NULL;
    Free_GPU(dx);
    return;
}

template<typename _T>void Warp_3(Image oImage, _T K[3 * 3], _T R[3 * 3], _T fScale,
    Image* poImage_Warped, Interpolation_Flag iImage_Inter_Type, Interpolation_Flag iMask_Inter_Type,
    int Corner[2][2], Border_Type iImage_Border_Type, Border_Type iMask_Border_Type, Point_Cloud<float>* poPC)
{//���ǳ�����ԭ���÷�����һֱ����Blender Prepare
    //_T* puxmap, * puymap;    //��ʱδ֪
    int Dest_roi[2][2];   //[0][0-1]: x,y [1][0-1]: w,h
    Sphere_Projector<_T> oProjector = {};

    Build_Map<_T>(oImage.m_iWidth, oImage.m_iHeight, K, R, fScale, /*&puxmap, &puymap,*/ Dest_roi, &oProjector, poPC);
    //ע�⣬����ǰ���warp�õ�roi��С�����ϸ�ã��ʴ˴˴�Ҫ����
    Init_Image_GPU(poImage_Warped, Dest_roi[1][0] + 1, Dest_roi[1][1] + 1, Image::IMAGE_TYPE_BMP, 32);
    Re_Map_3_GPU<_T>(oImage, *poImage_Warped, Dest_roi[0][0], Dest_roi[0][1], &oProjector,
        iImage_Inter_Type, iMask_Inter_Type, iImage_Border_Type, iMask_Border_Type);
    //Disp((int*)Dest_roi,2,2,"roi");

    //bSave_Image_GPU("c:\\tmp\\1.bmp", oImage);
    //bSave_Comp_GPU("c:\\tmp\\2.bmp", *poImage_Warped,3);
    if (Corner)
    {
        Corner[0][0] = Dest_roi[0][0];
        Corner[0][1] = Dest_roi[0][1];
        Corner[1][0] = Dest_roi[0][0] + Dest_roi[1][0];
        Corner[1][1] = Dest_roi[0][1] + Dest_roi[1][1];
    }
    //Disp((int*)Dest_roi, 2, 2);
    return;
}

__global__ void _Dilate_col_GPU(Image Warp[])
{
    __shared__ Image::Part_1 oImage;
    int iThread_ID = GET_THREAD_ID();

    if (threadIdx.x == 0)
        oImage = Warp[blockIdx.y].m_oPart_1;
    __syncthreads();
    if (iThread_ID >= oImage.m_iWidth)
        return;
    short iMove_To, y, y1, r = 1;
    unsigned char* pCur = &oImage.m_pChannel[3][iThread_ID + oImage.m_iWidth];

    for (y = 1; y < oImage.m_iHeight; y++, pCur += oImage.m_iWidth)
    {
        if (*pCur != pCur[-oImage.m_iWidth])
        {
            if (*pCur)
            {//��ɫ������
                iMove_To = y - r;
                if (iMove_To < 0)
                    iMove_To = 0;
                unsigned char* pPrev = pCur - oImage.m_iWidth;
                for (y1 = y - 1; y1 >= iMove_To && !*pPrev; y1--)
                    *pPrev = 0xFF, pPrev -= oImage.m_iWidth;
            }
            else
            {//��ɫ������
                iMove_To = y + r;
                if (iMove_To > oImage.m_iHeight)
                    iMove_To = oImage.m_iHeight;
                unsigned char* pNext = pCur;
                for (y1 = y; y1 < iMove_To && !(*pNext); y1++, pNext += oImage.m_iWidth)
                    *pNext = 0xFF;
                y = y1;
                pCur = &oImage.m_pChannel[3][y * oImage.m_iWidth + iThread_ID];
            }
        }
    }
}

__global__ void _Dilate_row_GPU(Image Warp[])
{//������
    __shared__ Image::Part_1 oImage;
    int iThread_ID = GET_THREAD_ID();

    if (threadIdx.x == 0)
        oImage = Warp[blockIdx.y].m_oPart_1;
    __syncthreads();

    if (iThread_ID >= oImage.m_iHeight)
        return;

    short iMove_To, x1, r = 1;
    //short w_8 = (oImage.m_iWidth >> 3) << 3;
    unsigned char* pLine = &oImage.m_pChannel[3][iThread_ID * oImage.m_iWidth];

    for (int x = 1; x < oImage.m_iWidth;)
    {
        if (pLine[x] != pLine[x - 1])
        {//�б仯��Ҫ����
            if (pLine[x])
            {//��ɫ���������
                iMove_To = x - r;
                if (iMove_To < 0)
                    iMove_To = 0;
                for (x1 = x - 1; x1 >= iMove_To && !pLine[x1]; x1--)
                    pLine[x1] = 0xFF;
            }
            else
            {//��ɫ����ǰ�ƽ�
                iMove_To = x + r;
                if (iMove_To > oImage.m_iWidth)
                    iMove_To = oImage.m_iWidth;
                for (x1 = x; x1 < iMove_To && !pLine[x1]; x1++)
                    pLine[x1] = 0xFF;
                x = x1;
            }
        }
        x++;
    }
}

__global__ void _Dilate_row_GPU_1(Image Warp[])
{
    __shared__ Image::Part_1 oImage;
    if (threadIdx.x == 0)
    {
        oImage = Warp[blockIdx.y].m_oPart_1;
    }
    __syncthreads();
    short x = threadIdx.x,
        y = blockIdx.x;

    if (y >= oImage.m_iHeight || x >= oImage.m_iWidth)
        return;

    Pixel_4 oPix;
    oPix = *(Pixel_4*)(&oImage.m_pChannel[3][y * oImage.m_iWidth + x - 1]);
    if (x == 0)
        oPix.Data[0] = oPix.Data[1];
    if (x == oImage.m_iWidth - 1)
        oPix.Data[2] = oPix.Data[1];
    oPix.Data[3] = oPix.Data[1];    //����һ��

    if (oPix.Data[1] == 0)
    {
        if (oPix.Data[0] == 0xFF || oPix.Data[2] == 0xFF)
            oPix.Data[1] = 0xFF;
    }
    __syncthreads();
    if (oPix.Data[1] != oPix.Data[3])
        oImage.m_pChannel[3][y * oImage.m_iWidth + x] = 0xFF;
    //if (y == 1 && x == 126 && blockIdx.y == 0)
        //printf("%d %d %d %d\n", oPix.Data[0], oPix.Data[1], oPix.Data[2], oPix.Data[3]);

}

void Dilate_GPU(Image Warp[], Image Warp_Header_GPU[], int iImage_Count)
{//ֻ��Image��alpha ͨ������
//��������Ѿ���ȫ��׼ opencv

    dim3 oGrid, oThread;
    const int iThread_Per_Block = 1024;
    int iMax_w = 0, iMax_h = 0;
    for (int i = 0; i < iImage_Count; i++)
    {
        Image::Part_1 oImage = Warp[i].m_oPart_1;
        if (oImage.m_iWidth > iMax_w)
            iMax_w = oImage.m_iWidth;
        if (oImage.m_iHeight > iMax_h)
            iMax_h = oImage.m_iHeight;
    }

    if (iMax_h > iThread_Per_Block)
    {
        //�ȸ��з���
        oThread.x = Min(iThread_Per_Block, iMax_h);
        oGrid.x = (iMax_h + oThread.x - 1) / oThread.x;
        oGrid.y = iImage_Count;
        _Dilate_row_GPU << <oGrid, oThread >> > (Warp_Header_GPU);
    }
    else
    {
        oThread.x = Min(iThread_Per_Block, iMax_w);
        oGrid.x = iMax_h;
        oGrid.y = iImage_Count;
        _Dilate_row_GPU_1 << <oGrid, oThread >> > (Warp_Header_GPU);
    }

    oThread.x = Min(iThread_Per_Block, iMax_w);
    oGrid.x = (iMax_w + oThread.x - 1) / oThread.x;
    oGrid.y = iImage_Count;
    _Dilate_col_GPU << <oGrid, oThread >> > (Warp_Header_GPU);
    return;
}

__global__ void _Resize_Bitwise_And(Image Source[], Image Dest[], Border_Type iBorder_Type)
{
    int iThread_ID = GET_THREAD_ID();
    __shared__ Image oSource, oDest;
    __shared__ float f_x, f_y;
    if (threadIdx.x == 0)
    {
        oSource = Source[blockIdx.y];
        oDest = Dest[blockIdx.y];
        f_x = (float)oSource.m_iWidth / oDest.m_iWidth;
        f_y = (float)oSource.m_iHeight / oDest.m_iHeight;
    }
    __syncthreads();

    if (iThread_ID >= oDest.m_iWidth * oDest.m_iHeight)
        return;
    short x_d = iThread_ID % oDest.m_iWidth,
        y_d = iThread_ID / oDest.m_iWidth;

    float w2, w3, x_s_f = (x_d + 0.5f) * f_x - 0.5f;
    unsigned char* pCur_Line, * pNext_Line;

    {
        float y_s_f = (y_d + 0.5f) * f_y - 0.5f;
        int y_s_0 = (int)floor(y_s_f);
        int y_s_1 = y_s_0 + 1;
        w2 = (y_s_1 - y_s_f), w3 = 1.f - w2;
        pCur_Line = &oSource.m_pChannel[3][iGet_Border_y_GPU(y_s_0, oSource.m_iHeight, iBorder_Type) * oSource.m_iWidth];
        pNext_Line = &oSource.m_pChannel[3][iGet_Border_y_GPU(y_s_1, oSource.m_iHeight, iBorder_Type) * oSource.m_iWidth];
    }

    float w0, w1;
    int x_s_0_r, x_s_1_r;
    {
        int x_s_0 = (int)floor(x_s_f);
        int x_s_1 = x_s_0 + 1;
        w0 = (x_s_1 - x_s_f), w1 = 1.f - w0;
        x_s_0_r = iGet_Border_x_GPU(x_s_0, oSource.m_iWidth, iBorder_Type);
        x_s_1_r = iGet_Border_x_GPU(x_s_1, oSource.m_iWidth, iBorder_Type);
    }

    float fValue_0, fValue_1;
    fValue_0 = w0 * pCur_Line[x_s_0_r] + w1 * pCur_Line[x_s_1_r];
    fValue_1 = w0 * pNext_Line[x_s_0_r] + w1 * pNext_Line[x_s_1_r];

    oDest.m_pChannel[3][iThread_ID] &= (unsigned char)(w2 * fValue_0 + w3 * fValue_1 + 0.5);

    return;
}

template<typename _T>void Resize_Bitwise_And(Stitch<_T>* poStitch,
    Image Blend_Warp[], Image Blend_Warp_Header_GPU[])
{
    int iMax_Size = 0;
    Image::Part_1 oImage;
    for (int i = 0; i < poStitch->m_iImage_Count; i++)
    {
        oImage = Blend_Warp[i].m_oPart_1;
        if (oImage.m_iWidth * oImage.m_iHeight > iMax_Size)
            iMax_Size = oImage.m_iWidth * oImage.m_iHeight;
    }

    dim3 oThread, oGrid;
    oThread.x = 512;
    oGrid.x = (iMax_Size + oThread.x - 1) / oThread.x;
    oGrid.y = poStitch->m_iImage_Count;
    //bSave_Comp_GPU("c:\\tmp\\2.bmp", Blend_Warp[2], 3);
    //Compare_Image("c:\\tmp\\1.bmp", "c:\\tmp\\2.bmp", 1);
    _Resize_Bitwise_And << <oGrid, oThread >> > (poStitch->m_pImage_Warp_Header_GPU,
        Blend_Warp_Header_GPU, BORDER_REFLECT);

    /*Disp_Cuda_Error();
    bSave_Comp_GPU("c:\\tmp\\2.bmp", Blend_Warp[2], 3);
    Compare_Image("c:\\tmp\\1.bmp", "c:\\tmp\\2.bmp",1);*/
    return;
}

template<typename _T>void Temp_Compare(const char File[], _T* pBuffer, int w, int h, _T iDiff_Threshold = 0)
{
    FILE* pFile = fopen(File, "rb");
    if (!pFile)
    {
        printf("Fail to open:%s\n", File);
        return;
    }
    _T* pBuffer_1 = (_T*)pMalloc(w * h * sizeof(_T));
    Disp_Cuda_Error();
    hipMemcpy(pBuffer_1, pBuffer, w * h * sizeof(_T), hipMemcpyDeviceToHost);
    const float eps = 0.000001f;
    int iCount = 0;
    for (int y = 0; y < h; y++)
        for (int x = 0; x < w; x++)
        {
            //if (y == 1 && x == 384)
                //printf("here");
            if (typeid(_T) == typeid(float))
            {
                float fValue;
                int iResult = (int)fread(&fValue, 1, 4, pFile);
                if (abs(fValue - pBuffer_1[y * w + x]) > eps)
                {
                    printf("y:%d x:%d Source:%f Dest:%f\n", y, x, fValue, pBuffer_1[y * w + x]);
                    iCount++;
                    return;
                }
            }
            else if (typeid(_T) == typeid(short))
            {
                short iValue;
                int iResult = (int)fread(&iValue, 1, 2, pFile);
                if (abs(iValue - pBuffer_1[y * w + x]) > iDiff_Threshold)
                {
                    printf("y:%d x:%d Source:%d Dest:%d\n", y, x, iValue, pBuffer_1[y * w + x]);
                    iCount++;
                    //return;
                }
            }
        }

    Free(pBuffer_1);
    fclose(pFile);
    printf("Mismatched Count:%d\n", iCount);

    return;
}
template<typename _T>void Temp_Load_Camera(const char* pcFile, Stitch<_T>* poStitch)
{
    Stitch<_T> oStitch = *poStitch;
    Match_Item<_T> oMatch = {};
    int i, j, iResult, iSize, bRet = 0,
        iMatch_Count;

    FILE* pFile = fopen(pcFile, "rb");
    if (!pFile)
    {
        printf("Fail to load:%s in Temp_Load_Camera\n", pcFile);
        exit(0);
        goto END;
    }
       

    //����iCamera_Count
    iResult = (int)fread(&oStitch.m_iImage_Count, 1, 4, pFile);
    if (!iResult || !oStitch.m_iImage_Count)
        goto END;

    iSize = oStitch.m_iImage_Count * sizeof(Camera<_T>);
    oStitch.m_pCamera = (Camera<_T>*)pMalloc(iSize);
    memset(oStitch.m_pCamera, 1, oStitch.m_iImage_Count * sizeof(Camera<_T>));
    for (i = 0; i < oStitch.m_iImage_Count; i++)
    {
        //�ȶ�K
        for (j = 0; j < 3 * 3; j++)
        {
            double fValue;
            iResult = (int)fread(&fValue, 1, sizeof(double), pFile);
            oStitch.m_pCamera[i].K[j] = fValue;
        }

        for (j = 0; j < 3 * 3; j++)
        {
            double fValue;
            iResult = (int)fread(&fValue, 1, sizeof(fValue), pFile);
            oStitch.m_pCamera[i].R[j] = fValue;
        }

        for (j = 0; j < 3; j++)
        {
            double fValue;
            iResult = (int)fread(&fValue, 1, sizeof(fValue), pFile);
            oStitch.m_pCamera[i].t[j] = fValue;
        }
    }

    iResult = (int)fread(&iMatch_Count, 1, 4, pFile);
    oStitch.m_pImage_Match = (Match_Item<_T>*)pMalloc(iMatch_Count * sizeof(Match_Item<_T>));
    for (i = 0; i < iMatch_Count; i++)
    {
        int A, B;
        iResult = (int)fread(&A, 1, sizeof(A), pFile);
        iResult = (int)fread(&B, 1, sizeof(B), pFile);
        oMatch.m_iImage_A = A;
        oMatch.m_iImage_B = B;
        oStitch.m_pImage_Match[i] = oMatch;
    }
    *poStitch = oStitch;
    bRet = 1;

END:
    if (!bRet)
        Free_Stitch(&oStitch);
    if (pFile)
        fclose(pFile);
    return;
}

template<typename _T>void bLoad_Image(Stitch<_T> oStitch)
{
    for (int i = 0; i < oStitch.m_iImage_Count; i++)
    {
        char File[256];
        sprintf(File, "data\\%d.bmp", i);
        if (!bLoad_Image_GPU(File, &oStitch.m_pImage_Source[i]))
            exit(0);
    }
    return;
}

//template<typename _T>void Temp_Load_Image(Stitch<_T>* poStitch)
//{
//    char File[256];
//    //FILE* pFile;
//    int i, iSize;
//    Stitch<_T> oStitch = *poStitch;
//}

template<typename _T>void Set_K(Stitch<_T>* poStitch,
    _T K_s[3 * 3], _T K[3 * 3], _T fAspect)
{
    /*_T fSeam_Work_Aspect = poStitch->seam_work_aspect;
    K[0] = K_s[0] * fSeam_Work_Aspect;
    K[2] = K_s[2] * fSeam_Work_Aspect;
    K[4] = K_s[4] * fSeam_Work_Aspect;
    K[5] = K_s[5] * fSeam_Work_Aspect;*/

    K[0] = K_s[0] * fAspect;
    K[2] = K_s[2] * fAspect;
    K[4] = K_s[4] * fAspect;
    K[5] = K_s[5] * fAspect;
    K[8] = 1.f;
    K[1] = K[3] = K[6] = K[7] = 0.f;
    return;
}


template<typename _T>void Free_Partial(Stitch<_T>* poStitch)
{//��Feed֮���ͷŲ����ڴ�
    //��Ӧ�ͷ��������ڴ棬Seam_Est��Image_Warped��Mask
    Free_Image_GPU(&poStitch->m_pSeam_Est[0]);  //�˴���MaskҲ�ͷ���
    //Free_Image_GPU(&poStitch->m_pImage_Warp[0]);
    //�����������������ɣ�����Ϊ����ʽ
    memset(poStitch->m_pSeam_Est, 0, poStitch->m_iImage_Count * sizeof(Image));
    //memset(poStitch->m_pImage_Warp, 0, poStitch->m_iImage_Count * sizeof(Image));
    //memset(poStitch->m_pMask, 0, poStitch->m_iImage_Count * sizeof(Image));
    //�����������������ɣ�����Ϊ����ʽ
    hipMemset(poStitch->m_pSeam_Est_Header_GPU, 0, poStitch->m_iImage_Count * sizeof(Image));
    //hipMemset(poStitch->m_pImage_Warp_Header_GPU, 0, poStitch->m_iImage_Count * sizeof(Image));
    //hipMemset(poStitch->m_pMask_Header_GPU, 0, poStitch->m_iImage_Count * sizeof(Image));
}


void Result_Roi(int Corner[][2][2], int Size[][2], int iImage_Count, int roi[2][2])
{//�о��������bounding box
    int tl[2] = { std::numeric_limits<int>::max(), std::numeric_limits<int>::max() },
        br[2] = { std::numeric_limits<int>::min(), std::numeric_limits<int>::min() };
    for (int i = 0; i < iImage_Count; i++)
    {
        if (Corner[i][0][0] < tl[0])
            tl[0] = Corner[i][0][0];
        if (Corner[i][0][1] < tl[1])
            tl[1] = Corner[i][0][1];

        if (Corner[i][0][0] + Size[i][0] > br[0])
            br[0] = Corner[i][0][0] + Size[i][0];
        if (Corner[i][0][1] + Size[i][1] > br[1])
            br[1] = Corner[i][0][1] + Size[i][1];
    }
    roi[0][0] = tl[0], roi[0][1] = tl[1],
        roi[1][0] = br[0] - tl[0], roi[1][1] = br[1] - tl[1];
    return;
}
__global__ void Disp_Image(Image::Part_1 Group_1[], Image::Part_1 Group_2[])
{
    int i;
    for (i = 0; i < 6; i++)
        printf("%d %d\n", Group_1[i].m_iHeight, Group_1[i].m_iWidth);
    for (i = 0; i < 6; i++)
        printf("%d %d\n", Group_2[i].m_iHeight, Group_2[i].m_iWidth);
    return;
}
template<typename _T>void Prepare_Blender_1(Stitch<_T>* poStitch, Blender* poBlender)
{
    int roi[2][2];
    for (int i = 0; i < poStitch->m_iImage_Count; i++)
    {
        poStitch->m_pSize[i][0] = poStitch->m_pCorner[i][1][0] - poStitch->m_pCorner[i][0][0] + 1;
        poStitch->m_pSize[i][1] = poStitch->m_pCorner[i][1][1] - poStitch->m_pCorner[i][0][1] + 1;
    }
    //Disp((int*)poStitch->m_pSize, 4, 2);
    Result_Roi(poStitch->m_pCorner, poStitch->m_pSize, poStitch->m_iImage_Count, roi);
    memcpy(poBlender->dst_roi_final, roi, sizeof(roi));

    double max_len = Max(roi[1][0], roi[1][1]);
    //ע�⣬log����eΪ��
    //log(max_len)/log(2) = log2(max_len) �����max_len��ռ����λ
    //������Ŀ�����󻮷�block�����ֵ32�����max_len������32����һ��block
    int num_bands_ = std::min(5, static_cast<int>(ceil(std::log(max_len) / std::log(2.0))));

    int iBlock_Size = 1 << num_bands_;
    roi[1][0] += (iBlock_Size - roi[1][0] % iBlock_Size) % iBlock_Size;
    roi[1][1] += (iBlock_Size - roi[1][1] % iBlock_Size) % iBlock_Size;

    //��ʱ��roi��[1]װ��Ŀ��ͼ�Ĵ�С��Ŀ��ͼ���ý�������
    memcpy(poBlender->dst_roi, roi, sizeof(roi));
    poBlender->m_iNum_Band = num_bands_;

    //����ʹ�ÿռ�
    //����Imageͷ�Ĵ�С
    //�����ڴ�
    int iTotal = 0, iSize = roi[1][0] * roi[1][1];      //Mask?
    //��������һ����Ҫ����
    iTotal += 6 * sizeof(short*);       //dst_pyr_laplace_Header_GPU
    iTotal += 6 * sizeof(float*);        //dst_band_weights
    iTotal += iSize * 3 * sizeof(short) + 128; //m_pDst 23003136
    iTotal += iSize + 128;                    //oMask; 3833856
    iTotal += iSize * sizeof(float) + 128;    //dst_band_weights   15335424 
    //����������Ŀռ���Ҫ
    {
        int Size_1[] = { roi[1][0],roi[1][1] };
        for (int i = 1; i <= num_bands_; ++i)
        {
            Size_1[0] = (Size_1[0] + 1) >> 1;
            Size_1[1] = (Size_1[1] + 1) >> 1;
            iSize = Size_1[0] * Size_1[1];
            iTotal += iSize * 3 * sizeof(short) + 128;    //dst_pyr_laplace[i]
            iTotal += iSize * sizeof(float) + 128;        //dst_band_weights
        }
    }

    Light_Ptr oPtr;
    unsigned char* p;
    Attach_Light_Ptr(oPtr, (unsigned char*)pMalloc_GPU(iTotal), iTotal, 0);
    //�γԱ�Ҫ��
    hipMemset(oPtr.m_pBuffer, 0, oPtr.m_iMax_Buffer_Size);
    poBlender->m_pBuffer = oPtr.m_pBuffer;

    //�ȷ���dst_pyr_laplace
    iSize = roi[1][0] * roi[1][1];
    Image::Part_1 oImage;
    oImage.m_iHeight = roi[1][1];
    oImage.m_iWidth = roi[1][0];
    Malloc(oPtr, iSize * 3 * sizeof(short), oImage.m_pChannel[0]);
    oImage.m_pChannel[1] = oImage.m_pChannel[0] + iSize * sizeof(short);
    oImage.m_pChannel[2] = oImage.m_pChannel[1] + iSize * sizeof(short);
    poBlender->dst_pyr_laplace[0] = oImage;
    for (int i = 1; i < 6; i++)
    {
        Image::Part_1 oUpper, oLower;
        oUpper = poBlender->dst_pyr_laplace[i - 1];
        oLower.m_iHeight = (oUpper.m_iHeight + 1) >> 1;
        oLower.m_iWidth = (oUpper.m_iWidth + 1) >> 1;
        iSize = oLower.m_iHeight * oLower.m_iWidth;
        Malloc(oPtr, iSize * 3 * sizeof(short), oLower.m_pChannel[0]);
        oLower.m_pChannel[1] = oLower.m_pChannel[0] + iSize * sizeof(short);
        oLower.m_pChannel[2] = oLower.m_pChannel[1] + iSize * sizeof(short);
        poBlender->dst_pyr_laplace[i] = oLower;
    }

    //�ٷ���
    iSize = roi[1][0] * roi[1][1];
    oImage.m_iHeight = roi[1][1];
    oImage.m_iWidth = roi[1][0];
    Malloc(oPtr, iSize * sizeof(float), oImage.m_pChannel[0]);
    poBlender->dst_band_weights[0] = oImage;
    for (int i = 1; i < 6; i++)
    {
        Image::Part_1 oUpper, oLower;
        oUpper = poBlender->dst_band_weights[i - 1];
        oLower.m_iHeight = (oUpper.m_iHeight + 1) >> 1;
        oLower.m_iWidth = (oUpper.m_iWidth + 1) >> 1;
        iSize = oLower.m_iHeight * oLower.m_iWidth;
        Malloc(oPtr, iSize * sizeof(float), oLower.m_pChannel[0]);
        poBlender->dst_band_weights[i] = oLower;
    }

    Malloc(oPtr, 12 * sizeof(Image::Part_1), p);
    poBlender->dst_pyr_laplace_Header_GPU = (Image::Part_1*)p;
    poBlender->dst_band_weights_Header_GPU = poBlender->dst_pyr_laplace_Header_GPU + 6;
    hipMemcpy(poBlender->dst_pyr_laplace_Header_GPU, poBlender->dst_pyr_laplace, 12 * sizeof(Image::Part_1), hipMemcpyHostToDevice);

    return;
}

//template<typename _T>void Prepare_Blender(Stitch<_T> *poStitch,Blender* poBlender)
//{//�������
//    int roi[2][2];
//    for (int i = 0; i < poStitch->m_iImage_Count; i++)
//    {
//        poStitch->m_pSize[i][0] = poStitch->m_pCorner[i][1][0] - poStitch->m_pCorner[i][0][0] + 1;
//        poStitch->m_pSize[i][1] = poStitch->m_pCorner[i][1][1] - poStitch->m_pCorner[i][0][1] + 1;
//    }
//    //Disp((int*)poStitch->m_pSize, 4, 2);
//    Result_Roi(poStitch->m_pCorner, poStitch->m_pSize, poStitch->m_iImage_Count, roi);
//    
//    double max_len = Max(roi[1][0], roi[1][1]);
//    //ע�⣬log����eΪ��
//    //log(max_len)/log(2) = log2(max_len) �����max_len��ռ����λ
//    //������Ŀ�����󻮷�block�����ֵ32�����max_len������32����һ��block
//    int num_bands_ = std::min(5, static_cast<int>(ceil(std::log(max_len) / std::log(2.0))));
//
//    int iBlock_Size = 1 << num_bands_;
//    roi[1][0] += (iBlock_Size - roi[1][0] % iBlock_Size) % iBlock_Size;
//    roi[1][1] += (iBlock_Size - roi[1][1] % iBlock_Size) % iBlock_Size;
//
//    memcpy(poBlender->dst_roi, roi, sizeof(roi));
//    poBlender->m_iNum_Band = num_bands_;
//
//    //�����ڴ�
//    int iTotal=0,iSize = roi[1][0] * roi[1][1];
//    //��������һ����Ҫ����
//    iTotal += 6 * sizeof(short*);       //dst_pyr_laplace_Header_GPU
//    iTotal += 6 * sizeof(float*);        //dst_band_weights
//    iTotal += iSize * 3 * sizeof(short) + 128; //m_pDst 23003136
//    iTotal += iSize + 128;                    //oMask; 3833856
//    iTotal += iSize * sizeof(float) + 128;    //dst_band_weights   15335424 
//    {
//        int Size_1[] = { roi[1][0],roi[1][1] };
//        for (int i = 1; i <= num_bands_; ++i)
//        {
//            Size_1[0] = (Size_1[0] + 1) >> 1;
//            Size_1[1] = (Size_1[1] + 1) >> 1;
//            iSize = Size_1[0] * Size_1[1];
//            iTotal += iSize * 3 * sizeof(short) + 128;    //dst_pyr_laplace[i]
//            iTotal += iSize * sizeof(float) + 128;        //dst_band_weights
//        }
//    }
//    Light_Ptr oPtr;
//    unsigned char* p;
//    Attach_Light_Ptr(oPtr, (unsigned char*)pMalloc_GPU(iTotal), iTotal, 0);
//    hipMemset(oPtr.m_pBuffer, 0, oPtr.m_iMax_Buffer_Size);
//    poBlender->m_pBuffer = oPtr.m_pBuffer;
//
//    //��ʽ��ʼ����
//    iSize = 6 * sizeof(short*) + 6 * sizeof(float*);
//    Malloc(oPtr, iSize, p);
//    poBlender->dst_pyr_laplace_Header_GPU = (short**)p;
//    poBlender->dst_band_weights_Header_GPU =(float**)(poBlender->dst_pyr_laplace_Header_GPU + 6);
//
//    //m_pDst
//    iSize = roi[1][0] * roi[1][1];
//    Malloc(oPtr, iSize * 3 * sizeof(short), p);    
//    poBlender->m_pDst = (short(*)[3])p;
//    poBlender->dst_pyr_laplace[0] = (short*)poBlender->m_pDst;
//    //hipMemset(poBlender->m_pDst, 0, iSize * 3 * sizeof(short));
//        
//    Init_Image_GPU(&poBlender->m_oMask, roi[1][0], roi[1][1], Image::IMAGE_TYPE_BMP, 8, &oPtr);
//    //Set_Color_GPU(poBlender->m_oMask);
//
//    //������˹���������ٿռ�
//    Malloc(oPtr, iSize * sizeof(float), p);
//    poBlender->dst_band_weights[0] = (float*)p;
//    //hipMemset(poBlender->dst_band_weights[0], 0, iSize * sizeof(float));
//    {
//        int Size_1[] = { roi[1][0],roi[1][1] };
//        for (int i = 1; i <= num_bands_; ++i)
//        {
//            Size_1[0] = (Size_1[0] + 1) >> 1;
//            Size_1[1] = (Size_1[1] + 1) >> 1;
//            iSize = Size_1[0] * Size_1[1];
//            Malloc(oPtr, iSize * 3 * sizeof(short), p);
//            poBlender->dst_pyr_laplace[i] = (short*)p;
//            //hipMemset(poBlender->dst_pyr_laplace[i], 0, iSize * 3 * sizeof(short));
//
//            Malloc(oPtr, iSize * sizeof(float), p);
//            poBlender->dst_band_weights[i] = (float*)p;
//            //hipMemset(poBlender->dst_band_weights[i], 0, iSize * sizeof(float));
//        }
//    }
//    hipMemcpy(poBlender->dst_pyr_laplace_Header_GPU, poBlender->dst_pyr_laplace, 6 * 2 * 8, hipMemcpyHostToDevice);
//
//    return;
//}

template<typename _T>void Feed_Blender_Get_Pos(Image Warp[], Stitch<_T>* poStitch, Blender* poBlender, int Pos[][2][2], int Size[][2], short LTRB_New[][2][2])
{
    int gap = 3 * (1 << poBlender->m_iNum_Band);
    for (int i = 0; i < poStitch->m_iImage_Count; i++)
    {
        Image::Part_1 oImage = Warp[i].m_oPart_1;
        int* tl = poStitch->m_pCorner[i][0];
        int tl_new[2] = { std::max(poBlender->dst_roi[0][0], tl[0] - gap),
                      std::max(poBlender->dst_roi[0][1], tl[1] - gap) };
        int dst_roi_br[2] = { poBlender->dst_roi[0][0] + poBlender->dst_roi[1][0],
                      poBlender->dst_roi[0][1] + poBlender->dst_roi[1][1] };
        int br_new[2] = { std::min(dst_roi_br[0] , tl[0] + oImage.m_iWidth + gap),
            std::min(dst_roi_br[1], tl[1] + oImage.m_iHeight + gap) };

        tl_new[0] = poBlender->dst_roi[0][0] + (((tl_new[0] - poBlender->dst_roi[0][0]) >> poBlender->m_iNum_Band) << poBlender->m_iNum_Band);
        tl_new[1] = poBlender->dst_roi[0][1] + (((tl_new[1] - poBlender->dst_roi[0][1]) >> poBlender->m_iNum_Band) << poBlender->m_iNum_Band);

        int width = br_new[0] - tl_new[0];
        int height = br_new[1] - tl_new[1];

        width += ((1 << poBlender->m_iNum_Band) - width % (1 << poBlender->m_iNum_Band)) % (1 << poBlender->m_iNum_Band);
        height += ((1 << poBlender->m_iNum_Band) - height % (1 << poBlender->m_iNum_Band)) % (1 << poBlender->m_iNum_Band);

        br_new[0] = tl_new[0] + width;
        br_new[1] = tl_new[1] + height;

        int dy = std::max(br_new[1] - dst_roi_br[1], 0);
        int dx = std::max(br_new[0] - dst_roi_br[0], 0);

        tl_new[0] -= dx; br_new[0] -= dx;
        tl_new[1] -= dy; br_new[1] -= dy;

        int top = tl[1] - tl_new[1];
        int left = tl[0] - tl_new[0];
        int bottom = br_new[1] - tl[1] - oImage.m_iHeight;
        int right = br_new[0] - tl[0] - oImage.m_iWidth;
        //printf("%d %d %d %d\n", top, left, bottom, right);
        Pos[i][0][0] = left + 5;
        Pos[i][0][1] = top;
        Pos[i][1][0] = right;
        Pos[i][1][1] = bottom;
        Size[i][0] = oImage.m_iWidth + Pos[i][0][0] + Pos[i][1][0];
        Size[i][1] = oImage.m_iHeight + Pos[i][0][1] + Pos[i][1][1];

        LTRB_New[i][0][0] = tl_new[0];
        LTRB_New[i][0][1] = tl_new[1];
        LTRB_New[i][1][0] = br_new[0];
        LTRB_New[i][1][1] = br_new[1];

        //if (i == 3)
            //printf("Here");
    }
    //Disp((int*)Pos, 4, 4, "Pos");
    return;
}
__global__ void Copy_Make_Border_1(Image Source[], Image Dest[], int Border[][2][2])
{//��дһ������ͼ���
    __shared__ Image oSource, oDest;
    __shared__ short iTop, iLeft, iRight, iBottom;;
    if (threadIdx.x == 0)
    {
        oSource = Source[blockIdx.y];
        oDest = Dest[blockIdx.y];
        iLeft = Border[blockIdx.y][0][0];
        iTop = Border[blockIdx.y][0][1];
        iRight = Border[blockIdx.y][1][0];
        iBottom = Border[blockIdx.y][1][1];
    }
    __syncthreads();
    short iWidth_Align_4 = (oSource.m_iWidth + 3) >> 2;
    if (threadIdx.x >= iWidth_Align_4 || blockIdx.x >= oSource.m_iHeight)
        return;             //û�õ��߳�����

    int iDest_Size = oDest.m_iWidth * oDest.m_iHeight;
    short x_s = threadIdx.x * 4;      //���̶߳�Ӧ��x����
    short x_d = x_s + iLeft,
        y_d = blockIdx.x + iTop;

    {//�ȳ��м䲿��
        //int iPos_s = blockIdx.x * oSource.m_iWidth + x_s;
        //int iPos_d = y_d * oDest.m_iWidth + x_d;
        int iSource_Size = oSource.m_iWidth * oSource.m_iHeight;
        short iRemain_x = oSource.m_iWidth - x_s;
        unsigned char* pDest = &oDest.m_pChannel[0][y_d * oDest.m_iWidth + x_d],
            * pSource = &oSource.m_pChannel[0][blockIdx.x * oSource.m_iWidth + x_s];

        for (short j = 0; j < oDest.m_iChannel_Count; j++, pSource += iSource_Size, pDest += iDest_Size)
        {
            if (iRemain_x < 4)
            {
                for (short i = 0; i < iRemain_x; i++)
                    pDest[i] = pSource[i];
            }
            else
                *(Pixel_4*)pDest = *(Pixel_4*)pSource;
        }
    }
    __syncthreads();

    //���м䲿�ֳ�����߽�
    if (threadIdx.x < iLeft)
    {
        /* unsigned char* pDest = &oDest.m_pChannel[0][y_d * oDest.m_iWidth + threadIdx.x],
             * pSource = &oDest.m_pChannel[0][y_d * oDest.m_iWidth + iLeft * 2 - threadIdx.x - 1];
         for (int j = 0; j < oDest.m_iChannel_Count; j++,pSource+= iDest_Size,pDest+= iDest_Size)
             *pDest = *pSource;*/
        for (int j = 0; j < oDest.m_iChannel_Count; j++)
            oDest.m_pChannel[j][y_d * oDest.m_iWidth + threadIdx.x] = oDest.m_pChannel[j][y_d * oDest.m_iWidth + iLeft * 2 - threadIdx.x - 1];
    }

    if (threadIdx.x < iRight)
    {
        unsigned char* pDest = &oDest.m_pChannel[0][y_d * oDest.m_iWidth + oDest.m_iWidth - threadIdx.x - 1],
            * pSource = &oDest.m_pChannel[0][y_d * oDest.m_iWidth + oDest.m_iWidth - iRight * 2 + threadIdx.x];
        for (int j = 0; j < oDest.m_iChannel_Count; j++, pSource += iDest_Size, pDest += iDest_Size)
            *pDest = *pSource;
        //for (int j = 0; j < oDest.m_iChannel_Count; j++)
            //oDest.m_pChannel[j][ y_d * oDest.m_iWidth + oDest.m_iWidth - threadIdx.x - 1] = oDest.m_pChannel[j][y_d * oDest.m_iWidth  + oDest.m_iWidth - iRight*2 + threadIdx.x];
    }
    __syncthreads();

    short iWidth_Div_4 = oDest.m_iWidth >> 2;
    //���м�����ݳ�������ȥ
    if (blockIdx.x < iTop)
    {
        //�˴�����Ӧ�ð��Լ������г���Ŀ���ϣ��������Լ�������Ŀ����
        //unsigned char* pDest = &oDest.m_pChannel[0][blockIdx.x * oDest.m_iWidth],
        //    * pSource = &oDest.m_pChannel[0][((iTop << 1) - blockIdx.x - 1) * oDest.m_iWidth];
        unsigned char* pSource = &oDest.m_pChannel[0][y_d * oDest.m_iWidth];
        unsigned char* pDest = &oDest.m_pChannel[0][(iTop - blockIdx.x - 1) * oDest.m_iWidth];

        /*if (blockIdx.y == 2 && threadIdx.x == 0 && blockIdx.x == 26)
            printf("%d\n", iTop - blockIdx.x - 1);*/

        int iDist = oDest.m_iWidth - (iWidth_Div_4 << 2);
        for (short j = 0; j < oDest.m_iChannel_Count; j++, pSource += iDest_Size, pDest += iDest_Size)
        {
            for (short x = threadIdx.x; x < iWidth_Div_4; x += iWidth_Align_4)
            {
                *(Pixel_4*)&pDest[x * 4] = *(Pixel_4*)&pSource[x * 4];
            }

            //for (short x = threadIdx.x; x < iWidth_Div_4; x += blockDim.x)
            //    *(Pixel_4*)&oDest.m_pChannel[j][blockIdx.x * oDest.m_iWidth + x * 4] = *(Pixel_4*)&oDest.m_pChannel[j][(iTop * 2 - blockIdx.x - 1) * oDest.m_iWidth + x * 4];

            //��β����
            if (threadIdx.x < iDist)
                pDest[oDest.m_iWidth - threadIdx.x - 1] = pDest[oDest.m_iWidth - iRight * 2 + threadIdx.x];
            //oDest.m_pChannel[j][blockIdx.x * oDest.m_iWidth + oDest.m_iWidth - threadIdx.x - 1] = oDest.m_pChannel[j][blockIdx.x * oDest.m_iWidth + oDest.m_iWidth - iRight * 2 + threadIdx.x];
        }
    }

    //���м����ݳ����±�ȥ
    if (blockIdx.x >= oSource.m_iHeight - iBottom)
    {
        int iDist_y = oSource.m_iHeight - 1 - blockIdx.x;
        int iDist = oDest.m_iWidth - (iWidth_Div_4 << 2);

        unsigned char* pDest = &oDest.m_pChannel[0][(oDest.m_iHeight - iBottom + iDist_y) * oDest.m_iWidth],
            * pSource = &oDest.m_pChannel[0][(oDest.m_iHeight - iBottom - iDist_y - 1) * oDest.m_iWidth];

        for (int j = 0; j < oDest.m_iChannel_Count; j++, pSource += iDest_Size, pDest += iDest_Size)
        {
            for (short x = threadIdx.x; x < iWidth_Div_4; x += iWidth_Align_4)
            {
                *(Pixel_4*)&pDest[x * 4] = *(Pixel_4*)&pSource[x * 4];
                //*(Pixel_4*)&oDest.m_pChannel[j][(oDest.m_iHeight - iBottom + iDist_y) * oDest.m_iWidth + x * 4] =
                    //*(Pixel_4*)&oDest.m_pChannel[j][(oDest.m_iHeight - iBottom - iDist_y - 1) * oDest.m_iWidth + x * 4];
            }
            //��β����
            if (threadIdx.x < iDist)
                pDest[oDest.m_iWidth - threadIdx.x - 1] = pDest[oDest.m_iWidth - iRight * 2 + threadIdx.x];
            //oDest.m_pChannel[j][(oDest.m_iHeight - iBottom + iDist_y) * oDest.m_iWidth + oDest.m_iWidth - threadIdx.x - 1] = oDest.m_pChannel[j][(oDest.m_iHeight - iBottom + iDist_y) * oDest.m_iWidth + oDest.m_iWidth - iRight * 2 + threadIdx.x];
        }
    }
    return;
}

__global__ void Copy_Make_Border(Image Source[], Image Dest[],
    int Border[][2][2])   //˳�ֽ�����Pytamid
{//Border_REFLECT

    int iThread_ID = GET_THREAD_ID();
    __shared__ Image::Part_1 oSource, oDest;
    __shared__ int iTop, iLeft;
    if (threadIdx.x == 0)
    {
        oSource = Source[blockIdx.y].m_oPart_1;
        oDest = Dest[blockIdx.y].m_oPart_1;
        iLeft = Border[blockIdx.y][0][0];
        iTop = Border[blockIdx.y][0][1];
        //iRight = Border[blockIdx.y][1][0];
        //iBottom = Border[blockIdx.y][1][1];
    }
    __syncthreads();

    if (iThread_ID >= oDest.m_iWidth * oDest.m_iHeight)
        return;
    short y_d = iThread_ID / oDest.m_iWidth,
        x_d = iThread_ID % oDest.m_iWidth;
    short x_s, y_s;
    if (x_d < iLeft)
        x_s = iLeft - x_d - 1;
    else if (x_d >= oSource.m_iWidth + iLeft)
        x_s = oSource.m_iWidth - 1 - (x_d - (oSource.m_iWidth + iLeft));
    else
        x_s = x_d - iLeft;

    if (y_d < iTop)
        y_s = iTop - y_d - 1;
    else if (y_d >= oSource.m_iHeight + iTop)   //ע�⣬�˴�ֻ���� Reflect,����101
        y_s = oSource.m_iHeight - 1 - (y_d - (oSource.m_iHeight + iTop));
    else
        y_s = y_d - iTop;

    //if (blockIdx.y == 0 && y_d == 0 && x_d == 82)
    //{
    //    printf("y_s:%d x_s:%d\n", y_s, x_s);
    //    //printf("%d\n", y_d >= oSource.m_iHeight + iTop);
    //}

    int iPos_d = y_d * oDest.m_iWidth + x_d,
        iPos_s = y_s * oSource.m_iWidth + x_s;
    oDest.m_pChannel[0][iPos_d] = oSource.m_pChannel[0][iPos_s];
    oDest.m_pChannel[1][iPos_d] = oSource.m_pChannel[1][iPos_s];
    oDest.m_pChannel[2][iPos_d] = oSource.m_pChannel[2][iPos_s];

    //if (blockIdx.y == 0 && iThread_ID < 6)
    //{
    //    //printf("%d Thread:%d \n", gridDim.y,iThread_ID);
    //    Pyramid[iThread_ID] = &Dest[gridDim.y * iThread_ID];
    //}
    return;
}
__global__ void _Pyr_Up_col_Add(Image::Part_1 oMid, Image::Part_1 oDest, int iChannel_Count = 3)
{
    int iThread_ID = GET_THREAD_ID();
    int iMid_Size = oMid.m_iHeight * oMid.m_iWidth;
    if (iThread_ID >= iMid_Size)
        return;

    short x = iThread_ID % oDest.m_iWidth,
        y = iThread_ID / oDest.m_iWidth;
    unsigned char bHas_Remain_y = oDest.m_iHeight > oMid.m_iHeight * 2 && y == oMid.m_iHeight - 1 ? 1 : 0;
    unsigned char bEven = (y * 2 + 1 < oDest.m_iHeight);

    //�ظ������ȫ����ȡ�������첻��
    short* r0 = &((short*)oMid.m_pChannel[0])[iGet_Border_y_GPU(y - 1, oMid.m_iHeight, BORDER_REFLECT101) * oDest.m_iWidth + x],
        * r1 = &((short*)oMid.m_pChannel[0])[iGet_Border_y_GPU(y, oMid.m_iHeight, BORDER_REFLECT) * oDest.m_iWidth + x],
        * r2 = &((short*)oMid.m_pChannel[0])[iGet_Border_y_GPU(y + 1, oMid.m_iHeight, BORDER_REFLECT) * oDest.m_iWidth + x];

    int iPos_d = (y * 2) * oDest.m_iWidth + x;

    for (int i = 0; i < iChannel_Count; i++, r0 += iMid_Size, r1 += iMid_Size, r2 += iMid_Size)
    {
        int iValue = ((short*)oDest.m_pChannel[i])[iPos_d] + ((*r0 + *r1 * 6 + *r2 + 32) >> 6);
        //if (y * 2 == 16 && x == 0)
            //printf("%d %d %d\n", ((short*)oDest.m_pChannel[i])[iPos_d], ((*r0 + *r1 * 6 + *r2 + 32) >> 6),iValue);

        ((short*)oDest.m_pChannel[i])[iPos_d] = iValue;
        if (bEven)
        {
            iValue = ((short*)oDest.m_pChannel[i])[iPos_d + oDest.m_iWidth] + ((((*r1 + *r2) << 2) + 32) >> 6);
            ((short*)oDest.m_pChannel[i])[iPos_d + oDest.m_iWidth] = iValue;    // Clip3(-128, 127, iValue);
            if (bHas_Remain_y)
            {
                iValue = ((short*)oDest.m_pChannel[i])[iPos_d + (oDest.m_iWidth << 1)] + (((short*)oDest.m_pChannel[i])[iPos_d]);
                ((short*)oDest.m_pChannel[i])[iPos_d + (oDest.m_iWidth << 1)] = iValue; // Clip3(-128, 127, iValue);
            }
        }
    }
    return;
}
__global__ void _Pyr_Up_col_Subtract_Batch_GPU(Image Source[], unsigned short* Mid[][3], Image Dest[], Border_Type iBorder_Type = BORDER_REFLECT101)
{
    int iThread_ID = GET_THREAD_ID();
    __shared__ Image oDest;
    __shared__ short iMid_Height;
    __shared__ int iMid_Size;
    if (threadIdx.x == 0)
    {
        oDest = Dest[blockIdx.y];
        iMid_Height = Source[blockIdx.y].m_iHeight;
        iMid_Size = (int)(Mid[blockIdx.y][1] - Mid[blockIdx.y][0]);
    }
    __syncthreads();
    if (iThread_ID >= iMid_Height * oDest.m_iWidth)
        return;

    short x = iThread_ID % oDest.m_iWidth,
        y = iThread_ID / oDest.m_iWidth;
    unsigned char bHas_Remain_y = oDest.m_iHeight > iMid_Height * 2 && y == iMid_Height - 1 ? 1 : 0;
    unsigned char bEven = (y * 2 + 1 < oDest.m_iHeight);

    //�ظ������ȫ����ȡ�������첻��
    unsigned short* r0 = &Mid[blockIdx.y][0][iGet_Border_y_GPU(y - 1, iMid_Height, BORDER_REFLECT101) * oDest.m_iWidth + x],
        * r1 = &Mid[blockIdx.y][0][iGet_Border_y_GPU(y, iMid_Height, BORDER_REFLECT) * oDest.m_iWidth + x],
        * r2 = &Mid[blockIdx.y][0][iGet_Border_y_GPU(y + 1, iMid_Height, BORDER_REFLECT) * oDest.m_iWidth + x];

    int iPos_d = (y * 2) * oDest.m_iWidth + x;
    for (int i = 0; i < oDest.m_iChannel_Count; i++, r0 += iMid_Size, r1 += iMid_Size, r2 += iMid_Size)
    {
        int iValue = oDest.m_pChannel[i][iPos_d] - ((*r0 + *r1 * 6 + *r2 + 32) >> 6);
        oDest.m_pChannel[i][iPos_d] = Clip3(-128, 127, iValue);
        if (bEven)
        {
            iValue = oDest.m_pChannel[i][iPos_d + oDest.m_iWidth] - ((((*r1 + *r2) << 2) + 32) >> 6);
            oDest.m_pChannel[i][iPos_d + oDest.m_iWidth] = Clip3(-128, 127, iValue);
            if (bHas_Remain_y)
            {
                iValue = oDest.m_pChannel[i][iPos_d + (oDest.m_iWidth << 1)] - (oDest.m_pChannel[i][iPos_d]);
                oDest.m_pChannel[i][iPos_d + (oDest.m_iWidth << 1)] = Clip3(-128, 127, iValue);
            }
        }
    }
}
__global__ void _Pyr_Up_col_Batch_GPU(Image Source[], unsigned short* Mid[][3], Image Dest[], Border_Type iBorder_Type = BORDER_REFLECT101)
{
    int iThread_ID = GET_THREAD_ID();
    __shared__ Image oDest;
    __shared__ short iMid_Height;
    __shared__ int iMid_Size;
    if (threadIdx.x == 0)
    {
        oDest = Dest[blockIdx.y];
        iMid_Height = Source[blockIdx.y].m_iHeight;
        iMid_Size = (int)(Mid[blockIdx.y][1] - Mid[blockIdx.y][0]);
    }
    __syncthreads();
    if (iThread_ID >= iMid_Height * oDest.m_iWidth)
        return;

    short x = iThread_ID % oDest.m_iWidth,
        y = iThread_ID / oDest.m_iWidth;
    unsigned char bHas_Remain_y = oDest.m_iHeight > iMid_Height * 2 && y == iMid_Height - 1 ? 1 : 0;
    unsigned char bEven = (y * 2 + 1 < oDest.m_iHeight);

    //�ظ������ȫ����ȡ�������첻��
    unsigned short* r0 = &Mid[blockIdx.y][0][iGet_Border_y_GPU(y - 1, iMid_Height, BORDER_REFLECT101) * oDest.m_iWidth + x],
        * r1 = &Mid[blockIdx.y][0][iGet_Border_y_GPU(y, iMid_Height, BORDER_REFLECT) * oDest.m_iWidth + x],
        * r2 = &Mid[blockIdx.y][0][iGet_Border_y_GPU(y + 1, iMid_Height, BORDER_REFLECT) * oDest.m_iWidth + x];

    int iPos_d = (y * 2) * oDest.m_iWidth + x;
    /*if (iThread_ID == 0 && blockIdx.y == 0)
        printf("%d\n", iMid_Size);*/

        //int i = 0;
    for (int i = 0; i < oDest.m_iChannel_Count; i++, r0 += iMid_Size, r1 += iMid_Size, r2 += iMid_Size)
    {
        oDest.m_pChannel[i][iPos_d] = (*r0 + *r1 * 6 + *r2 + 32) >> 6;
        if (bEven)
        {
            //d1�� =   [(r1 + r2)*4 + 32]>>6
            oDest.m_pChannel[i][iPos_d + oDest.m_iWidth] = (((*r1 + *r2) << 2) + 32) >> 6;
            if (bHas_Remain_y)
                oDest.m_pChannel[i][iPos_d + (oDest.m_iWidth << 1)] = oDest.m_pChannel[i][iPos_d];
        }
    }
}

__global__ void _Pyr_Down_col_Batch_GPU(Image Source[], unsigned short* Mid[][3], Image Dest[], Border_Type iBorder_Type = BORDER_REFLECT101)
{
    int iThread_ID = GET_THREAD_ID();
    __shared__ Image oDest;
    __shared__ short iMid_Height;
    if (threadIdx.x == 0)
    {
        oDest = Dest[blockIdx.y];
        iMid_Height = Source[blockIdx.y].m_iHeight;
    }
    __syncthreads();

    //�����ճ���
    if (iThread_ID >= oDest.m_iHeight * oDest.m_iWidth)
        return;
    short x = iThread_ID % oDest.m_iWidth,
        y = iThread_ID / oDest.m_iWidth;

    short y1 = y << 1;
    int iMid_Pos = y1 * oDest.m_iWidth;
    //int iMid_Size = Mid[blockIdx]
    //unsigned short* pSource = &Mid[blockIdx.y][0][x];

    int Mid_Pos[4] = { iGet_Border_y_GPU(y1 - 2, iMid_Height, iBorder_Type) * oDest.m_iWidth,
        iGet_Border_y_GPU(y1 - 1, iMid_Height, iBorder_Type) * oDest.m_iWidth,
        iGet_Border_y_GPU(y1 + 1, iMid_Height, iBorder_Type) * oDest.m_iWidth,
        iGet_Border_y_GPU(y1 + 2, iMid_Height, iBorder_Type) * oDest.m_iWidth };

    //for (int i = 0; i < oDest.m_iChannel_Count; i++/*, pSource += iMid_Size*/)
    //{
    unsigned short* pSource = &Mid[blockIdx.y][0][x];
    unsigned short iValue = pSource[iMid_Pos] * 6 +
        ((pSource[Mid_Pos[1]] + pSource[Mid_Pos[2]]) << 2) +
        pSource[Mid_Pos[0]] + pSource[Mid_Pos[3]];
    oDest.m_pChannel[0][iThread_ID] = (iValue + 128) >> 8;
    //}

    if (oDest.m_iChannel_Count > 1)
    {
        pSource = &Mid[blockIdx.y][1][x];
        iValue = pSource[iMid_Pos] * 6 +
            ((pSource[Mid_Pos[1]] + pSource[Mid_Pos[2]]) << 2) +
            pSource[Mid_Pos[0]] + pSource[Mid_Pos[3]];
        oDest.m_pChannel[1][iThread_ID] = (iValue + 128) >> 8;

        if (oDest.m_iBit_Count > 2)
        {
            pSource = &Mid[blockIdx.y][2][x];
            iValue = pSource[iMid_Pos] * 6 +
                ((pSource[Mid_Pos[1]] + pSource[Mid_Pos[2]]) << 2) +
                pSource[Mid_Pos[0]] + pSource[Mid_Pos[3]];
            oDest.m_pChannel[2][iThread_ID] = (iValue + 128) >> 8;
        }
    }
    return;
}

__global__ void _Pyr_Up_row_Batch_GPU(Image Source[], Image Dest[], unsigned short* Mid[][3], Border_Type iBorder_Type = BORDER_REFLECT101)
{//�˴�Ҫ�õ�Dest��Width���ʴ�ҲҪ����
    int iThread_ID = GET_THREAD_ID();
    __shared__ Image oSource;
    __shared__ short iMid_Width;
    if (threadIdx.x == 0)
    {
        oSource = Source[blockIdx.y];
        iMid_Width = Dest[blockIdx.y].m_iWidth;
    }
    __syncthreads();

    int iSize_s = oSource.m_iWidth * oSource.m_iHeight;
    if (iThread_ID >= iSize_s)
        return;

    short x = iThread_ID % oSource.m_iWidth,
        y = iThread_ID / oSource.m_iWidth;
    unsigned char bHas_Remain_x = iMid_Width > (oSource.m_iWidth << 1) && (x == oSource.m_iWidth - 1) ? 1 : 0;
    unsigned char bIs_Source_Border = (x == oSource.m_iWidth - 1);
    unsigned char bEven = bIs_Source_Border && ((x << 1) + 1 < iMid_Width);

    int iPos_m = y * iMid_Width + (x << 1),
        iPos_s = y * oSource.m_iWidth;

    for (short i = 0; i < oSource.m_iChannel_Count; i++)
    {
        unsigned short* pMid = &Mid[blockIdx.y][i][iPos_m];
        unsigned short Mid[3];
        //�м��
        unsigned char iPix = oSource.m_pChannel[i][iPos_s + x];
        if (x == 0)
        {//�������
            Mid[0] = iPix * 6 + (oSource.m_pChannel[i][iPos_s + 1] << 1);
            Mid[1] = (iPix + oSource.m_pChannel[i][iPos_s + 1]) << 2;
        }
        else if (bIs_Source_Border)
        {//�ұ����㣬2���������
            Mid[0] = oSource.m_pChannel[i][iPos_s + x - 1] + iPix * 7;
            if (bEven)  //Դ�����һ����Զ�����ӦĿ��Ҳ������
                Mid[1] = iPix << 3;
        }
        else
        {//һ�����
            Mid[0] = oSource.m_pChannel[i][iPos_s + x - 1] + iPix * 6 + oSource.m_pChannel[i][iPos_s + x + 1];
            Mid[1] = (iPix + oSource.m_pChannel[i][iPos_s + x + 1]) << 2;
        }

        pMid[0] = Mid[0];
        if (!bIs_Source_Border || bEven)
        {
            pMid[1] = Mid[1];
            if (bHas_Remain_x)  //���������
                pMid[2] = Mid[1];
        }
    }
}
__global__ void _Pyr_Down_row_Batch_GPU(Image Source[], unsigned short* Mid[][3], Border_Type iBorder_Type = BORDER_REFLECT101)
{
    int iThread_ID = GET_THREAD_ID();
    __shared__ Image oSource;
    if (threadIdx.x == 0)
        oSource = Source[blockIdx.y];
    __syncthreads();

    int iMid_Width = (oSource.m_iWidth + 1) >> 1;
    //Ȼ������ճ���
    if (iThread_ID >= oSource.m_iHeight * iMid_Width)
        return;

    short x = iThread_ID % iMid_Width,
        y = iThread_ID / iMid_Width;
    short x1 = x << 1;
    int iSize_s = oSource.m_iWidth * oSource.m_iHeight;
    unsigned char* pSource = &oSource.m_pChannel[0][y * oSource.m_iWidth];
    short Source_Pos[4] = { (short)iGet_Border_x_GPU(x1 - 2, oSource.m_iWidth, iBorder_Type),
        (short)iGet_Border_x_GPU(x1 - 1, oSource.m_iWidth, iBorder_Type),
        (short)iGet_Border_x_GPU(x1 + 1, oSource.m_iWidth, iBorder_Type),
        (short)iGet_Border_x_GPU(x1 + 2, oSource.m_iWidth, iBorder_Type) };

    Mid[blockIdx.y][0][iThread_ID] = pSource[x1] * 6 +
        ((pSource[Source_Pos[1]] + pSource[Source_Pos[2]]) << 2) +
        pSource[Source_Pos[0]] + pSource[Source_Pos[3]];
    pSource += iSize_s;

    if (oSource.m_iChannel_Count > 1)
    {
        Mid[blockIdx.y][1][iThread_ID] = pSource[x1] * 6 +
            ((pSource[Source_Pos[1]] + pSource[Source_Pos[2]]) << 2) +
            pSource[Source_Pos[0]] + pSource[Source_Pos[3]];
        pSource += iSize_s;

        if (oSource.m_iChannel_Count > 2)
            Mid[blockIdx.y][2][iThread_ID] = pSource[x1] * 6 +
            ((pSource[Source_Pos[1]] + pSource[Source_Pos[2]]) << 2) +
            pSource[Source_Pos[0]] + pSource[Source_Pos[3]];
    }
}

void Pyr_Down_Batch_GPU(Image Source[], Image Source_Header_GPU[], int iImage_Count,
    unsigned short* pMid_Header[][3], unsigned short* pMid_Header_GPU[][3],
    Image Dest[], Image Dest_Header_GPU[])
{
    //Ѱ�����Mid���
    int iSize, iMax_Size = 0;
    for (int j = 0; j < iImage_Count; j++)
    {
        Image::Part_1 oSource = Source[j].m_oPart_1;
        iSize = ((oSource.m_iWidth + 1) / 2) * oSource.m_iHeight;
        if (iSize > iMax_Size)
            iMax_Size = iSize;
    }

    //��ʼ�������²���
    dim3 oThread, oGrid;
    oThread.x = Min(512, iMax_Size);
    oGrid.x = (iMax_Size + oThread.x - 1) / oThread.x;
    oGrid.y = iImage_Count;
    _Pyr_Down_row_Batch_GPU << <oGrid, oThread >> > (Source_Header_GPU, pMid_Header_GPU);

    //Ѱ�����ͼ�����
    iMax_Size = 0;
    for (int j = 0; j < iImage_Count; j++)
    {
        Image::Part_1 oDest = Dest[j].m_oPart_1;
        iSize = oDest.m_iWidth * oDest.m_iHeight;
        if (iSize > iMax_Size)
            iMax_Size = iSize;
    }

    oThread.x = Min(512, iMax_Size);
    oGrid.x = (iMax_Size + oThread.x - 1) / oThread.x;
    oGrid.y = iImage_Count;
    _Pyr_Down_col_Batch_GPU << <oGrid, oThread >> > (Source_Header_GPU, pMid_Header_GPU, Dest_Header_GPU);
}
void Pyr_Up_Batch_GPU(Image Source[], Image Source_Header_GPU[], int iImage_Count,
    unsigned short* pMid_Header[][3], unsigned short* pMid_Header_GPU[][3],
    Image Dest[], Image Dest_Header_GPU[])
{
    //Ѱ�����Mid���
    int iSize, iMax_Size = 0;
    for (int j = 0; j < iImage_Count; j++)
    {
        Image::Part_1 oSource = Source[j].m_oPart_1;
        iSize = oSource.m_iWidth * oSource.m_iHeight;
        if (iSize > iMax_Size)
            iMax_Size = iSize;
    }

    //��ʼ�������²���
    dim3 oThread, oGrid;
    oThread.x = Min(512, iMax_Size);
    oGrid.x = (iMax_Size + oThread.x - 1) / oThread.x;
    oGrid.y = iImage_Count;
    _Pyr_Up_row_Batch_GPU << <oGrid, oThread >> > (Source_Header_GPU, Dest_Header_GPU, pMid_Header_GPU);
    //Disp_Cuda_Error();

    iMax_Size = 0;
    for (int j = 0; j < iImage_Count; j++)
    {
        Image::Part_1 oSource = Source[j].m_oPart_1,
            oDest = Dest[j].m_oPart_1;
        iSize = oDest.m_iWidth * oSource.m_iHeight;
        if (iSize > iMax_Size)
            iMax_Size = iSize;
    }

    oThread.x = Min(512, iMax_Size);
    oGrid.x = (iMax_Size + oThread.x - 1) / oThread.x;
    oGrid.y = iImage_Count;

    //�˴�Ӧ��������������
    //_Pyr_Up_col_Batch_GPU << <oGrid, oThread >> > (Source_Header_GPU, pMid_Header_GPU, Dest_Header_GPU);
    _Pyr_Up_col_Subtract_Batch_GPU << <oGrid, oThread >> > (Source_Header_GPU, pMid_Header_GPU, Dest_Header_GPU);
    //Disp_Cuda_Error();
    //bSave_Image_GPU("c:\\tmp\\2.bmp", Dest[0]);
    return;
}

__global__ void _Pyr_Down_row_Batch_float_GPU(Image::Part_1 Source[], Image::Part_1 Mid[], Border_Type iBorder_Type = BORDER_REFLECT101)
{//�д����������
    int iThread_ID = GET_THREAD_ID();
    __shared__ Image::Part_1 oSource, oMid;
    if (threadIdx.x == 0)
    {
        oSource = Source[blockIdx.y];
        oMid = Mid[blockIdx.y];
    }
    __syncthreads();
    if (iThread_ID >= oSource.m_iHeight * oMid.m_iWidth)
        return;

    short x = iThread_ID % oMid.m_iWidth,
        y = iThread_ID / oMid.m_iWidth;
    short x1 = x << 1;
    float* pSource = &((float*)(oSource.m_pChannel[0]))[y * oSource.m_iWidth];
    short Source_Pos[4] = { (short)iGet_Border_x_GPU(x1 - 2, oSource.m_iWidth, iBorder_Type),
        (short)iGet_Border_x_GPU(x1 - 1, oSource.m_iWidth, iBorder_Type),
        (short)iGet_Border_x_GPU(x1 + 1, oSource.m_iWidth, iBorder_Type),
        (short)iGet_Border_x_GPU(x1 + 2, oSource.m_iWidth, iBorder_Type) };

    ((float*)oMid.m_pChannel[0])[iThread_ID] = pSource[x1] * 6 +
        ((pSource[Source_Pos[1]] + pSource[Source_Pos[2]]) * 4) +
        pSource[Source_Pos[0]] + pSource[Source_Pos[3]];

    //if (blockIdx.y == 1 && y == 1 && x == 384)
    //{
    //    //printf("%f\n", ((float*)oMid.m_pChannel[0])[iThread_ID]);
    //    printf("%f %f %f %f %f\n", pSource[Source_Pos[0]], pSource[Source_Pos[1]],
    //        pSource[x1],
    //        pSource[Source_Pos[2]], pSource[Source_Pos[3]]);
    //}
}

__global__ void _Pyr_Down_row_float_GPU(Image::Part_1 oSource, int iMid_Width, float* pMid, Border_Type iBorder_Type = BORDER_REFLECT101)
{//�д����������
    int iThread_ID = GET_THREAD_ID();
    if (iThread_ID >= oSource.m_iHeight * iMid_Width)
        return;

    short x = iThread_ID % iMid_Width,
        y = iThread_ID / iMid_Width;
    short x1 = x << 1;
    float* pSource = &((float*)(oSource.m_pChannel[0]))[y * oSource.m_iWidth];
    short Source_Pos[4] = { (short)iGet_Border_x_GPU(x1 - 2, oSource.m_iWidth, iBorder_Type),
        (short)iGet_Border_x_GPU(x1 - 1, oSource.m_iWidth, iBorder_Type),
        (short)iGet_Border_x_GPU(x1 + 1, oSource.m_iWidth, iBorder_Type),
        (short)iGet_Border_x_GPU(x1 + 2, oSource.m_iWidth, iBorder_Type) };

    pMid[iThread_ID] = pSource[x1] * 6 +
        ((pSource[Source_Pos[1]] + pSource[Source_Pos[2]]) * 4) +
        pSource[Source_Pos[0]] + pSource[Source_Pos[3]];

}

__global__ void _Pyr_Down_col_Batch_float_GPU(Image::Part_1 Mid[], Image::Part_1 Dest[], Border_Type iBorder_Type = BORDER_REFLECT101)
{//�з���
    int iThread_ID = GET_THREAD_ID();
    __shared__ Image::Part_1 oMid, oDest;
    if (threadIdx.x == 0)
    {
        oMid = Mid[blockIdx.y];
        oDest = Dest[blockIdx.y];
    }
    __syncthreads();

    if (iThread_ID >= oDest.m_iHeight * oDest.m_iWidth)
        return;

    short x = iThread_ID % oDest.m_iWidth,
        y = iThread_ID / oDest.m_iWidth;

    short y1 = y << 1;
    int iMid_Pos = y1 * oDest.m_iWidth;
    //int iMid_Size = iMid_Height * oDest.m_iWidth;
    float* pSource = &((float*)oMid.m_pChannel[0])[x];

    int Mid_Pos[4] = { iGet_Border_y_GPU(y1 - 2, oMid.m_iHeight, iBorder_Type) * oDest.m_iWidth,
        iGet_Border_y_GPU(y1 - 1, oMid.m_iHeight, iBorder_Type) * oDest.m_iWidth,
        iGet_Border_y_GPU(y1 + 1, oMid.m_iHeight, iBorder_Type) * oDest.m_iWidth,
        iGet_Border_y_GPU(y1 + 2, oMid.m_iHeight, iBorder_Type) * oDest.m_iWidth };

    float fValue = pSource[iMid_Pos] * 6 +
        ((pSource[Mid_Pos[1]] + pSource[Mid_Pos[2]]) * 4) +
        pSource[Mid_Pos[0]] + pSource[Mid_Pos[3]];
    ((float*)(oDest.m_pChannel[0]))[iThread_ID] = fValue * (1.f / 256.f);

    /*if (blockIdx.y == 1 && y == 0 && x == 271)
        printf("%f %f %f %f %f\n", pSource[Mid_Pos[0]], pSource[Mid_Pos[1]],
            pSource[iMid_Pos],
            pSource[Mid_Pos[2]], pSource[Mid_Pos[3]]);*/
            //printf("%f\n", ((float*)(oDest.m_pChannel[0]))[iThread_ID]);

    return;
}

__global__ void _Pyr_Down_col_float_GPU(float* pMid, int iMid_Height, Image::Part_1 oDest, Border_Type iBorder_Type = BORDER_REFLECT101)
{//�з���
    int iThread_ID = GET_THREAD_ID();
    if (iThread_ID >= oDest.m_iHeight * oDest.m_iWidth)
        return;
    short x = iThread_ID % oDest.m_iWidth,
        y = iThread_ID / oDest.m_iWidth;

    short y1 = y << 1;
    int iMid_Pos = y1 * oDest.m_iWidth;
    //int iMid_Size = iMid_Height * oDest.m_iWidth;
    float* pSource = &pMid[x];
    int Mid_Pos[4] = { iGet_Border_y_GPU(y1 - 2, iMid_Height, iBorder_Type) * oDest.m_iWidth,
        iGet_Border_y_GPU(y1 - 1, iMid_Height, iBorder_Type) * oDest.m_iWidth,
        iGet_Border_y_GPU(y1 + 1, iMid_Height, iBorder_Type) * oDest.m_iWidth,
        iGet_Border_y_GPU(y1 + 2, iMid_Height, iBorder_Type) * oDest.m_iWidth };

    float fValue = pSource[iMid_Pos] * 6 +
        ((pSource[Mid_Pos[1]] + pSource[Mid_Pos[2]]) * 4) +
        pSource[Mid_Pos[0]] + pSource[Mid_Pos[3]];

    ((float*)(oDest.m_pChannel[0]))[iThread_ID] = fValue * (1.f / 256.f);
}

void Pyr_Down_float_GPU(Image::Part_1 oSource, Image::Part_1 oDest, float* pAux)
{//��ͼ������˹�²���
    Data_Block<float*, 1>oMid;
    int iSize = oSource.m_iHeight * oDest.m_iWidth;

    if (pAux)
        oMid.Data[0] = pAux;
    else
        oMid.Data[0] = (float*)pMalloc_GPU(iSize * sizeof(float));

    dim3 oThread, oGrid;
    //�ȸ��з���
    oThread.x = Min(512, iSize);
    oGrid.x = (iSize + oThread.x - 1) / oThread.x;
    _Pyr_Down_row_float_GPU << <oGrid, oThread >> > (oSource, oDest.m_iWidth, oMid.Data[0]);
    //Disp_Part_GPU(oMid.Data[0], oDest.m_iWidth, 342, 12, 10, 1);

    //�ٵ��з���
    iSize = oDest.m_iWidth * oDest.m_iHeight;
    oThread.x = Min(512, iSize);
    oGrid.x = (iSize + oThread.x - 1) / oThread.x;
    _Pyr_Down_col_float_GPU << <oGrid, oThread >> > (oMid.Data[0], oSource.m_iHeight, oDest);

    //Disp_Part_GPU(oMid.Data[0], oDest.m_iWidth, 342, 12, 10, 1);
    //Disp_Part_GPU((float*)oDest.m_pChannel[0], oDest.m_iWidth, 342, 5, 10, 1);
    //Disp_Cuda_Error();
    //Temp_Compare("c:\\tmp\\Dest_1.bin", (float*)oDest.m_pChannel[0], oDest.m_iWidth, oDest.m_iHeight);

    //Disp_Part_GPU((float*)oDest.m_pChannel[0], oDest.m_iWidth, 0, 100, oDest.m_iWidth, 1);
    return;
}
__global__ void Disp_Image_Header(Image::Part_1 Image_Header_GPU[], int iImage_Count)
{
    for (int i = 0; i < iImage_Count; i++)
    {
        Image::Part_1 oImage = Image_Header_GPU[i];
        printf("%d %d\n", oImage.m_iWidth, oImage.m_iHeight);
        for (int j = 0; j < 100; j++)
            printf("%d\n", oImage.m_pChannel[0][j]);
    }

}
void Pyr_Down_Batch_1_Leve_float(Image::Part_1 Source[], Image::Part_1 Dest[], Image::Part_1 Mid_Header[],
    Image::Part_1 Source_Header_GPU[], Image::Part_1 Dest_Header_GPU[], Image::Part_1 Mid_Header_GPU[],
    int iImage_Count, int iLevel = -1)
{
    /*Disp_Image_Header<<<1,1>>>(Source_Header_GPU,iImage_Count);
    Disp_Cuda_Error();*/

    static int iCount = 0;
    {
        int iMax_Size = 0;
        for (int j = 0; j < iImage_Count; j++)
        {
            Image::Part_1 oSource = Source[j],
                oMid = Mid_Header[j];
            oMid.m_iHeight = oSource.m_iHeight;
            oMid.m_iWidth = (oSource.m_iWidth + 1) / 2;
            if (oMid.m_iHeight * oMid.m_iWidth > iMax_Size)
                iMax_Size = oMid.m_iHeight * oMid.m_iWidth;
            Mid_Header[j] = oMid;
        }
        //hipMemcpy(Mid_Header_GPU, Mid_Header, iImage_Count * sizeof(Image::Part_1), hipMemcpyHostToDevice);

        dim3 oThread, oGrid;
        oThread.x = Min(iMax_Size, 512);
        oGrid.x = (iMax_Size + oThread.x - 1) / oThread.x;
        oGrid.y = iImage_Count;
        _Pyr_Down_row_Batch_float_GPU << <oGrid, oThread >> > (Source_Header_GPU, Mid_Header);

        /*Disp_Cuda_Error();
        for (int j = 0; j < iImage_Count; j++)
        {
            Image::Part_1 oMid = Mid_Header[j];
            char File[256];
            sprintf(File, "c:\\tmp\\Weight_Mid_Level_%d_%d.bin", iLevel, j);
            Temp_Compare(File, (float*)oMid.m_pChannel[0], oMid.m_iWidth, oMid.m_iHeight);
        }*/

        //Disp_Part_GPU((float*)pMid_Header[0].m_pChannel[0], pMid_Header[0].m_iWidth, 0, 100, pMid_Header[0].m_iWidth, 1);
        //Disp_Cuda_Error();
        //Temp_Compare("c:\\tmp\\1.bin", (float*)pMid_Header[0].m_pChannel[0], pMid_Header[0].m_iWidth, Src_Pyr_Laplace[0][0].m_iHeight);
        iMax_Size = 0;
        for (int j = 0; j < iImage_Count; j++)
        {
            Image::Part_1 oDest = Dest[j];
            if (oDest.m_iHeight * oDest.m_iWidth > iMax_Size)
                iMax_Size = oDest.m_iHeight * oDest.m_iWidth;
        }
        oThread.x = Min(iMax_Size, 512);
        oGrid.x = (iMax_Size + oThread.x - 1) / oThread.x;
        oGrid.y = iImage_Count;
        _Pyr_Down_col_Batch_float_GPU << <oGrid, oThread >> > (Mid_Header, Dest_Header_GPU);

        /*Disp_Cuda_Error();
        for (int j = 0; j < iImage_Count; j++)
        {
            Image::Part_1 oDest = Dest[j];
            char File[256];
            sprintf(File, "c:\\tmp\\Weight_Level_%d_%d.bin", iLevel, j);
            Temp_Compare(File, (float*)oDest.m_pChannel[0], oDest.m_iWidth, oDest.m_iHeight);
        }*/
    }

    //Disp_Cuda_Error();
    iCount++;
    return;
}

void Pyr_Down_Batch_float(Image::Part_1* Src_Pyr_Laplace[6], Image::Part_1* Src_Pyr_Laplace_Header_GPU[6], int iImage_Count)
{
    Image::Part_1* pMid_Header, * pMid_Header_GPU;
    pMid_Header = (Image::Part_1*)pMalloc(iImage_Count * sizeof(Image::Part_1));
    pMid_Header_GPU = (Image::Part_1*)pMalloc_GPU(iImage_Count * sizeof(Image::Part_1));

    //����Mid �õĿռ�
    int iMid_Size = 0;
    for (int i = 0; i < iImage_Count; i++)
    {
        Image::Part_1 oSource = Src_Pyr_Laplace[0][i],
            oMid;
        oMid.m_iHeight = oSource.m_iHeight;
        oMid.m_iWidth = (oSource.m_iWidth + 1) / 2;
        int iMid_1 = oMid.m_iWidth * oSource.m_iHeight;
        //�м�Mid����ռ䣬�������к��д���
        //�ʴ�Mid����ռ�ʱ Դ��һ�� * Դ��
        //iMid_Size += (oSource.m_iWidth + 1) / 2 * oSource.m_iHeight;
        iMid_Size += iMid_1;
        //oMid.m_pChannel[0] = (unsigned char*)((unsigned long long)iMid_Size*4); //����һ�£������
        pMid_Header[i] = oMid;
    }
    float* p, * pMid = (float*)pMalloc_GPU(iMid_Size * sizeof(float));
    p = pMid;
    for (int i = 0; i < iImage_Count; i++)
    {
        Image::Part_1 oMid = pMid_Header[i];
        oMid.m_pChannel[0] = (unsigned char*)p;
        p += oMid.m_iHeight * oMid.m_iWidth;
        pMid_Header[i] = oMid;
    }
    //hipMemcpy(pMid_Header_GPU, pMid_Header, iImage_Count * sizeof(Image::Part_1), hipMemcpyHostToDevice);

    int iDown_Sample_Thresold = 120000;
    for (int i = 0; i < 5; i++)
    {
        //Disp_Cuda_Error();
        //unsigned long long tStart = iGet_Tick_Count();
        //for (int k = 0; k < 10000; k++)
        {
            Image::Part_1 oImage = Src_Pyr_Laplace[i + 1][0];
            if (oImage.m_iHeight * oImage.m_iWidth < iDown_Sample_Thresold)
            {
                Pyr_Down_Batch_1_Leve_float(Src_Pyr_Laplace[i], Src_Pyr_Laplace[i + 1], pMid_Header,
                    Src_Pyr_Laplace_Header_GPU[i], Src_Pyr_Laplace_Header_GPU[i + 1], pMid_Header_GPU,
                    iImage_Count, i + 1);
                //�˴��и����⣬һ��Ҫ��ͬ��������������pMid_Header��pMid_Header_GPU��Ե��
                Disp_Cuda_Error();
            }
            else
            {
                for (int j = 0; j < iImage_Count; j++)
                    Pyr_Down_float_GPU(Src_Pyr_Laplace[i][j], Src_Pyr_Laplace[i + 1][j], pMid);
            }
        }
        //Disp_Cuda_Error();
        //printf("%lld\n", iGet_Tick_Count() - tStart);
    }

    //exit(0);

    //for (int i = 0; i < 5; i++)
    //{
    //    for (int j = 0; j < iImage_Count; j++)
    //    {
    //        char File[256];
    //        sprintf(File, "c:\\tmp\\Weight_Level_%d_%d.bin", i + 1, j);
    //        Image::Part_1 oDest = Src_Pyr_Laplace[i+1][j];
    //        //printf("Level:%d Image:%d\n", i + 1, j);
    //        Temp_Compare(File, (float*)oDest.m_pChannel[0], oDest.m_iWidth, oDest.m_iHeight);
    //    }
    //}

    //Image::Part_1 oDest = Src_Pyr_Laplace[1][0];
    //Temp_Compare("c:\\tmp\\Dest_1.bin", (float*)oDest.m_pChannel[0], oDest.m_iWidth, oDest.m_iHeight);

    if (pMid_Header)
        Free(pMid_Header);
    if (pMid_Header_GPU)
        Free_GPU(pMid_Header_GPU);
    if (pMid)
        Free_GPU(pMid);
    return;
}
void Pyr_Down_1_float(Image::Part_1* Src_Pyr_Laplace[6], Image::Part_1* Src_Pyr_Laplace_Header_GPU[6], int iImage_Count)
{
    float** pMid_Header, ** pMid_Header_GPU;
    pMid_Header = (float**)pMalloc(iImage_Count * sizeof(float*));
    pMid_Header_GPU = (float**)pMalloc_GPU(iImage_Count * sizeof(float*));

    //����Mid �õĿռ�
    int iMid_Size = 0;
    for (int i = 0; i < iImage_Count; i++)
    {
        Image::Part_1 oSource = Src_Pyr_Laplace[0][i],
            oDest = Src_Pyr_Laplace[1][i];;
        int iMid_1 = (oSource.m_iWidth + 1) / 2 * oSource.m_iHeight,
            iMid_2 = oDest.m_iHeight * oSource.m_iWidth;

        if (iMid_2 > iMid_1)
            printf("Odd value in Create_Laplace_Pyramid\n");
        //�м�Mid����ռ䣬�������к��д���
        //�ʴ�Mid����ռ�ʱ Դ��һ�� * Դ��
        //iMid_Size += (oSource.m_iWidth + 1) / 2 * oSource.m_iHeight;
        iMid_Size += Max(iMid_1, iMid_2);
        pMid_Header[i] = (float*)((unsigned long long)iMid_Size); //����һ�£������
    }
    float* p, * pMid = (float*)pMalloc_GPU(iMid_Size * sizeof(float));

    //Ȼ����pMid_Header ����
    p = pMid;
    for (int i = 0; i < iImage_Count; i++)
    {
        int iMid_Size = (int)((unsigned long long)(pMid_Header[i]));
        pMid_Header[i] = p, p += iMid_Size;
        //printf("here");
    }
    hipMemcpy(pMid_Header_GPU, pMid_Header, iImage_Count * sizeof(float*), hipMemcpyHostToDevice);
    for (int i = 0; i < 5; i++)
    {// //�����²��� �ܺ�ʱ 4900ms
        for (int j = 0; j < iImage_Count; j++)
        {
            //Disp_Cuda_Error();
            //unsigned long long tStart = iGet_Tick_Count();
            //for(int k=0;k<10000;k++)
            Pyr_Down_float_GPU(Src_Pyr_Laplace[i][j], Src_Pyr_Laplace[i + 1][j], pMid);
            //Disp_Cuda_Error();
            //printf("%lld\n", iGet_Tick_Count() - tStart);
            //exit(0);
        }
    }

    //Disp_Part_GPU((float*)Src_Pyr_Laplace[0][1].m_pChannel[0], Src_Pyr_Laplace[0][1].m_iWidth, 
    //    0, 100, Src_Pyr_Laplace[0][1].m_iWidth, 1);
    //Temp_Compare("c:\\tmp\\Dest_1.bin", (float*)(Src_Pyr_Laplace[1][1].m_pChannel[0]), Src_Pyr_Laplace[1][1].m_iWidth, Src_Pyr_Laplace[1][1].m_iHeight);

    ////Pyr_Down_float_GPU(Src_Pyr_Laplace[0][1], Src_Pyr_Laplace[1][1], pMid);
    //for (int i = 1; i < 6; i++)
    //{
    //    char File[256];
    //    sprintf(File, "c:\\tmp\\Dest_%d.bin", i);
    //    int j = 3;
    //    Temp_Compare(File, (float*)(Src_Pyr_Laplace[i][j].m_pChannel[0]), Src_Pyr_Laplace[i][j].m_iWidth, Src_Pyr_Laplace[i][j].m_iHeight);
    //}
    if (pMid_Header)
        Free(pMid_Header);
    if (pMid_Header_GPU)
        Free_GPU(pMid_Header_GPU);
    if (pMid)
        Free_GPU(pMid);
    return;
}
void Create_Laplace_Pyramid(Image* Src_Pyr_Laplace[6], Image* Src_Pyr_Laplace_Header_GPU[6], int iImage_Count)
{
    unsigned short* (*pMid_Header)[3], * (*pMid_Header_GPU)[3];
    pMid_Header = (unsigned short* (*)[3])pMalloc(iImage_Count * 3 * sizeof(unsigned short*));
    //pKernel_GPU = (float*)pMalloc_GPU((r * 2 + 1)*sizeof(float));
    pMid_Header_GPU = (unsigned short* (*)[3])pMalloc_GPU(iImage_Count * 3 * sizeof(unsigned short*));

    //����Mid �õĿռ�
    int iMid_Size = 0;
    for (int i = 0; i < iImage_Count; i++)
    {
        Image::Part_1 oSource = Src_Pyr_Laplace[0][i].m_oPart_1,
            oDest = Src_Pyr_Laplace[1][i].m_oPart_1;
        int iMid_1 = (oSource.m_iWidth + 1) / 2 * oSource.m_iHeight,
            iMid_2 = oDest.m_iHeight * oSource.m_iWidth;

        if (iMid_2 > iMid_1)
            printf("Odd value in Create_Laplace_Pyramid\n");
        //�м�Mid����ռ䣬�������к��д���
        //�ʴ�Mid����ռ�ʱ Դ��һ�� * Դ��
        //iMid_Size += (oSource.m_iWidth + 1) / 2 * oSource.m_iHeight;
        iMid_Size += Max(iMid_1, iMid_2);
        pMid_Header[i][0] = (unsigned short*)((unsigned long long)(iMid_Size)); //����һ�£������
    }
    unsigned short* p, * pMid = (unsigned short*)pMalloc_GPU(iMid_Size * 3 * sizeof(unsigned short));

    //Ȼ����pMid_Header ����
    p = pMid;
    for (int i = 0; i < iImage_Count; i++)
    {
        int iMid_Size = (int)((unsigned long long)(pMid_Header[i][0]));
        for (int j = 0; j < Src_Pyr_Laplace[0][i].m_iChannel_Count; j++)
            pMid_Header[i][j] = p, p += iMid_Size;
        //printf("here");
    }
    hipMemcpy(pMid_Header_GPU, pMid_Header, iImage_Count * 3 * sizeof(unsigned short*), hipMemcpyHostToDevice);

    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();

    //С�������ֵ�����������������ֵ�ô���
    //���ֵֻ��4060�ϳ���������GPU���ò�

    int iDown_Sample_Thresold = 250000;
    //�����²������ӵ�һ�����¸�
    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int k=0;k<10000;k++)
    for (int i = 0; i < 5; i++)
    {// //�����²��� �ܺ�ʱ 4900ms
        //Disp_Cuda_Error();
        //unsigned long long tStart = iGet_Tick_Count();
        //for(int k=0;k<10000;k++)
        {
            Image::Part_1 oImage = Src_Pyr_Laplace[i + 1][0].m_oPart_1;
            if (oImage.m_iHeight * oImage.m_iWidth < iDown_Sample_Thresold)
                Pyr_Down_Batch_GPU(Src_Pyr_Laplace[i], Src_Pyr_Laplace_Header_GPU[i], iImage_Count,
                    pMid_Header, pMid_Header_GPU, Src_Pyr_Laplace[i + 1], Src_Pyr_Laplace_Header_GPU[i + 1]);
            else
            {
                for (int j = 0; j < iImage_Count; j++)
                    Pyr_Down_GPU(Src_Pyr_Laplace[i][j], Src_Pyr_Laplace[i + 1][j], pMid);
            }

            /*char File[256];
            for (int j = 0; j < iImage_Count; j++)
            {
                sprintf(File, "c:\\tmp\\Dest_%d.bmp", j);
                bSave_Image_GPU(File, &Src_Pyr_Laplace_Header_GPU[i + 1][j]);
            }*/

            //for (int j = 0; j < iImage_Count; j++)
            //{
            //    sprintf(File, "c:\\tmp\\Source_%d.bmp", j);
            //    bSave_Image_GPU(File, &Src_Pyr_Laplace_Header_GPU[i + 1][j]);
            //}
            //Compare_Image("c:\\tmp\\Source_0.bmp", "c:\\tmp\\Dest_0.bmp");
            //Compare_Image("c:\\tmp\\Source_1.bmp", "c:\\tmp\\Dest_1.bmp");
            //Compare_Image("c:\\tmp\\Source_2.bmp", "c:\\tmp\\Dest_2.bmp");
            //Compare_Image("c:\\tmp\\Source_3.bmp", "c:\\tmp\\Dest_3.bmp");
            //////printf("Here");
        }
        //Disp_Cuda_Error();
        //printf("%lld\n", iGet_Tick_Count() - tStart);
    }

    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //�ٵ��ϲ���
    iDown_Sample_Thresold = 120000;     //�ϲ������Է�ֵ
    for (int i = 0; i < 5; i++)
    {
        //Disp_Cuda_Error();
        //unsigned long long tStart = iGet_Tick_Count();
        //for (int k = 0; k < 10000; k++)
        {
            Image::Part_1 oImage = Src_Pyr_Laplace[i + 1][0].m_oPart_1;
            if (oImage.m_iHeight * oImage.m_iWidth < iDown_Sample_Thresold)
                Pyr_Up_Batch_GPU(Src_Pyr_Laplace[i + 1], Src_Pyr_Laplace_Header_GPU[i + 1], iImage_Count,
                    pMid_Header, pMid_Header_GPU, Src_Pyr_Laplace[i], Src_Pyr_Laplace_Header_GPU[i]);
            else
            {
                for (int j = 0; j < iImage_Count; j++)
                    Pyr_Up_GPU(Src_Pyr_Laplace[i + 1][j], Src_Pyr_Laplace[i][j]);
            }
        }
        //Disp_GPU((char*)Src_Pyr_Laplace[0][0].m_pChannel[0], 1, 2);

        //Disp_Cuda_Error();
        //printf("%lld\n", iGet_Tick_Count() - tStart);

        /*for (int j = 0; j < iImage_Count; j++)
        {
            char File_2[256], File_1[256];
            sprintf(File_2, "c:\\tmp\\Dest_%d.bmp", j);
            bSave_Image_GPU(File_2, Src_Pyr_Laplace[j]);

            Set_Color_GPU(Src_Pyr_Laplace[i][j]);
            Pyr_Up_GPU(Src_Pyr_Laplace[i + 1][j], Src_Pyr_Laplace[i][j]);
            sprintf(File_1, "c:\\tmp\\Source_%d.bmp", j);
            bSave_Image_GPU(File_1, Src_Pyr_Laplace[i][j]);
            Compare_Image(File_1, File_2);
        }*/
    }

    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);
    //bSave_Image_GPU("c:\\tmp\\1.bmp", Src_Pyr_Laplace[5][0]);


    if (pMid)
        Free_GPU(pMid);
    if (pMid_Header)
        Free(pMid_Header);
    if (pMid_Header_GPU)
        Free_GPU(pMid_Header_GPU);
    return;
}
__global__ void Set_Pyramid_1(Image* Buffer, Image* Pyramid[6], int iImage_Count)
{
    Pyramid[threadIdx.x] = &Buffer[threadIdx.x * iImage_Count];
    return;
}

__global__ void _Copy_Make_Border_float_GPU(Image oSource, Image::Part_1 oDest, short iLeft, short iTop/*, short iRight, short iBottom*/)
{//Դ��unsigned char, һ�����Mask, Ŀ���� float������ oDest.m_pChannel[0]
    //�������ⷽ�����ʴ�û��ͨ�ü�ֵ������Image��, ���߷����� BORDER_CONSTANT
#define FACTOR  1.f/255.f   //ע�⣬�˴��չ�����float, ����6��

    int iThread_ID = GET_THREAD_ID();
    if (iThread_ID >= oSource.m_iWidth * oSource.m_iHeight)
        return;

    int y = iThread_ID / oSource.m_iWidth,
        x = iThread_ID % oSource.m_iWidth;
    //float fValue = oSource.m_pChannel[3][iThread_ID];
    //((float*)(oDest.m_pChannel[0]))[(y + iTop) * oDest.m_iWidth + x + iLeft] = fValue * FACTOR;

    ((float*)(oDest.m_pChannel[0]))[(y + iTop) * oDest.m_iWidth + x + iLeft] = (float)oSource.m_pChannel[3][iThread_ID] * (float)FACTOR;
    return;
#undef FACTOR
}


void Copy_Make_Border_float_GPU(Image oSource, Image::Part_1 oDest, short iLeft, short iTop, short iRight, short iBottom)
{//�������ü�ֵ��ֱ����BORDER_CONSTANT

    int iSize = oSource.m_iHeight * oSource.m_iWidth;
    dim3 oThread, oGrid;
    oThread.x = Min(512, iSize);
    oGrid.x = (iSize + oThread.x) / oThread.x;
    oGrid.y = 1;

    //������0���ڲ��ڱ�
    //hipMemset(oDest.m_pChannel[0], 0, oDest.m_iWidth * oDest.m_iHeight * sizeof(float));
    //�򵥷�������������ʰ��
    _Copy_Make_Border_float_GPU << <oGrid, oThread >> > (oSource, oDest, iLeft, iTop);

    //Disp_Cuda_Error();
    //Disp_Part_GPU(oSource.m_pChannel[3], oSource.m_iWidth, 686, 1, 10, 1);
    //Disp_Part_GPU((float*)oDest.m_pChannel[0], oDest.m_iWidth, 0, 100, oDest.m_iWidth, 1);
    //Disp_Cuda_Error();
    //Temp_Compare("c:\\tmp\\1.bin", (float*)oDest.m_pChannel[0], oDest.m_iWidth, oDest.m_iHeight);

    return;
}
__global__ void _Feed_Blender_1_GPU(Image oSource, Image::Part_1 oSource_Weight,
    Image::Part_1 oDest, Image::Part_1 oDest_Weight, short iLeft, short iTop, char bSigned)
{//��oWeight�Ĵ�СΪ׼
    int iThread_ID = GET_THREAD_ID();
    if (iThread_ID >= oSource_Weight.m_iHeight * oSource_Weight.m_iWidth)
        return;
    unsigned short y = iThread_ID / oSource_Weight.m_iWidth,
        x = iThread_ID % oSource_Weight.m_iWidth;
    float fWeight = ((float*)oSource_Weight.m_pChannel[0])[y * oSource_Weight.m_iWidth + x];
    int iSource_Pos = y * oSource.m_iWidth + x;
    //int iWeight_Pos = y * oSource_Weight.m_iWidth + x;
    int iDest_Pos = (y + iTop) * oDest.m_iWidth + x + iLeft;

    if (bSigned)
    {
        ((short*)oDest.m_pChannel[0])[iDest_Pos] += (short)(((char*)oSource.m_pChannel[0])[iSource_Pos] * fWeight);
        ((short*)oDest.m_pChannel[1])[iDest_Pos] += (short)(((char*)oSource.m_pChannel[1])[iSource_Pos] * fWeight);
        ((short*)oDest.m_pChannel[2])[iDest_Pos] += (short)(((char*)oSource.m_pChannel[2])[iSource_Pos] * fWeight);
    }
    else
    {
        ((short*)oDest.m_pChannel[0])[iDest_Pos] += (short)(((unsigned char*)oSource.m_pChannel[0])[iSource_Pos] * fWeight);
        ((short*)oDest.m_pChannel[1])[iDest_Pos] += (short)(((unsigned char*)oSource.m_pChannel[1])[iSource_Pos] * fWeight);
        ((short*)oDest.m_pChannel[2])[iDest_Pos] += (short)(((unsigned char*)oSource.m_pChannel[2])[iSource_Pos] * fWeight);
    }
    ((float*)oDest_Weight.m_pChannel[0])[iDest_Pos] += fWeight;

    return;
}
void Feed_Blender_1_GPU(Image oSource, Image::Part_1 oSource_Weight,
    Image::Part_1 oDest, Image::Part_1 oDest_Weight, short iLeft, short iTop, char bSigned = 1)
{
    static int iCount = 0;
    dim3 oGrid, oThread;
    int iSize = oSource_Weight.m_iWidth * oSource_Weight.m_iHeight;
    oThread.x = Min(512, iSize);
    oGrid.x = (iSize + oThread.x - 1) / oThread.x;
    _Feed_Blender_1_GPU << <oGrid.x, oThread.x >> > (oSource, oSource_Weight, oDest, oDest_Weight, iLeft, iTop, bSigned);
    //Disp_Cuda_Error();
    //if (iCount == 5)
    //{
    //    //Disp_Part_GPU((unsigned char *)oSource.m_pChannel[0], oSource.m_iWidth, 18, 0, 1, 1);
    //    //Disp_Part_GPU((short*)oDest.m_pChannel[0], oDest.m_iWidth, 59, 5, 1, 1);
    //    //Temp_Compare<short>("c:\\tmp\\1.bin", (short*)oDest.m_pChannel[0], oDest.m_iWidth, oDest.m_iHeight); 
    //    //Disp_Part_GPU((float*)oSource_Weight.m_pChannel[0], oSource_Weight.m_iWidth, 59, 5, 1, 1);
    //    //Disp_Part_GPU((float*)oDest_Weight.m_pChannel[0], oDest_Weight.m_iWidth, 882, 15, 1, 1);
    //    //Temp_Compare<float>("c:\\tmp\\0_float.bin", (float*)oDest_Weight.m_pChannel[0], oDest_Weight.m_iWidth, oDest_Weight.m_iHeight);
    //}
    //Disp_Cuda_Error();
    iCount++;
    return;
}

void _Feed_Blender(Image* Src_Pyr_Laplace[6], Image::Part_1* pPyr_Weight[6],
    Image::Part_1 dst_pyr_laplace[6], Image::Part_1 dst_band_weights[6],
    Image* Src_Pyr_Laplace_Header_GPU[6], Image::Part_1* Pyr_Weight_Header_GPU[6],
    Image::Part_1 dst_pyr_laplace_Header_GPU[6], Image::Part_1 dst_band_weights_Header_GPU[6],
    short (*pLTRB)[2][2], int dst_roi[2][2], int iImage_Count)
{
    const int MAX_IMAGE_COUNT = 8;
    if (iImage_Count > MAX_IMAGE_COUNT)
    {
        printf("exceeds MAX_IMAGE_COUNT in _Feed_Blender\n");
        exit(0);
    }
    short LTBR[8][2][2], roi[8][2][2];  // , (*roi_Header_GPU)[2][2];

    for (int j = 0; j < iImage_Count; j++)
    {
        LTBR[j][0][0] = pLTRB[j][0][0] - dst_roi[0][0];
        LTBR[j][0][1] = pLTRB[j][0][1] - dst_roi[0][1];
        LTBR[j][1][0] = pLTRB[j][1][0] - dst_roi[0][0];
        LTBR[j][1][1] = pLTRB[j][1][1] - dst_roi[0][1];
    }

    //���ִ��
    for (int i = 0; i <= 5; i++)
    {
        for (int j = 0; j < iImage_Count; j++)
        {
            roi[j][0][0] = LTBR[j][0][0];
            roi[j][0][1] = LTBR[j][0][1];
            roi[j][1][0] = LTBR[j][1][0] - LTBR[j][0][0];
            roi[j][1][1] = LTBR[j][1][1] - LTBR[j][0][1];

            /* Disp_Cuda_Error();
             unsigned long long tStart = iGet_Tick_Count();
             for(int k=0;k<10000;k++)*/
            Feed_Blender_1_GPU(Src_Pyr_Laplace[i][j], pPyr_Weight[i][j],
                dst_pyr_laplace[i], dst_band_weights[i], roi[j][0][0], roi[j][0][1], i < 5);
            /* Disp_Cuda_Error();
             printf("%lld\n", iGet_Tick_Count() - tStart);
             exit(0);*/
             //Temp_Compare<short>("c:\\tmp\\1.bin", (short*)dst_pyr_laplace[0].m_pChannel[0], dst_pyr_laplace[0].m_iWidth, dst_pyr_laplace[0].m_iHeight);
            LTBR[j][0][0] >>= 1, LTBR[j][0][1] >>= 1, LTBR[j][1][0] >>= 1, LTBR[j][1][1] >>= 1;
        }
    }

    ////���������ڴ���д�ص����˴��Ǹ��ѵ㣬�������޷����ֽڽ���Դ�Բ���
    //for (int i = 0; i <= 5; i++)
    //{
    //    for (int j = 0; j < iImage_Count; j++)
    //    {
    //        roi[j][0][0] = LTBR[j][0][0];
    //        roi[j][0][1] = LTBR[j][0][1];
    //        roi[j][1][0] = LTBR[j][1][0] - LTBR[j][0][0];
    //        roi[j][1][1] = LTBR[j][1][1] - LTBR[j][0][1];
    //    }
    //}
    //roi_Header_GPU = (short(*)[2][2])pMalloc_GPU(iImage_Count * 2 * 2 * sizeof(short));
    //hipMemcpy(roi_Header_GPU, roi, iImage_Count * 2 * 2 * sizeof(short), hipMemcpyHostToDevice);

   /* char File[256];
    for (int i = 0; i < 6; i++)
    {
        printf("Level:%d\n", i);
        for (int j = 0; j < 3; j++)
        {
            sprintf(File, "c:\\tmp\\Level_%d_Channel_%d.bin", i, j);
            Temp_Compare<short>(File, (short*)dst_pyr_laplace[i].m_pChannel[j], dst_pyr_laplace[i].m_iWidth, dst_pyr_laplace[i].m_iHeight,0);
        }
        sprintf(File, "c:\\tmp\\Level_%d_Weight.bin", i);
        Temp_Compare<float>(File, (float*)dst_band_weights[i].m_pChannel[0], dst_band_weights[i].m_iWidth, dst_band_weights[i].m_iHeight, 0);
    }*/

    return;
}
template<typename _T>void Feed_Blender(Image Warp[], Image Warp_Header_GPU[], Stitch<_T>* poStitch, Blender* poBlender)
{
    int iMax_Size,                                  //iMax_Size��������Borderͼ�����
        iMax_Source_Width, iMax_Source_Height,      //������Դͼ��ĳ���
        iSize, iImage_Count = poStitch->m_iImage_Count;
    int (*pLTRB)[2][2], //����װLeft, Top, Right, Bottom 4��Margin
        (*pLTRB_GPU)[2][2], (*pSize)[2];
    short (*pLTRB_New)[2][2]; //tl_new, br_new

    //Image** Src_Pyr_Laplace;
    Image* Src_Pyr_Laplace[6], * Src_Pyr_Laplace_Header_GPU[6];
    Image* pImage_Buffer, * pImage_Buffer_GPU;

    //�ȷ����ڴ�
    Light_Ptr oPtr, oPtr_GPU;
    unsigned char* p;
    iSize = iImage_Count * 2 * sizeof(int) +                //Size
        iImage_Count * 2 * 2 * sizeof(int) +                //pLTRB
        iImage_Count * 2 * 2 * sizeof(short) +                //pLTRB_New
        iImage_Count * 6 * sizeof(Image) + 128 * 2;              //Src_Pyr_Laplace

    iSize = ALIGN_SIZE_128(iSize);
    Attach_Light_Ptr(oPtr, (unsigned char*)pMalloc(iSize), iSize, 0);

    iSize = iImage_Count * 2 * 2 * sizeof(int) +
        iImage_Count * 2 * sizeof(int) +
        iImage_Count * 2 * 2 * sizeof(int);
    Malloc(oPtr, iSize, p);
    pLTRB = (int(*)[2][2])p;
    pSize = (int(*)[2])(pLTRB + iImage_Count);
    pLTRB_New = (short(*)[2][2])(pSize + iImage_Count);

    iSize = iImage_Count * 6 * sizeof(Image);
    Malloc(oPtr, iSize, p);
    Src_Pyr_Laplace[0] = pImage_Buffer = (Image*)p;
    for (int i = 0; i < 6; i++)
        Src_Pyr_Laplace[i] = &pImage_Buffer[i * iImage_Count];

    Feed_Blender_Get_Pos(Warp, poStitch, poBlender, pLTRB, pSize, pLTRB_New);

    iMax_Size = iSize = iMax_Source_Width = iMax_Source_Height = 0;  //iMax_Size��������Borderͼ�����
    for (int i = 0; i < iImage_Count; i++)
    {
        int iSize_1 = pSize[i][0] * pSize[i][1];
        iSize += iSize_1 * 3;             //src_pyr_laplace
        if (iSize_1 > iMax_Size)
            iMax_Size = iSize_1;
        if (Warp[i].m_iWidth > iMax_Source_Width)
            iMax_Source_Width = Warp[i].m_iWidth;
        if (Warp[i].m_iHeight > iMax_Source_Height)
            iMax_Source_Height = Warp[i].m_iHeight;

        //iSize += pSize[i][0] * pSize[i][1] * sizeof(float); //weight_pyr_gaus
        int Size_1[2] = { (pSize[i][0] + 1) / 2,(pSize[i][1] + 1) / 2 };
        for (int j = 0; j < 6; j++)
        {
            iSize += Size_1[0] * Size_1[1] * 3;             //src_pyr_laplace
            //iSize += Size_1[0] * Size_1[1] * sizeof(float); //weight_pyr_gaus
            Size_1[0] = (Size_1[0] + 1) / 2;
            Size_1[1] = (Size_1[1] + 1) / 2;
        }
    }

    iSize += iImage_Count * 2 * 2 * sizeof(int) +   //pSize_GPU,
        iImage_Count * 6 * sizeof(Image);       //Src_Pyr_Laplace_Header_GPU
    Attach_Light_Ptr(oPtr_GPU, (unsigned char*)pMalloc_GPU(iSize), iSize, 0);
    iSize = 0;

    for (int i = 0; i < iImage_Count; i++)
        Init_Image_GPU(&Src_Pyr_Laplace[0][i], pSize[i][0], pSize[i][1], Image::IMAGE_TYPE_BMP, 24, &oPtr_GPU);

    for (int i = 1; i < 6; i++)
    {
        for (int j = 0; j < iImage_Count; j++)
        {
            Image::Part_1 oPrev = Src_Pyr_Laplace[i - 1][j].m_oPart_1;
            Init_Image_GPU(&Src_Pyr_Laplace[i][j], (oPrev.m_iWidth + 1) / 2,
                (oPrev.m_iHeight + 1) / 2, Image::IMAGE_TYPE_BMP, 24, &oPtr_GPU);
        }
        //printf("%d %d\n", Src_Pyr_Laplace[i][0].m_iWidth, Src_Pyr_Laplace[i][0].m_iHeight);
    }

    //��pPos_4���ݳ���GPU
    iSize = iImage_Count * 2 * 2 * sizeof(int) +    //Pos_4
        iImage_Count * 6 * sizeof(Image) +          //pImage_Buffer_GPU
        6 * sizeof(Image*);                         //Src_Pyr_Laplace_Header_GPU
    Malloc(oPtr_GPU, iSize, p);
    pLTRB_GPU = (int (*)[2][2])p;
    pImage_Buffer_GPU = (Image*)(pLTRB_GPU + iImage_Count);
    //Src_Pyr_Laplace_Header_GPU = (Image**)(pImage_Buffer_GPU + iImage_Count * 6);
    hipMemcpy(pLTRB_GPU, pLTRB, iSize, hipMemcpyHostToDevice);
    hipMemcpy(pImage_Buffer_GPU, pImage_Buffer, iImage_Count * 6 * sizeof(Image), hipMemcpyHostToDevice);
    for (int i = 0; i < 6; i++)
        Src_Pyr_Laplace_Header_GPU[i] = &pImage_Buffer_GPU[i * iImage_Count];

    ////�ɰ����棬�е���
    //dim3 oThread, oGrid;
    //oThread.x = 512;
    //oGrid.x = (iMax_Size + oThread.x - 1) / oThread.x;
    //oGrid.y = iImage_Count;
    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //for (int i = 0; i < 10000; i++)
    ////3600 ms ����
    //Copy_Make_Border<<<oGrid,oThread>>>(Warp_Header_GPU, pImage_Buffer_GPU, pLTRB_GPU);
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

   /* dim3 oThread, oGrid;
    oThread.x = (iMax_Source_Width + 3) / 4;
    oGrid.x = iMax_Source_Height;
    oGrid.y = iImage_Count;*/

    //bLoad_Image_GPU("c:\\tmp\\1.bmp", &Warp[3]);
    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int j=0;j<10000;j++)
    {
        //3100ms, �е���
        //Copy_Make_Border_1 << <oGrid, oThread >> > (Warp_Header_GPU, pImage_Buffer_GPU, pLTRB_GPU);

        //ע�⣬�������в���ҪҪGPUͷ 
        //2900ms������
        for (int i = 0; i < iImage_Count; i++)
            Copy_Make_Border_GPU(Warp[i], pImage_Buffer[i], pLTRB[i][0][0], pLTRB[i][0][1], pLTRB[i][1][0], pLTRB[i][1][1]);
    }
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);
    //bSave_Image_GPU("c:\\tmp\\3.bmp", &pImage_Buffer_GPU[3]);
    //Compare_Image("c:\\tmp\\2.bmp", "c:\\tmp\\3.bmp");

    //60-70 ms
    ////Set_Pyramid_1 << <1, 6 >> > (pImage_Buffer_GPU, Src_Pyr_Laplace_Header_GPU, iImage_Count);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\img_with_border_0.bmp", &Src_Pyr_Laplace[0][0]);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\img_with_border_1.bmp", &Src_Pyr_Laplace[0][1]);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\img_with_border_2.bmp", &Src_Pyr_Laplace[0][2]);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\img_with_border_3.bmp", &Src_Pyr_Laplace[0][3]);

    Create_Laplace_Pyramid(Src_Pyr_Laplace, Src_Pyr_Laplace_Header_GPU, iImage_Count);
    /*for (int i = 0; i < 6; i++)
    {
        char Dest_File[256];
        sprintf(Dest_File, "c:\\tmp\\Dest_Level_%d.bmp", i);
        Image oImage = Src_Pyr_Laplace[i][0];
        bSave_Image_GPU(Dest_File, oImage);

        char Source_File[256];
        sprintf(Source_File, "c:\\tmp\\Source_Level_%d.bmp", i);
        Compare_Image(Source_File, Dest_File);
    }*/

    //�ֵ�����Weight�ˣ���ӦMask
    //��һ���ļ�ͷװ��Weight����
    Image::Part_1* pPyr_Weight[6], * pPyr_Weight_Header_GPU[6];
    pPyr_Weight[0] = (Image::Part_1*)pMalloc(iImage_Count * 6 * sizeof(Image::m_oPart_1));
    pPyr_Weight_Header_GPU[0] = (Image::Part_1*)pMalloc_GPU(iImage_Count * 6 * sizeof(Image::m_oPart_1));

    for (int i = 1; i < 6; i++)
        pPyr_Weight[i] = pPyr_Weight[i - 1] + 4;
    iSize = 0;
    for (int i = 0; i < iImage_Count; i++)
    {
        pLTRB[i][0][0] -= 5;
        pSize[i][0] -= 5;
        Image::Part_1 oImage;
        oImage.m_iWidth = pSize[i][0];
        oImage.m_iHeight = pSize[i][1];
        oImage.m_pChannel[0] = (unsigned char*)((unsigned long long)(iSize * 4));
        iSize += oImage.m_iHeight * oImage.m_iWidth;
        pPyr_Weight[0][i] = oImage;
    }
    for (int i = 1; i < 6; i++)
    {
        for (int j = 0; j < iImage_Count; j++)
        {
            Image::Part_1 oUpper = pPyr_Weight[i - 1][j];
            Image::Part_1 oLower;
            oLower.m_iHeight = (oUpper.m_iHeight + 1) >> 1;
            oLower.m_iWidth = (oUpper.m_iWidth + 1) >> 1;
            oLower.m_pChannel[0] = (unsigned char*)((unsigned long long)(iSize * 4));
            pPyr_Weight[i][j] = oLower;
            iSize += oLower.m_iHeight * oLower.m_iWidth;
        }
        pPyr_Weight_Header_GPU[i] = pPyr_Weight_Header_GPU[i - 1] + iImage_Count;
    }

    float* pWeight = (float*)pMalloc_GPU(iSize * sizeof(float));
    //�൱���Ȳ��ڱ�
    hipMemset(pWeight, 0, iSize * sizeof(float));
    for (int i = 0; i < 6; i++)
        for (int j = 0; j < iImage_Count; j++)
            pPyr_Weight[i][j].m_pChannel[0] += (unsigned long long)pWeight;

    hipMemcpy(pPyr_Weight_Header_GPU[0], pPyr_Weight[0], iImage_Count * 6 * sizeof(Image::Part_1), hipMemcpyHostToDevice);

    //bSave_Comp_GPU("c:\\tmp\\1.bmp", Warp[0],3);
    /*Disp_Cuda_Error();
    unsigned long long tStart = iGet_Tick_Count();
    for(int k=0;k<10000;k++)*/
    //1800ms
    for (int i = 0; i < iImage_Count; i++)
        Copy_Make_Border_float_GPU(Warp[i], pPyr_Weight[0][i], pLTRB[i][0][0], pLTRB[i][0][1], pLTRB[i][1][0], pLTRB[i][1][1]);
    /*Disp_Cuda_Error();
    printf("%lld\n", iGet_Tick_Count() - tStart);*/

    /*Temp_Compare("c:\\tmp\\Dest_0.bin", (float*)pPyr_Weight[0][0].m_pChannel[0], pPyr_Weight[0][0].m_iWidth, pPyr_Weight[0][0].m_iHeight);
    Temp_Compare("c:\\tmp\\Dest_1.bin", (float*)pPyr_Weight[0][1].m_pChannel[0], pPyr_Weight[0][1].m_iWidth, pPyr_Weight[0][1].m_iHeight);
    Temp_Compare("c:\\tmp\\Dest_2.bin", (float*)pPyr_Weight[0][2].m_pChannel[0], pPyr_Weight[0][2].m_iWidth, pPyr_Weight[0][2].m_iHeight);
    Temp_Compare("c:\\tmp\\Dest_3.bin", (float*)pPyr_Weight[0][3].m_pChannel[0], pPyr_Weight[0][3].m_iWidth, pPyr_Weight[0][3].m_iHeight);*/
    //Disp_Part_GPU((float*)pPyr_Weight[0][0].m_pChannel[0], pPyr_Weight[0][0].m_iWidth, 
        //342*2, 12, 10, 1);

    //һ��������������ͼ�Ľ�������
    Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    {
        //Pyr_Down_1_float(pPyr_Weight, pPyr_Weight_Header_GPU,iImage_Count);
        //ĿǰΪ 4100ms
        Pyr_Down_Batch_float(pPyr_Weight, pPyr_Weight_Header_GPU, iImage_Count);
    }
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);


    //////������Weight �˴���Ҫ  poBlender->dst_roi
    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    //�˴�������12000ms
    _Feed_Blender(Src_Pyr_Laplace, pPyr_Weight, poBlender->dst_pyr_laplace, poBlender->dst_band_weights,
        Src_Pyr_Laplace_Header_GPU, pPyr_Weight_Header_GPU,
        poBlender->dst_pyr_laplace_Header_GPU, poBlender->dst_band_weights_Header_GPU,
        pLTRB_New, poBlender->dst_roi, iImage_Count);

    ////����������������ͼ�Ľ�������һ��ԭͼ�Ĳһ����Ȩ��
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);
    if (oPtr.m_pBuffer)
        Free(oPtr.m_pBuffer);
    if (pWeight)
        Free_GPU(pWeight);
    if (pPyr_Weight[0])
        Free(pPyr_Weight[0]);
    if (pPyr_Weight_Header_GPU[0])
        Free_GPU(pPyr_Weight_Header_GPU[0]);
    if (Src_Pyr_Laplace[0][0].m_pChannel[0])
        Free_GPU(Src_Pyr_Laplace[0][0].m_pChannel[0]);

    return;
}

__global__ void _Normalize_Using_Weight_Map(Image::Part_1 oSource, Image::Part_1 oWeight)
{
    int iThread_ID = GET_THREAD_ID();
    if (iThread_ID >= oSource.m_iHeight * oSource.m_iWidth)
        return;
    const float eps = 1e-5f;
    //row[x].x = static_cast<short>(row[x].x / (weight_row[x] + WEIGHT_EPS));
    float fWeight = 1.f / (((float*)oWeight.m_pChannel[0])[iThread_ID] + eps);
    short Value_3[] = { ((short*)oSource.m_pChannel[0])[iThread_ID],
        ((short*)oSource.m_pChannel[1])[iThread_ID],
        ((short*)oSource.m_pChannel[2])[iThread_ID] };

    ((short*)oSource.m_pChannel[0])[iThread_ID] = Value_3[0] * fWeight;
    ((short*)oSource.m_pChannel[1])[iThread_ID] = Value_3[1] * fWeight;
    ((short*)oSource.m_pChannel[2])[iThread_ID] = Value_3[2] * fWeight;

    return;
}

void Normalize_Using_Weight_Map(Image::Part_1 oSource, Image::Part_1 oWeight)
{
    int iSize = oSource.m_iHeight * oSource.m_iWidth;
    dim3 oThread, oGrid;
    oThread.x = Min(iSize, 512);
    oGrid.x = (iSize + oThread.x - 1) / oThread.x;
    _Normalize_Using_Weight_Map << <oGrid, oThread >> > (oSource, oWeight);

    //������ֿ�3Ƭ�����ˣ���һ��
    return;
}
__global__ void _Py_Up_row_short(Image::Part_1 oSource, Image::Part_1 oMid, int iChannel_Count = 3)
{
    int iThread_ID = GET_THREAD_ID();
    if (iThread_ID >= oMid.m_iHeight * oMid.m_iWidth)
        return;
    short x = iThread_ID % oSource.m_iWidth,
        y = iThread_ID / oSource.m_iWidth;

    unsigned char bHas_Remain_x = oMid.m_iWidth > (oSource.m_iWidth << 1) && (x == oSource.m_iWidth - 1) ? 1 : 0;
    unsigned char bIs_Source_Border = (x == oSource.m_iWidth - 1);
    unsigned char bEven = bIs_Source_Border && ((x << 1) + 1 < oMid.m_iWidth);

    int iPos_m = y * oMid.m_iWidth,
        iPos_s = y * oSource.m_iWidth;
    int iSize_m = oMid.m_iWidth * oMid.m_iHeight;

    short* pMid = &((short*)oMid.m_pChannel[0])[iPos_m + (x << 1)];
    for (short i = 0; i < iChannel_Count; i++, pMid += iSize_m)
    {
        short Mid[3];
        //�м��
        short iPix = ((short*)oSource.m_pChannel[i])[iPos_s + x];
        if (x == 0)
        {//�������
            Mid[0] = iPix * 6 + (((short*)oSource.m_pChannel[i])[iPos_s + 1] << 1);
            Mid[1] = (iPix + ((short*)oSource.m_pChannel[i])[iPos_s + 1]) << 2;
        }
        else if (bIs_Source_Border)
        {//�ұ����㣬2���������
            Mid[0] = ((short*)oSource.m_pChannel[i])[iPos_s + x - 1] + iPix * 7;
            if (bEven)  //Դ�����һ����Զ�����ӦĿ��Ҳ������
                Mid[1] = iPix << 3;
        }
        else
        {//һ�����
            Mid[0] = ((short*)oSource.m_pChannel[i])[iPos_s + x - 1] + iPix * 6 + ((short*)oSource.m_pChannel[i])[iPos_s + x + 1];
            Mid[1] = (iPix + ((short*)oSource.m_pChannel[i])[iPos_s + x + 1]) << 2;
        }

        pMid[0] = Mid[0];
        if (!bIs_Source_Border || bEven)
        {
            pMid[1] = Mid[1];
            if (bHas_Remain_x)  //���������
                pMid[2] = Mid[1];
        }
    }
}
__global__ void _Pyr_Up_col_Add_Crop(Image::Part_1 oMid, Image::Part_1 oDest,
    Image::Part_1 oCrop, int iChannel_Count = 3)
{//��Crop������
    int iThread_ID = GET_THREAD_ID();
    int iMid_Size = oMid.m_iHeight * oMid.m_iWidth;

    short x = iThread_ID % oDest.m_iWidth,
        y = iThread_ID / oDest.m_iWidth;
    if (iThread_ID >= iMid_Size || x >= oCrop.m_iWidth)
        return;

    unsigned char bHas_Remain_y = oDest.m_iHeight > oMid.m_iHeight * 2 && y == oMid.m_iHeight - 1 ? 1 : 0;
    unsigned char bEven = (y * 2 + 1 < oDest.m_iHeight);

    //�ظ������ȫ����ȡ�������첻��
    short* r0 = &((short*)oMid.m_pChannel[0])[iGet_Border_y_GPU(y - 1, oMid.m_iHeight, BORDER_REFLECT101) * oDest.m_iWidth + x],
        * r1 = &((short*)oMid.m_pChannel[0])[iGet_Border_y_GPU(y, oMid.m_iHeight, BORDER_REFLECT) * oDest.m_iWidth + x],
        * r2 = &((short*)oMid.m_pChannel[0])[iGet_Border_y_GPU(y + 1, oMid.m_iHeight, BORDER_REFLECT) * oDest.m_iWidth + x];
    short y1 = y * 2;

    if (y1 >= oCrop.m_iHeight)
        return;

    int iPos_d = y1 * oDest.m_iWidth + x;
    for (int i = 0; i < iChannel_Count; i++, r0 += iMid_Size, r1 += iMid_Size, r2 += iMid_Size)
    {
        int iValue = ((short*)oDest.m_pChannel[i])[iPos_d] + ((*r0 + *r1 * 6 + *r2 + 32) >> 6);
        //if (y * 2 == 0 && x == 35)
        //    printf("%d %d %d\n", ((short*)oDest.m_pChannel[i])[iPos_d], ((*r0 + *r1 * 6 + *r2 + 32) >> 6),iValue);

        //((short*)oDest.m_pChannel[i])[iPos_d] = iValue;
        //((short*)oCrop.m_pChannel[i])[y1 * oCrop.m_iWidth + x] = iValue;
        int iPos_Crop = y1 * oCrop.m_iWidth + x;
        oCrop.m_pChannel[i][iPos_Crop] = oCrop.m_pChannel[3][iPos_Crop] ? iValue : 0;
        //if (y1 == 16 && x == 0)
            //printf("%d %d %d %d\n", iPos_d, ((short*)oDest.m_pChannel[i])[iPos_d], ((*r0 + *r1 * 6 + *r2 + 32) >> 6), iValue);
        if (y1 + 1 >= oCrop.m_iHeight)
            continue;
        if (bEven)
        {
            iValue = ((short*)oDest.m_pChannel[i])[iPos_d + oDest.m_iWidth] + ((((*r1 + *r2) << 2) + 32) >> 6);
            //((short*)oDest.m_pChannel[i])[iPos_d + oDest.m_iWidth] = iValue;    // Clip3(-128, 127, iValue);
            //((short*)oCrop.m_pChannel[i])[(y1 +1) * oCrop.m_iWidth + x] = iValue;
            iPos_Crop = (y1 + 1) * oCrop.m_iWidth + x;
            oCrop.m_pChannel[i][iPos_Crop] = oCrop.m_pChannel[3][iPos_Crop] ? iValue : 0;
            if (y1 + 2 >= oCrop.m_iHeight)
                continue;
            if (bHas_Remain_y)
            {
                iValue = ((short*)oDest.m_pChannel[i])[iPos_d + (oDest.m_iWidth << 1)] + (((short*)oDest.m_pChannel[i])[iPos_d]);
                //((short*)oDest.m_pChannel[i])[iPos_d + (oDest.m_iWidth << 1)] = iValue; // Clip3(-128, 127, iValue);
                //((short*)oCrop.m_pChannel[i])[(y1 + 2) * oCrop.m_iWidth + x] = iValue;
                iPos_Crop = (y1 + 2) * oCrop.m_iWidth + x;
                oCrop.m_pChannel[i][iPos_Crop] = oCrop.m_pChannel[3][iPos_Crop] ? iValue : 0;
            }
        }
    }
    return;
}
void Pyr_Up_Add_Crop_short(Image::Part_1 oSource, Image::Part_1 oDest, short* pMid, Image::Part_1 oCrop)
{//���һ����Crop������
    Image::Part_1 oMid;
    oMid.m_iWidth = oDest.m_iWidth;
    oMid.m_iHeight = oSource.m_iHeight;
    int iSize = oSource.m_iHeight * oSource.m_iWidth;
    oMid.m_pChannel[0] = (unsigned char*)pMid;
    oMid.m_pChannel[1] = oMid.m_pChannel[0] + iSize * 2;
    oMid.m_pChannel[2] = oMid.m_pChannel[1] + iSize * 2;

    dim3 oThread, oGrid;
    //��ʼ�����н����ϲ���
    oThread.x = Min(iSize, 256);
    oGrid.x = (iSize + oThread.x - 1) / oThread.x;
    _Py_Up_row_short << <oGrid, oThread >> > (oSource, oMid);

    iSize = oMid.m_iHeight * oMid.m_iWidth;
    oThread.x = Min(iSize, 512);
    oGrid.x = (iSize + oThread.x - 1) / oThread.x;
    _Pyr_Up_col_Add_Crop << <oGrid, oThread >> > (oMid, oDest, oCrop);

    return;
}

void Pyr_Up_Add_short(Image::Part_1 oSource, Image::Part_1 oDest, short* pMid)
{//���ģ������3x3�����²�����һ��
    Image::Part_1 oMid;
    oMid.m_iWidth = oDest.m_iWidth;
    oMid.m_iHeight = oSource.m_iHeight;
    int iSize = oSource.m_iHeight * oSource.m_iWidth;
    oMid.m_pChannel[0] = (unsigned char*)pMid;
    oMid.m_pChannel[1] = oMid.m_pChannel[0] + iSize * 2;
    oMid.m_pChannel[2] = oMid.m_pChannel[1] + iSize * 2;

    //Temp_Compare("c:\\tmp\\Py_Up_5_0.bin", (short*)oSource.m_pChannel[0], oSource.m_iWidth, oSource.m_iHeight);
    //Temp_Compare("c:\\tmp\\Py_Up_4_0.bin", (short*)oDest.m_pChannel[0], oDest.m_iWidth, oDest.m_iHeight);

    //Disp_Part_GPU((short*)oSource.m_pChannel[0], oSource.m_iWidth, 17, 0, 3, 2, "Org Source");
    //Disp_Part_GPU((short*)oDest.m_pChannel[0], oDest.m_iWidth, 34, 0, 3, 2,"Org Dest");

    dim3 oThread, oGrid;
    //��ʼ�����н����ϲ���
    oThread.x = Min(iSize, 512);
    oGrid.x = (iSize + oThread.x - 1) / oThread.x;
    _Py_Up_row_short << <oGrid, oThread >> > (oSource, oMid);
    //Disp_Cuda_Error();
    //Disp_Part_GPU((short*)oMid.m_pChannel[0], oMid.m_iWidth, 6, 0, 1, 2);

    iSize = oMid.m_iHeight * oMid.m_iWidth;
    oThread.x = Min(iSize, 512);
    oGrid.x = (iSize + oThread.x - 1) / oThread.x;
    _Pyr_Up_col_Add << <oGrid, oThread >> > (oMid, oDest);


    //Disp_Cuda_Error();
    //Disp_Part_GPU((short*)oDest.m_pChannel[0], oDest.m_iWidth, 35, 0, 10, 1);
    //Temp_Compare("c:\\tmp\\Py_Up_4_0.bin", (short*)oDest.m_pChannel[0], oDest.m_iWidth, oDest.m_iHeight);

    return;
}

void Restore_Image_From_Laplace(Image::Part_1 Pyr[], Image::Part_1 oDst)
{//��һ���������ָ����Ѿ���ȫ��׼opencv

    //�ȷ���һ��Mid
    Image::Part_1 oUpper = Pyr[0],
        oLower = Pyr[1];    // , oMid;

    short* pMid = (short*)pMalloc_GPU(oLower.m_iHeight * oUpper.m_iWidth * 3 * sizeof(short));

    ////��ʱװ���ļ�,װ�����ݾ���ȫ������
    //for (int i = 0; i < 6; i++)
    //{
    //    char File[256];
    //    int iSize = Pyr[i].m_iHeight * Pyr[i].m_iWidth * sizeof(short);
    //    for (int j = 0; j < 3; j++)
    //    {
    //        sprintf(File, "c:\\tmp\\normalizeUsingWeightMap_%d_%d.bin", i, j);
    //        bLoad_Raw_Data_GPU(File, &Pyr[i].m_pChannel[j], &iSize);
    //    }
    //}

    for (int i = 5; i > 1; i--)
    {
        //�Ȱ��ź�oMid
        oUpper = Pyr[i - 1];
        oLower = Pyr[i];
        //�ϲ���
        Pyr_Up_Add_short(oLower, oUpper, pMid);
    }
    //Disp_Part_GPU(((short*)Pyr[0].m_pChannel[0]), Pyr[0].m_iWidth, 0, 16, 1, 1);

    oUpper = Pyr[0];
    oLower = Pyr[1];
    Pyr_Up_Add_Crop_short(oLower, oUpper, pMid, oDst);


    //bSave_Image_GPU("c:\\tmp\\1.bmp", oDst);
    //Disp_Cuda_Error();
    //Disp_Part_GPU(((short*)oDst.m_pChannel[0]), oDst.m_iWidth, 0, 16, 1, 1);
    //Temp_Compare<short>("c:\\tmp\\dst_0.bin", (short*)oDst.m_pChannel[0], oDst.m_iWidth, oDst.m_iHeight);
    //Temp_Compare<short>("c:\\tmp\\dst_1.bin", (short*)oDst.m_pChannel[1], oDst.m_iWidth, oDst.m_iHeight);
    //Temp_Compare<short>("c:\\tmp\\dst_2.bin", (short*)oDst.m_pChannel[2], oDst.m_iWidth, oDst.m_iHeight);

    ////����
    //for (int i = 0; i < 6; i++)
    //{
    //    char File[256];
    //    for (int j = 0; j < 3; j++)
    //    {
    //        sprintf(File, "c:\\tmp\\Py_Up_%d_%d.bin", i, j);
    //        Temp_Compare(File, (short*)Pyr[i].m_pChannel[j], Pyr[i].m_iWidth, Pyr[i].m_iHeight);
    //    }
    //}
    if (pMid)
        Free_GPU(pMid);
    return;
}

__global__ void _Weight_Compare_Crop(Image::Part_1 oWeight, float eps, Image::Part_1 oMask)
{
    int iThread_ID = GET_THREAD_ID();
    short y = iThread_ID / oMask.m_iWidth,
        x = iThread_ID % oMask.m_iWidth;

    oMask.m_pChannel[0][iThread_ID] = ((float*)oWeight.m_pChannel[0])[y * oWeight.m_iWidth + x] > eps ? 255 : 0;
    return;
}
void Weight_Compare_Crop(Image::Part_1 oWeight, float eps, Image oMask)
{
    int iSize = oMask.m_iWidth * oMask.m_iHeight;
    dim3 oThread, oGrid;
    oThread.x = Min(512, iSize);
    oGrid.x = (iSize + oThread.x - 1) / oThread.x;
    _Weight_Compare_Crop << <oGrid, oThread >> > (oWeight, eps, oMask.m_oPart_1);
    /*Disp_Cuda_Error();
    bSave_Image_GPU("c:\\tmp\\2.bmp", oMask);
    Compare_Image("c:\\tmp\\mask.bmp", "c:\\tmp\\2.bmp");*/

    return;
}

//__global__ void _Blend_GPU(Image::Part_1 oDst, Image::Part_1 oMask, Image::Part_1 oResult)
//{
//    int iThread_ID = GET_THREAD_ID();
//    if (iThread_ID > oDst.m_iHeight * oDst.m_iWidth)
//        return;
//
//    int iMask = oMask.m_pChannel[0][iThread_ID] & 1;
//    //short Value[3] = {};
//    //if (iMask)
//    //{
//    //    Value[0] = ((short*)oDst.m_pChannel[0])[iThread_ID];
//    //    Value[1] = ((short*)oDst.m_pChannel[1])[iThread_ID];
//    //    Value[2] = ((short*)oDst.m_pChannel[2])[iThread_ID];
//    //    /*if (Value[0] < 0 || Value[0]>255 ||
//    //        Value[1] < 0 || Value[1]>255 ||
//    //        Value[2] < 0 || Value[2]>255)
//    //    {
//    //        printf("*");
//    //    }*/
//    //}
//    oResult.m_pChannel[0][iThread_ID] = ((short*)oDst.m_pChannel[0])[iThread_ID] *iMask;
//    oResult.m_pChannel[1][iThread_ID] = ((short*)oDst.m_pChannel[1])[iThread_ID] *iMask;
//    oResult.m_pChannel[2][iThread_ID] = ((short*)oDst.m_pChannel[2])[iThread_ID] *iMask;
//}
//void Blend(Image::Part_1 oDst, Image oMask, Image oResult)
//{//���Ļ��
//    //1���ҳ�Mask�� =0�ĵ�
//    // 2����dst�ж�ӦMask�е�0�㣬��Ϊ0
//    //3������ҪClipһ��     
//    int iSize = oDst.m_iHeight * oDst.m_iWidth;
//    dim3 oThread, oGrid;
//    oThread.x = Min(iSize, 512);
//    oGrid.x = (iSize + oThread.x - 1) / oThread.x;
//    _Blend_GPU << <oGrid, oThread >> > (oDst, oMask.m_oPart_1, oResult.m_oPart_1);
//    //Disp_Cuda_Error();
//    //bSave_Image_GPU("c:\\tmp\\dst.bmp", oResult);
//    return;
//}

template<typename _T>void Blend(Stitch<_T>* poStitch, Blender* poBlender)
{
    //�ȹ��
    //Disp_Cuda_Error();

    for (int i = 0; i < 6; i++)
    {
        //unsigned long long tStart = iGet_Tick_Count();
        //for(int j=0;j<10000;j++)
        Normalize_Using_Weight_Map(poBlender->dst_pyr_laplace[i], poBlender->dst_band_weights[i]);
        //Disp_Cuda_Error();
        //printf("%lld\n", iGet_Tick_Count() - tStart);

        ////�����ݣ��˴���Щ������������ģ��Ǹ������������
        //char File[256];
        //Image::Part_1 oImage = poBlender->dst_pyr_laplace[i];
        /*if(i==5)
        for (int j = 0; j < 3; j++)
        {
            sprintf(File, "c:\\tmp\\normalizeUsingWeightMap_%d_%d.bin", i, j);
            Temp_Compare<short>(File, (short*)oImage.m_pChannel[j], oImage.m_iWidth, oImage.m_iHeight);
        }*/
    }

    /*Image::Part_1 oDst;
    oDst.m_iHeight = poBlender->dst_roi_final[1][1];
    oDst.m_iWidth = poBlender->dst_roi_final[1][0];
    int iSize = oDst.m_iHeight * oDst.m_iWidth;
    oDst.m_pChannel[0] = (unsigned char*)pMalloc_GPU(iSize * 3 *sizeof(short));
    oDst.m_pChannel[1] = oDst.m_pChannel[0] + iSize * sizeof(short);
    oDst.m_pChannel[2] = oDst.m_pChannel[1] + iSize * sizeof(short);*/

    Image oDst;
    Init_Image_GPU(&oDst, poBlender->dst_roi_final[1][0], poBlender->dst_roi_final[1][1], Image::IMAGE_TYPE_BMP, 32);
    poBlender->m_oResult = oDst;

    //short Dst_rc[4] = { 0,0, (short)poBlender->dst_roi_final[1][0],(short)poBlender->dst_roi_final[1][1] };
    //Init_Image_GPU(&poBlender->m_oMask, poBlender->dst_roi_final[1][0], poBlender->dst_roi_final[1][1], Image::IMAGE_TYPE_BMP, 8);
    Attach_Buffer(&poBlender->m_oMask, oDst.m_pChannel[3], poBlender->dst_roi_final[1][0], poBlender->dst_roi_final[1][1],
        1, Image::IMAGE_TYPE_BMP);

    Weight_Compare_Crop(poBlender->dst_band_weights[0], 1e-5f, poBlender->m_oMask);
    //bSave_Image_GPU("c:\\tmp\\1.bmp", poBlender->m_oMask);

    //�������ϲ���
    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    Restore_Image_From_Laplace(poBlender->dst_pyr_laplace, oDst.m_oPart_1);
    if (poBlender->m_pBuffer)
    {
        Free_GPU(poBlender->m_pBuffer);
        poBlender->m_pBuffer = NULL;
    }

    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    //bSave_Image_GPU("c:\\tmp\\1.bmp", oDst);
    ////���ĺϳ�
    //Image oResult;
    //Init_Image_GPU(&oResult, oDst.m_iWidth, oDst.m_iHeight, Image::IMAGE_TYPE_BMP, 24);
    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    //Blend(oDst, poBlender->m_oMask, oResult);
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    return;
}
template<typename _T>void Compose_Panorama(Stitch<_T>* poStitch,Image *poResult)
{
    int i;
    Stitch<_T> oStitch = *poStitch;
    Point_Cloud<float> oPC;
    Init_Point_Cloud(&oPC, 100000, 1);

    //������С��ģͶӰ
    for (i = 0; i < oStitch.m_iImage_Count; i++)
    {
        _T K[3 * 3];
        Camera<_T> oCamera = oStitch.m_pCamera[i];

        //�˴�ʵ�����ǽ�K��λ����ͶӰ����һ��scale�������һ��ͼ
        //�����Ǽ���zͶӰ����?
        Set_K(&oStitch, oCamera.K, K, poStitch->seam_work_aspect);
        _T fScale = oStitch.warped_image_scale * oStitch.seam_work_aspect;

        //unsigned long long tStart = iGet_Tick_Count();
        //for(int j=0;j<10000;j++)
        {//�϶�Ϊһ
            Warp_2<_T>(oStitch.m_pSeam_Est[i], oStitch.m_pMask[i], K, oCamera.R, fScale,
                &oStitch.m_pImage_Warp[i], &oStitch.m_pMasks_Warped[i],
                INTER_LINEAR, INTER_NEAREST, oStitch.m_pCorner[i]);
            //bSave_Image_GPU("c:\\tmp\\2.bmp", oStitch.m_pMasks_Warped[i]);
            //Free_Image_GPU(&oStitch.m_pImage_Warp[i]);
            //Free_Image_GPU(&oStitch.m_pMasks_Warped[i]);
        }
        //Disp_Cuda_Error();
        //printf("%lld\n", iGet_Tick_Count() - tStart);
    }
    //ͶӰ�����Ժ�Ҫ�����ݳ���GPU��
    hipMemcpy(oStitch.m_pCorner_GPU, oStitch.m_pCorner, oStitch.m_iImage_Count * 2 * 2 * sizeof(int), hipMemcpyHostToDevice);

    //Disp_Cuda_Error();
    //���Դ��룬װ������
    /*bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\images_warped_0.bmp", &poStitch->m_pImage_Warp[0]);
    bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\images_warped_1.bmp", &poStitch->m_pImage_Warp[1]);
    bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\images_warped_2.bmp", &poStitch->m_pImage_Warp[2]);
    bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\images_warped_3.bmp", &poStitch->m_pImage_Warp[3]);*/

    //��ǰ���ͶӰͼ��ȫ���ó����ֿ飬ÿһ��������󽻼�
    //���ݽ����������ֵ��һ��һֵ���Ը�ͼ���⣿
    Feed<_T>(&oStitch);

    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\image_warp_f_0.bmp", &poStitch->m_pImage_Warp[0]);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\image_warp_f_1.bmp", &poStitch->m_pImage_Warp[1]);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\image_warp_f_2.bmp", &poStitch->m_pImage_Warp[2]);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\image_warp_f_3.bmp", &poStitch->m_pImage_Warp[3]);
    //�Ը���ͶӰͼ�����󽻣�
    //bSave_Image_GPU("c:\\tmp\\1.bmp", oStitch.m_pImage_Warp[0]);
    //�˴���Ӧ�����������ʱδ������������
    Find<_T>(&oStitch);

    //���ˣ�����opencv�ù۵㣬�����ͷŲ����ڴ���
    Free_Partial(&oStitch);

    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\0.bmp", &oStitch.m_pImage_Source[0]);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\1.bmp", &oStitch.m_pImage_Source[1]);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\2.bmp", &oStitch.m_pImage_Source[2]);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\3.bmp", &oStitch.m_pImage_Source[3]);
    //bSave_Image_GPU("c:\\tmp\\1.bmp", oStitch.m_pImage_Source[0]);
    //int bIs_Compose_Scale_Set = 0;
    //Camera<_T> Cameras_Scaled[2];
    _T compose_work_aspect = oStitch.compose_scale / oStitch.work_scale;
    _T warp_scale = oStitch.warped_image_scale * compose_work_aspect;
    Image* pBlend_Warp = (Image*)pMalloc(poStitch->m_iImage_Count * sizeof(Image));
    Image* pBlend_Warp_Header_GPU = (Image*)pMalloc_GPU(poStitch->m_iImage_Count * sizeof(Image));

    for (int i = 0; i < oStitch.m_iImage_Count; i++)
    {
        _T K[3 * 3];
        Camera<_T> oCamera = oStitch.m_pCamera[i];

        //�˴�ʵ�����ǽ�K��λ����ͶӰ����һ��scale�������һ��ͼ
        //�����Ǽ���zͶӰ����?
        Set_K(&oStitch, oCamera.K, K, compose_work_aspect);
        //Disp(K, 3, 3, "K");


        //����ԭ��poStitch->m_pWarp
        Image oImg_Warp;
        //Disp_Cuda_Error();
        //unsigned long long tStart = iGet_Tick_Count();
        //for(int j=0;j<10000;j++)
        {
            Warp_3<_T>(poStitch->m_pImage_Source[i], K, oCamera.R, warp_scale,
                &oImg_Warp, INTER_LINEAR, INTER_NEAREST, poStitch->m_pCorner[i],
                BORDER_REFLECT, BORDER_CONSTANT, NULL);
            //Free_Image_GPU(&oImg_Warp);
        }
        //Disp_Cuda_Error();
        //printf("%lld\n", iGet_Tick_Count() - tStart);
        pBlend_Warp[i] = oImg_Warp;
        //�ֵ�����������
    }
    hipMemcpy(pBlend_Warp_Header_GPU, pBlend_Warp, poStitch->m_iImage_Count * sizeof(Image), hipMemcpyHostToDevice);

    ////װ��Warpͼ�񣬲ⲹ��
    //bLoad_Image_GPU("C:\\tmp\\Warp_0.bmp", &pBlend_Warp[0]);
    //bLoad_Image_GPU("C:\\tmp\\Warp_1.bmp", &pBlend_Warp[1]);
    //bLoad_Image_GPU("C:\\tmp\\Warp_2.bmp", &pBlend_Warp[2]);
    //bLoad_Image_GPU("C:\\tmp\\Warp_3.bmp", &pBlend_Warp[3]);
    //����
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    Block_Compensate(&oStitch, pBlend_Warp, pBlend_Warp_Header_GPU);
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    //bSave_Image_GPU("c:\\tmp\\0.bmp", &pBlend_Warp_Header_GPU[0]);
    //Compare_Image("c:\\tmp\\Warp_Comp_0.bmp", "c:\\tmp\\0.bmp",1);

    ////װ��Warpͼ��
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\Warp_0.bmp", &poStitch->m_pImage_Warp[0]);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\Warp_1.bmp", &poStitch->m_pImage_Warp[1]);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\Warp_2.bmp", &poStitch->m_pImage_Warp[2]);
    //bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\Warp_3.bmp", &poStitch->m_pImage_Warp[3]);
    //hipMemcpy(poStitch->m_pImage_Warp[0].m_pChannel[3], poStitch->m_pImage_Warp[0].m_pChannel[0], 
    //    poStitch->m_pImage_Warp[0].m_iWidth * poStitch->m_pImage_Warp[0].m_iHeight, hipMemcpyDeviceToDevice);
    //hipMemcpy(poStitch->m_pImage_Warp[1].m_pChannel[3], poStitch->m_pImage_Warp[1].m_pChannel[0],
    //    poStitch->m_pImage_Warp[1].m_iWidth * poStitch->m_pImage_Warp[1].m_iHeight, hipMemcpyDeviceToDevice);
    //hipMemcpy(poStitch->m_pImage_Warp[2].m_pChannel[3], poStitch->m_pImage_Warp[2].m_pChannel[0],
    //    poStitch->m_pImage_Warp[2].m_iWidth * poStitch->m_pImage_Warp[2].m_iHeight, hipMemcpyDeviceToDevice);
    //hipMemcpy(poStitch->m_pImage_Warp[3].m_pChannel[3], poStitch->m_pImage_Warp[3].m_pChannel[0],
    //    poStitch->m_pImage_Warp[3].m_iWidth * poStitch->m_pImage_Warp[3].m_iHeight, hipMemcpyDeviceToDevice);

    Dilate_GPU(poStitch->m_pImage_Warp, poStitch->m_pImage_Warp_Header_GPU, poStitch->m_iImage_Count);

    /*Draw_Point<float>(&oPC, 0, 0, 0);
    bSave_PLY("c:\\tmp\\1.ply", oPC);*/

    //���Ž�Image_Warp�е�Maskͨ�����ŵ�Warp��ô������Warp��Mask���� Bitwise And
    //�˴�������3500ms
    Resize_Bitwise_And(poStitch, pBlend_Warp, pBlend_Warp_Header_GPU);
    Blender oBlender = {};

    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    {
        //�˴�Ҫ�޸ģ���������ڴ棬���˺���զ�ã������ٷ���ۺ϶ȸ���
        Prepare_Blender_1(&oStitch, &oBlender);
    }
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    ////�������ݣ�װ��Mask_Warp
    //bLoad_Comp_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\Mask_Warp_0.bmp", pBlend_Warp[0], 3);
    //bLoad_Comp_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\Mask_Warp_1.bmp", pBlend_Warp[1], 3);
    //bLoad_Comp_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\Mask_Warp_2.bmp", pBlend_Warp[2], 3);
    //bLoad_Comp_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\Mask_Warp_3.bmp", pBlend_Warp[3], 3);

    //bSave_Image_GPU("c:\\tmp\\1.bmp", pBlend_Warp[0]);
    Feed_Blender(pBlend_Warp, pBlend_Warp_Header_GPU, poStitch, &oBlender);

    //�����Blend
    Blend(poStitch, &oBlender);
    *poResult = oBlender.m_oResult;

    //����ͷ�
    if (pBlend_Warp)
    {
        for (int i = 0; i < oStitch.m_iImage_Count; i++)
            Free_Image_GPU(&pBlend_Warp[i]);
        Free(pBlend_Warp);
    }

    if (pBlend_Warp_Header_GPU)
        Free_GPU(pBlend_Warp_Header_GPU);
    if (oBlender.m_pBuffer)
        Free_GPU(oBlender.m_pBuffer);
    //Free_Image_GPU(&oBlender.m_oResult);

    *poStitch = oStitch;
    Free_Point_Cloud(&oPC);
    return;
}



void Pry_Down_Test()
{
    Image oSource = {}, oDest;
    //bLoad_Image_GPU("C:\\tmp\\1.bmp", &oSource);
    bLoad_Image_GPU("C:\\tmp\\temp\\stitch\\1706x1279\\1.bmp", &oSource);
    Init_Image_GPU(&oDest, (oSource.m_iWidth + 1) >> 1, (oSource.m_iHeight + 1) >> 1, Image::IMAGE_TYPE_BMP, oSource.m_iBit_Count);

    Disp_Cuda_Error();
    unsigned long long tStart = iGet_Tick_Count();
    for (int i = 0; i < 10000; i++)
        Pyr_Down_GPU(oSource, oDest);
    Disp_Cuda_Error();
    printf("%lld\n", iGet_Tick_Count() - tStart);

    bSave_Image_GPU("c:\\tmp\\3.bmp", oDest);
    Compare_Image("c:\\tmp\\2.bmp", "c:\\tmp\\3.bmp");

    Free_Image_GPU(&oSource);
    Free_Image_GPU(&oDest);
    return;
}

void Pyr_Up_Test()
{//�������ϲ���ʵ��
    Image oSource = {}, oDest;
    bLoad_Image_GPU("c:\\tmp\\1.bmp", &oSource);
    Init_Image_GPU(&oDest, oSource.m_iWidth * 2 - 1, oSource.m_iHeight * 2 - 1, Image::IMAGE_TYPE_BMP, oSource.m_iBit_Count);
    Disp_Cuda_Error();
    unsigned long long tStart = iGet_Tick_Count();
    for (int i = 0; i < 10000; i++)
        Pyr_Up_GPU(oSource, oDest);
    Disp_Cuda_Error();
    printf("%lld\n", iGet_Tick_Count() - tStart);

    Free_Image_GPU(&oSource);
    Free_Image_GPU(&oDest);
    return;
}
void Copy_Make_Border_Test()
{
    int iLeft = 5, iTop = 11, iRight = 125 + 2, iBottom = 85;
    Image oSource = {}, oDest;
    bLoad_Image_GPU("c:\\tmp\\1.bmp", &oSource);
    Init_Image_GPU(&oDest, oSource.m_iWidth + iLeft + iRight, oSource.m_iHeight + iTop + iBottom, Image::IMAGE_TYPE_BMP, oSource.m_iBit_Count);
    Copy_Make_Border_GPU(oSource, oDest, iLeft, iTop, iRight, iBottom);
    return;
}

static void Test_1()
{
    typedef double _T;
    Stitch<_T> oStitch = {};
    //���ڽ��Ӿ����룬�˴�װ���������
    Temp_Load_Camera("data\\camera.bin", &oStitch);

    //��ʼ��ƴ����
    Init_Stitch<double>(&oStitch, 1706, 1279);

    //װ��ԭͼ���Ժ�����˴�
    bLoad_Image(oStitch);
    Resize_Seam_Image(&oStitch);

    Disp_Cuda_Error();
    unsigned long long tStart = iGet_Tick_Count();
    Image oResult;
    Compose_Panorama(&oStitch, &oResult);
    Disp_Cuda_Error();
    Free_Stitch(&oStitch);
    printf("Overall %lld ms\n", iGet_Tick_Count() - tStart);

    bSave_Image_GPU("c:\\tmp\\1.bmp", oResult);
    Free_Image_GPU(&oResult);
}
int main()
{
    Init_Env_All();
    //Copy_Make_Border_Test();
    //Gauss_Test();
    Test_1();
    //Test_2();
    //Pry_Down_Test();
    //Pyr_Up_Test();
    Free_Env_All();
    return 0;
}

