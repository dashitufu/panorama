#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Common.h"
#include "Common_Cuda.cuh"
//#include "Image.h"
#include "image_cuda.h"
#include "Matrix.h"

void SB_Image_Cuda()
{

}
//__device__ static int iGet_Border_y_GPU(int y, int iHeight, Border_Type iBorder_Type)
//{
//    if (y < 0)
//    {
//        switch (iBorder_Type)
//        {
//        case Border_Type::BORDER_CONSTANT:
//            return -1;
//        case Border_Type::BORDER_REFLECT:
//            return -y - 1;
//        case Border_Type::BORDER_REFLECT_101:
//            return -y;
//        }
//    }
//    else if (y >= iHeight)
//    {
//        switch (iBorder_Type)
//        {
//        case Border_Type::BORDER_CONSTANT:
//            return -1;
//        case Border_Type::BORDER_REFLECT:
//            return iHeight - (y - iHeight + 1);
//        case Border_Type::BORDER_REFLECT_101:
//            return iHeight - (y - iHeight + 1);
//        }
//    }
//    return y;
//}
__device__ static int iGet_Border_y_GPU(int y, int iHeight, Border_Type iBorder_Type, int iThread_ID = 0)
{
    if (y < 0)
    {
        switch (iBorder_Type)
        {
        case Border_Type::BORDER_REFLECT:
            return -y - 1;
        case Border_Type::BORDER_REFLECT_101:
            return -y;
        case Border_Type::BORDER_CONSTANT:
        case Border_Type::BORDER_REPLICATE:
            return 0;
        }
    }
    else if (y >= iHeight)
    {
        /*if (iThread_ID == 77803)
            printf("Here");*/
        switch (iBorder_Type)
        {

        case Border_Type::BORDER_REFLECT:
            return iHeight - (y - iHeight + 1);
        case Border_Type::BORDER_REFLECT_101:
            return iHeight - (y - iHeight + 2);
        case Border_Type::BORDER_CONSTANT:
        case Border_Type::BORDER_REPLICATE:
            return iHeight - 1;
        }
    }
    return y;
}

//__device__ static int iGet_Border_x_GPU(int x, int iWidth, Border_Type iBorder_Type)
//{
//    if (x < 0)
//    {
//        switch (iBorder_Type)
//        {
//        case Border_Type::BORDER_REFLECT:
//            return -x - 1;
//        case Border_Type::BORDER_REFLECT_101:
//            return -x;
//        }
//    }
//    else if (x >= iWidth)
//    {
//        switch (iBorder_Type)
//        {
//        case Border_Type::BORDER_REFLECT:
//            return iWidth - (x - iWidth + 1);
//        case Border_Type::BORDER_REFLECT_101:
//            return iWidth - (x - iWidth + 2);
//        }
//    }
//    return x;
//}

__device__ static int iGet_Border_x_GPU(int x, int iWidth, Border_Type iBorder_Type)
{
    if (x < 0)
    {
        switch (iBorder_Type)
        {
        case Border_Type::BORDER_REFLECT:
            return -x - 1;
        case Border_Type::BORDER_REFLECT_101:
            return -x;
        case Border_Type::BORDER_CONSTANT:
        case Border_Type::BORDER_REPLICATE:
            return 0;
        }
    }
    else if (x >= iWidth)
    {
        switch (iBorder_Type)
        {
        case Border_Type::BORDER_REFLECT:
            return iWidth - (x - iWidth + 1);
        case Border_Type::BORDER_REFLECT_101:
            return iWidth - (x - iWidth + 2);
        case Border_Type::BORDER_CONSTANT:
        case Border_Type::BORDER_REPLICATE:
            return iWidth - 1;
        }
    }
    return x;
}
void Init_Image_GPU(Image* poImage, int iWidth, int iHeight, Image::Type iType, int iBit_Count,Light_Ptr *poPtr,int iGPU_ID)
{
    Light_Ptr oPtr;
	int i,iSize, iSize_With_Remain;
	if (!poImage || iWidth == 0 || iHeight == 0)
	{
		printf("Invalid parameter in Init_Image_GPU\n");
		*poImage = { {0 } };
		return;
	}

	poImage->m_iWidth = iWidth;
	poImage->m_iHeight = iHeight;
	poImage->m_iBit_Count = iBit_Count;
	poImage->m_iChannel_Count = iBit_Count >> 3;
	poImage->m_iMem_Src = Mem_Src::GPU;
	poImage->m_iGPU_ID = iGPU_ID;

	iSize = iWidth * iHeight;
	poImage->m_iMax_Buffer_Size = iSize * poImage->m_iChannel_Count;

	//������أ�1��������64�ֽ����ϣ�2��128�ֽڱ������Ա�GPU����
	iSize_With_Remain = ((poImage->m_iMax_Buffer_Size + 127) / 128) * 128;
	if (iSize_With_Remain - poImage->m_iMax_Buffer_Size < 64)
		iSize_With_Remain += 128;

	if (poPtr)
	{
        oPtr = *poPtr;
		Malloc(oPtr, iSize_With_Remain, poImage->m_pBuffer);
		poImage->m_pChannel[0]= poImage->m_pBuffer;
	}
	else
		poImage->m_pChannel[0] = poImage->m_pBuffer = (unsigned char*)pMalloc_GPU(iSize_With_Remain);
	
	for (i = 1; i < poImage->m_iChannel_Count; i++)
		poImage->m_pChannel[i] = poImage->m_pChannel[i - 1] + iSize;
	for (; i < 4; i++)
		poImage->m_pChannel[i] = NULL;

	if (poPtr)
		*poPtr = oPtr;
}
void Copy_Image_To_CPU(Image oOrg, Image oNew)
{
	int iSize;
	iSize = oOrg.m_iWidth * oOrg.m_iHeight * Min(oOrg.m_iChannel_Count, oNew.m_iChannel_Count);
	if (iSize <= 0 || !oOrg.m_pChannel[0] || !oNew.m_pChannel[0])
	{
		printf("Invalid parameter in Copy_Image_To_GPU\n");
		return;
	}
    /*for(int i=0;i<oNew.m_iChannel_Count;i++)
	    hipMemcpyAsync(oNew.m_pChannel[i], oOrg.m_pChannel[i], oNew.m_iWidth * oNew.m_iHeight, hipMemcpyDeviceToHost);*/
    hipMemcpyAsync(oNew.m_pChannel[0], oOrg.m_pChannel[0], iSize, hipMemcpyDeviceToHost);
    //Disp_Cuda_Error();
}
void Copy_Image_To_GPU(Image oOrg, Image oNew)
{
	int iSize;
	iSize = oOrg.m_iWidth * oOrg.m_iHeight * Min(oOrg.m_iChannel_Count, oNew.m_iChannel_Count);
	if (iSize <= 0 || !oOrg.m_pChannel[0] || !oNew.m_pChannel[0] ||
		oNew.m_iMem_Src != Mem_Src::GPU)
	{
		printf("Invalid parameter in Copy_Image_To_GPU\n");
		return;
	}
	hipMemcpyAsync(oNew.m_pChannel[0], oOrg.m_pChannel[0], iSize, oOrg.m_iMem_Src == Mem_Src::CPU ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice);
}
void Free_Image_GPU(Image* poImage)
{
	if (poImage && poImage->m_pBuffer)
	{
		Free_GPU(poImage->m_pBuffer);
		*poImage = {};
	}
	return;
}
int bLoad_Comp_GPU(const char* pcFile, Image oDest, int iComp)
{//��ʱ����ɫͼ����Ŀ��ͼĳͨ����
    Image oImage;
    if (!bLoad_Image(pcFile, &oImage))
        return 0;
    if ((oDest.m_iWidth != oImage.m_iWidth) ||
        (oDest.m_iHeight != oImage.m_iHeight) ||
        (!oDest.m_pChannel[iComp]))
    {
        printf("Invalid image parameter in bLoad_Comp_GPU\n");
        return 0;
    }
    hipMemcpy(oDest.m_pChannel[iComp], oImage.m_pChannel[0], oImage.m_iWidth * oImage.m_iHeight, hipMemcpyHostToDevice);
    Disp_Cuda_Error();
    Free_Image(&oImage);
    return 0;
}
int bLoad_Image_GPU(const char* pcFile, Image* poImage, int iWidth, int iHeight, int iFrame)
{//�򻯲��������poImage->m_pBuffer�Ѿ��ж��������ÿ��ڴ�
	Image oImage;
	if (!bLoad_Image(pcFile, &oImage))
		return 0;

	if (!poImage->m_pBuffer)
		Init_Image_GPU(poImage, oImage.m_iWidth, oImage.m_iHeight, Image::IMAGE_TYPE_BMP, oImage.m_iBit_Count);
	else if (oImage.m_iWidth - poImage->m_iWidth +
		oImage.m_iHeight - poImage->m_iHeight /*+
		oImage.m_iChannel_Count - poImage->m_iChannel_Count*/
        )
	{
		printf("Invalid image parameter in bLoad_Image_GPU\n");
		return 0;
	}
	Copy_Image_To_GPU(oImage, *poImage);
	Disp_Cuda_Error();
    Free_Image(&oImage);
	return 1;
}

int bSave_Image_GPU(const char* pcFile, Image oImage)
{//ͷ���ڴ棬�������Դ�
	Image oDest;
	Init_Image(&oDest, oImage.m_iWidth, oImage.m_iHeight, Image::IMAGE_TYPE_BMP, oImage.m_iBit_Count);
	Copy_Image_To_CPU(oImage, oDest);
    Disp_Cuda_Error();
	int bResult=bSave_Image(pcFile, oDest);
	Free_Image(&oDest);
	return bResult;
}

int bSave_Image_GPU(const char* pcFile, Image *poHeader_GPU)
{//��ͷ����GPU
    Image oImage;
    hipMemcpy(&oImage, poHeader_GPU, sizeof(Image), hipMemcpyDeviceToHost);
    return bSave_Image_GPU(pcFile, oImage);
}
int bSave_Comp_GPU(const char* pcFile, Image * poHeader_GPU, int iComp)
{
    Image oImage;
    hipMemcpy(&oImage, poHeader_GPU, sizeof(Image), hipMemcpyDeviceToHost);
    return bSave_Comp_GPU(pcFile, oImage,iComp);
}
int bSave_Comp_GPU(const char* pcFile, Image oImage,int iComp)
{
    Image oImage_1;
    Attach_Buffer(&oImage_1, oImage.m_pChannel[iComp], oImage.m_iWidth, oImage.m_iHeight, 1, Image::IMAGE_TYPE_BMP);
    return bSave_Image_GPU(pcFile, oImage_1);
}


//__global__ void Bi_Linear_cv_Reflect_GPU(Image Source[],Image Dest[], 
//    int iCount, float f_x, float f_y)
//{
//    //threadIdx: threadId in a block
//    //blockDim.x = Thread_Per_Block
//    //gridDim.x = Size/blockDim.x
//    //gridDim.y = Channel_Count
//    //gridDim.z = Image_Count
//    //blockIdx.x = Block_ID 
//    //blockIdx.y = Channel in Image
//    //blockIdx.z = Image ID in Group
//
//    int iThread_ID = blockIdx.x * blockDim.x + threadIdx.x;
//    //if (iThread_ID == 0)
//    //{
//    //    //printf("%d %d %d\n", blockDim.x, blockDim.y, blockDim.z);
//    //    //printf("%d %d %d\n", gridDim.x, gridDim.y, gridDim.z);
//    //    printf("blockIdx: %d %d %d\n", blockIdx.x, blockIdx.y, blockIdx.z);
//    //}
//    
//    unsigned short w_d = Dest[blockIdx.z].m_iWidth,
//        h_d = Dest[blockIdx.z].m_iHeight,
//        w_s = Source[blockIdx.z].m_iWidth,
//        h_s = Source[blockIdx.z].m_iHeight;
//
//    if (iThread_ID > w_d * h_d)
//    {
//        //printf("exit");
//        return;
//    }
//      
//    int x_d = iThread_ID % w_d,
//        y_d = iThread_ID / w_d;
//        
//    float x_s_f = (x_d + 0.5f) * f_x - 0.5f;
//    float y_s_f = (y_d + 0.5f) * f_y - 0.5f,
//
//    int y_s_0 = (int)floor(y_s_f);
//    int y_s_1 = y_s_0 + 1;
//    float w2 = (y_s_1 - y_s_f), w3 = 1.f - w2;
//
//    unsigned char* pCur_Line = &Source[blockIdx.z].m_pChannel[blockIdx.y][iGet_Border_y(y_s_0, h_s, BORDER_REFLECT) * w_s];
//    unsigned char* pNext_Line = &Source[blockIdx.z].m_pChannel[blockIdx.y][iGet_Border_y(y_s_1, h_s, BORDER_REFLECT) * w_s];
//
//    int x_s_0 = (int)floor(x_s_f);
//    int x_s_1 = x_s_0 + 1;
//    float w0 = (x_s_1 - x_s_f), w1 = 1.f - w0;
//
//    int x_s_0_r = iGet_Border_x(x_s_0, w_s, BORDER_REFLECT),
//        x_s_1_r = iGet_Border_x(x_s_1, w_s, BORDER_REFLECT);
//
//    float fValue_0, fValue_1;
//    fValue_0 = w0 * pCur_Line[x_s_0_r] + w1 * pCur_Line[x_s_1_r];
//    fValue_1 = w0 * pNext_Line[x_s_0_r] + w1 * pNext_Line[x_s_1_r];
//
//    Dest[blockIdx.z].m_pChannel[blockIdx.y][iThread_ID] = (unsigned char)(w2 * fValue_0 + w3 * fValue_1 + 0.5);
//    
//    return;
//}

__global__ void Bi_Linear_cv_Reflect_GPU(Image Source[], Image Dest[],
    int iCount, float f_x, float f_y,
    short w_s, short h_s, short w_d, short h_d)
{
    //threadIdx: threadId in a block
    //blockDim.x = Thread_Per_Block
    //gridDim.x = Size/blockDim.x
    //gridDim.y = Channel_Count
    //gridDim.z = Image_Count
    //blockIdx.x = Block_ID 
    //blockIdx.y = Channel in Image
    //blockIdx.z = Image ID in Group

    int iThread_ID = blockIdx.x * blockDim.x + threadIdx.x;
    /*unsigned short w_d = Dest[blockIdx.z].m_iWidth,
        h_d = Dest[blockIdx.z].m_iHeight,
        w_s = Source[blockIdx.z].m_iWidth,
        h_s = Source[blockIdx.z].m_iHeight;*/

    if (iThread_ID >= w_d * h_d)
        return;

    int x_d = iThread_ID % w_d,
        y_d = iThread_ID / w_d;

    float w2, w3, x_s_f = (x_d + 0.5f) * f_x - 0.5f;

    /*__shared__ unsigned char* pSource, * pDest;
    if (threadIdx.x == 0)
    {
        pSource = Source[blockIdx.z].m_pChannel[blockIdx.y];
        pDest = Dest[blockIdx.z].m_pChannel[blockIdx.y];
    }*/

    __syncthreads();

    unsigned char* pCur_Line, * pNext_Line;
    {
        float y_s_f = (y_d + 0.5f) * f_y - 0.5f;
        int y_s_0 = (int)floor(y_s_f);
        int y_s_1 = y_s_0 + 1;
        w2 = (y_s_1 - y_s_f), w3 = 1.f - w2;
        //pCur_Line = &pSource[iGet_Border_y_GPU(y_s_0, h_s, BORDER_REFLECT) * w_s];
        //pNext_Line = &pSource[iGet_Border_y_GPU(y_s_1, h_s, BORDER_REFLECT) * w_s];
        pCur_Line = &Source[blockIdx.z].m_pChannel[blockIdx.y][iGet_Border_y_GPU(y_s_0, h_s, BORDER_REFLECT) * w_s];
        pNext_Line = &Source[blockIdx.z].m_pChannel[blockIdx.y][iGet_Border_y_GPU(y_s_1, h_s, BORDER_REFLECT) * w_s];
    }

    float w0, w1;
    int x_s_0_r, x_s_1_r;
    {
        int x_s_0 = (int)floor(x_s_f);
        int x_s_1 = x_s_0 + 1;
        w0 = (x_s_1 - x_s_f), w1 = 1.f - w0;
        x_s_0_r = iGet_Border_x_GPU(x_s_0, w_s, BORDER_REFLECT);
        x_s_1_r = iGet_Border_x_GPU(x_s_1, w_s, BORDER_REFLECT);
    }

    float fValue_0, fValue_1;
    fValue_0 = w0 * pCur_Line[x_s_0_r] + w1 * pCur_Line[x_s_1_r];
    fValue_1 = w0 * pNext_Line[x_s_0_r] + w1 * pNext_Line[x_s_1_r];

    Dest[blockIdx.z].m_pChannel[blockIdx.y][iThread_ID] = (unsigned char)(w2 * fValue_0 + w3 * fValue_1 + 0.5);
    //pDest[iThread_ID] = (unsigned char)(w2 * fValue_0 + w3 * fValue_1 + 0.5);
    return;
}

void Bi_Linear_cv_GPU(Image Source[], Image Dest[], int iCount, 
    int w_s, int h_s, int w_d, int h_d, int iChannel,
    float fScale_x, float fScale_y, Border_Type iBorder_Type)
{//��opencv��˫���Բ�ֵд��GPU�汾
//�޿���������ӿں���������̽���������ܿ��
    int iSize;  // = iCount * 2 * sizeof(Image);
    //�ȶ�Seam_est����Bi_Linear
    int iThread_Per_Block = 256;
    //iSize = Dest[0].m_iWidth * Dest[0].m_iHeight;
    iSize = w_d * h_d;
    dim3 oGrid;
    oGrid.x = (iSize + iThread_Per_Block - 1) / iThread_Per_Block;
    oGrid.y = iChannel;
    oGrid.z = iCount;

    //Disp_Cuda_Error();
    Bi_Linear_cv_Reflect_GPU << <oGrid, iThread_Per_Block >> > (Source, Dest, iCount, 1.f / fScale_x, 1.f / fScale_y, w_s, h_s, w_d, h_d);
    //Disp_Cuda_Error();
    return;
}

__global__ void _Set_Color_GPU(Image oBitMap, unsigned int C0, unsigned int C1, unsigned int C2)
{//������첽����hipMemcpyAsync��ܶ�
    typedef struct Pixel_4 {
        char Data[4];
    }Pixel_4;

    int iThread_ID = GET_THREAD_ID() << 2;
    union {
        Pixel_4 oValue;
        unsigned int iValue;
    };
    if (iThread_ID + 4 <= oBitMap.m_iWidth * oBitMap.m_iHeight)
    {
        iValue = C0;
        *(Pixel_4*)&oBitMap.m_pChannel[0][iThread_ID] = oValue;
        if(oBitMap.m_iChannel_Count==3)
        {
            iValue = C1;
            *(Pixel_4*)&oBitMap.m_pChannel[1][iThread_ID] = oValue;
            iValue = C2;
            *(Pixel_4*)&oBitMap.m_pChannel[2][iThread_ID] = oValue;
        }
    }else
    {
        while (iThread_ID < oBitMap.m_iWidth * oBitMap.m_iHeight)
        {
            oBitMap.m_pChannel[0][iThread_ID] = C0 & 0xFF;
            if (oBitMap.m_iChannel_Count == 3)
            {
                oBitMap.m_pChannel[1][iThread_ID] = C1 & 0xFF;
                oBitMap.m_pChannel[2][iThread_ID] = C2 & 0xFF;
            }
            iThread_ID++;
        }
    }
}

void Set_Color_GPU(Image oImage, int R, int G, int B, unsigned long long iStream)
{
    int iThread_Per_Block = 1024, iSize = oImage.m_iWidth * oImage.m_iHeight,
        iBlock_Count;
    int Color[3];
    if (oImage.m_iImage_Type == Image::IMAGE_TYPE_BMP)
    {
        Color[0] = (R << 24) + (R << 16) + (R << 8) + R;
        Color[1] = (G << 24) + (G << 16) + (G << 8) + G;
        Color[2] = (B << 24) + (B << 16) + (B << 8) + B;
    }
    else
    {
        _RGB_2_YUV(R, G, B, Color[0], Color[1], Color[2]);
        Color[0] = (Color[0] << 24) + (Color[0] << 16) + (Color[0] << 8) + Color[0];
        Color[1] = (Color[1] << 24) + (Color[1] << 16) + (Color[1] << 8) + Color[1];
        Color[2] = (Color[2] << 24) + (Color[2] << 16) + (Color[2] << 8) + Color[2];
    }

    iBlock_Count = (((iSize + 3) >> 2) + iThread_Per_Block - 1) / iThread_Per_Block;
    _Set_Color_GPU << <iBlock_Count, iThread_Per_Block >> > (oImage, Color[0], Color[1], Color[2]);
    return;
}

__global__ void _Pyr_Down_col_GPU(Data_Block<unsigned short*, 3>oMid, int iMid_Height, Image oDest, Border_Type iBorder_Type = BORDER_REFLECT101)
{//�з���
    int iThread_ID = GET_THREAD_ID();
    if (iThread_ID >= oDest.m_iHeight * oDest.m_iWidth)
        return;
    short x = iThread_ID % oDest.m_iWidth,
        y = iThread_ID / oDest.m_iWidth;

    short y1 = y << 1;
    int iMid_Pos = y1 * oDest.m_iWidth;
    int iMid_Size = iMid_Height * oDest.m_iWidth;
    unsigned short* pSource = &oMid.Data[0][x];
    unsigned int Mid_Pos[4] = { (unsigned int) iGet_Border_y_GPU(y1 - 2, iMid_Height, iBorder_Type) * oDest.m_iWidth,
        (unsigned int)iGet_Border_y_GPU(y1 - 1, iMid_Height, iBorder_Type)* oDest.m_iWidth,
        (unsigned int)iGet_Border_y_GPU(y1 + 1, iMid_Height, iBorder_Type)* oDest.m_iWidth,
        (unsigned int)iGet_Border_y_GPU(y1 + 2, iMid_Height, iBorder_Type)* oDest.m_iWidth };

    //for(int i=0;i<oDest.m_iChannel_Count;i++, pSource += iMid_Size)
    //{
        unsigned short iValue = pSource[iMid_Pos] * 6 +
            ((pSource[Mid_Pos[1]] + pSource[Mid_Pos[2]]) << 2) +
            pSource[Mid_Pos[0]] + pSource[Mid_Pos[3]];
        oDest.m_pChannel[0][iThread_ID] = (iValue + 128) >> 8;
    //}
        if(oDest.m_iChannel_Count >1)
        {
            pSource += iMid_Size;
            iValue = pSource[iMid_Pos] * 6 +
                ((pSource[Mid_Pos[1]] + pSource[Mid_Pos[2]]) << 2) +
                pSource[Mid_Pos[0]] + pSource[Mid_Pos[3]];
            oDest.m_pChannel[1][iThread_ID] = (iValue + 128) >> 8;

            if (oDest.m_iChannel_Count > 2)
            {
                pSource += iMid_Size;
                    iValue = pSource[iMid_Pos] * 6 +
                    ((pSource[Mid_Pos[1]] + pSource[Mid_Pos[2]]) << 2) +
                    pSource[Mid_Pos[0]] + pSource[Mid_Pos[3]];
                oDest.m_pChannel[2][iThread_ID] = (iValue + 128) >> 8;
            }
        }

    /*for (int i = 0; i < oDest.m_iChannel_Count; i++)
    {
        unsigned short* pSource = &oMid.Data[i][x];
        unsigned short iValue = pSource[y1 * oDest.m_iWidth] * 6 +
            (pSource[iGet_Border_y_GPU(y1 - 1, iMid_Height, iBorder_Type) * oDest.m_iWidth] + pSource[iGet_Border_y_GPU(y1 + 1, iMid_Height, iBorder_Type) * oDest.m_iWidth]) * 4 +
            pSource[iGet_Border_y_GPU(y1 - 2, iMid_Height, iBorder_Type) * oDest.m_iWidth] + pSource[iGet_Border_y_GPU(y1 + 2, iMid_Height, iBorder_Type) * oDest.m_iWidth];
        oDest.m_pChannel[i][iThread_ID] = (iValue + 128) >> 8;
    }*/
    return;
}

__global__ void _Pyr_Down_row_Ref_GPU(Image oSource, int iMid_Width, Data_Block<unsigned short*, 3>oMid, Border_Type iBorder_Type = BORDER_REFLECT101)
{//�д����������
    int iThread_ID = GET_THREAD_ID();
    if (iThread_ID >= oSource.m_iHeight * iMid_Width)
        return;

    short x = iThread_ID % iMid_Width,
        y = iThread_ID / iMid_Width;
    short x1 = x << 1;
    int iSize_s = oSource.m_iWidth * oSource.m_iHeight;
    unsigned char* pSource = &oSource.m_pChannel[0][y * oSource.m_iWidth];
    unsigned short Source_Pos[4] = { (unsigned short)iGet_Border_x_GPU(x1 - 2, oSource.m_iWidth, iBorder_Type),
        (unsigned short)iGet_Border_x_GPU(x1 - 1, oSource.m_iWidth, iBorder_Type),
       (unsigned short)iGet_Border_x_GPU(x1 + 1, oSource.m_iWidth, iBorder_Type),
       (unsigned short)iGet_Border_x_GPU(x1 + 2, oSource.m_iWidth, iBorder_Type) };

    oMid.Data[0][iThread_ID] = pSource[x1] * 6 +
        ((pSource[Source_Pos[1]] + pSource[Source_Pos[2]]) << 2) +
        pSource[Source_Pos[0]] + pSource[Source_Pos[3]];
    pSource += iSize_s;

    if (oSource.m_iChannel_Count > 1)
    {
        oMid.Data[1][iThread_ID] = pSource[x1] * 6 +
            ((pSource[Source_Pos[1]] + pSource[Source_Pos[2]]) << 2) +
            pSource[Source_Pos[0]] + pSource[Source_Pos[3]];
        pSource += iSize_s;

        if (oSource.m_iChannel_Count > 2)
            oMid.Data[2][iThread_ID] = pSource[x1] * 6 +
            ((pSource[Source_Pos[1]] + pSource[Source_Pos[2]]) << 2) +
            pSource[Source_Pos[0]] + pSource[Source_Pos[3]];
    }

    return;
}

void Pyr_Down_GPU(Image oSource, Image oDest,unsigned short *pAux)
{//������˹�������²���
    Data_Block<unsigned short*, 3>oMid;
    int iSize = oSource.m_iHeight * oDest.m_iWidth;

    if (pAux)
        oMid.Data[0] = pAux;
    else
        oMid.Data[0] = (unsigned short*)pMalloc_GPU(iSize * oSource.m_iChannel_Count * sizeof(unsigned short));
    
    oMid.Data[1] = oMid.Data[0] + iSize;
    oMid.Data[2] = oMid.Data[1] + iSize;

    dim3 oThread, oGrid;
    //�ȸ��з���
    oThread.x = Min(512, iSize);
    oGrid.x = (iSize + oThread.x - 1) / oThread.x;
    //oGrid.y = oDest.m_iChannel_Count;
    //Disp_GPU(oSource.m_pChannel[0], oSource.m_iHeight, oSource.m_iWidth, "Source");
    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    _Pyr_Down_row_Ref_GPU << <oGrid, oThread >> > (oSource, oDest.m_iWidth, oMid);
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);
    //Disp_GPU(oMid.Data[0], 1, 4);

    //�ٵ��з���
    iSize = oDest.m_iWidth * oDest.m_iHeight;
    oThread.x = Min(512, iSize);
    oGrid.x = (iSize + oThread.x - 1) / oThread.x;

    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    _Pyr_Down_col_GPU << <oGrid, oThread >> > (oMid, oSource.m_iHeight, oDest);
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    //Disp_GPU(oDest.m_pChannel[0], oDest.m_iHeight, oDest.m_iWidth, "Dest");
    ////����Commit�ƺ�����ʡ�������������ΪMid�������ͷų�������
    ////Disp_Cuda_Error();
    //bSave_Image_GPU("c:\\tmp\\4.bmp", oDest);
    //Compare_Image("c:\\tmp\\3.bmp", "c:\\tmp\\4.bmp");
    if (!pAux && oMid.Data[0])
        Free_GPU(oMid.Data[0]);
    return;
}
__global__ void _Pyr_Up_row_GPU(Image oSource, int iMid_Width, Data_Block<unsigned short*, 3> oMid)
{
    int iThread_ID = GET_THREAD_ID();
    int iSize_s = oSource.m_iWidth * oSource.m_iHeight;
    if (iThread_ID >= iSize_s)
        return;

    short x = iThread_ID % oSource.m_iWidth,
        y = iThread_ID / oSource.m_iWidth;
    unsigned char bHas_Remain_x = iMid_Width > (oSource.m_iWidth << 1) && (x == oSource.m_iWidth - 1) ? 1 : 0;
    unsigned char bIs_Source_Border = (x == oSource.m_iWidth - 1);
    unsigned char bEven = bIs_Source_Border && ((x << 1) + 1 < iMid_Width);

    int iPos_m = y * iMid_Width,
        iPos_s = y * oSource.m_iWidth;
    int iSize_m = iMid_Width * oSource.m_iHeight;
    
    unsigned short* pMid = &oMid.Data[0][iPos_m + (x << 1)];
    for (short i = 0; i < oSource.m_iChannel_Count; i++,pMid+=iSize_m)
    {
        //�Ż��������첻��
        
        unsigned short Mid[3];
        //�м��
        unsigned char iPix = oSource.m_pChannel[i][iPos_s + x];
        if (x == 0)
        {//�������
            Mid[0] = iPix * 6 + (oSource.m_pChannel[i][iPos_s + 1] << 1);
            Mid[1] = (iPix + oSource.m_pChannel[i][iPos_s + 1]) << 2;
        }
        else if (bIs_Source_Border)
        {//�ұ����㣬2���������
            Mid[0] = oSource.m_pChannel[i][iPos_s + x - 1] + iPix * 7;
            if (bEven)  //Դ�����һ����Զ�����ӦĿ��Ҳ������
                Mid[1] = iPix << 3;
        }
        else
        {//һ�����
            Mid[0] = oSource.m_pChannel[i][iPos_s + x - 1] + iPix * 6 + oSource.m_pChannel[i][iPos_s + x + 1];
            Mid[1] = (iPix + oSource.m_pChannel[i][iPos_s + x + 1]) << 2;
        }

        pMid[0] = Mid[0];
        if (!bIs_Source_Border || bEven)
        {
            pMid[1] = Mid[1];
            if (bHas_Remain_x)  //���������
                pMid[2] = Mid[1];
        }

        //if (x == 0)
        //{//�������
        //    oMid.Data[i][iPos_m] = oSource.m_pChannel[i][iPos_s] * 6 + oSource.m_pChannel[i][iPos_s + 1] * 2;
        //    oMid.Data[i][iPos_m + 1] = (oSource.m_pChannel[i][iPos_s] + oSource.m_pChannel[i][iPos_s + 1]) * 4;
        //}else if (x == oSource.m_iWidth - 1)
        //{//�ұ�����
        //    oMid.Data[i][iPos_m + (x << 1)] = oSource.m_pChannel[i][iPos_s + x - 1] + oSource.m_pChannel[i][iPos_s + x] * 7;
        //    if ((x << 1) + 1 < iMid_Width)
        //    {
        //        oMid.Data[i][iPos_m + (x << 1) + 1] = oSource.m_pChannel[i][iPos_s + x] * 8;
        //        if (bHas_Remain_x)  //���������
        //            oMid.Data[i][iPos_m + (x << 1) + 2] = oMid.Data[i][iPos_m + (x << 1) + 1];
        //    }
        //}else
        //{//�м��
        //    oMid.Data[i][iPos_m + (x << 1)] = oSource.m_pChannel[i][iPos_s + x - 1] + oSource.m_pChannel[i][iPos_s + x] * 6 + oSource.m_pChannel[i][iPos_s + x + 1];
        //    oMid.Data[i][iPos_m + (x << 1) + 1] = (oSource.m_pChannel[i][iPos_s + x] + oSource.m_pChannel[i][iPos_s + x + 1]) * 4;
        //}
    }
}
__global__ void _Pyr_Up_col_Subtract_GPU(Data_Block<unsigned short*, 3>oMid, int iMid_Height, Image oDest)
{//�в����Ժ��ټ��ϼ�������
    int iThread_ID = GET_THREAD_ID();
    int iSize = iMid_Height * oDest.m_iWidth;

    if (iThread_ID >= iSize)
        return;

    short x = iThread_ID % oDest.m_iWidth,
        y = iThread_ID / oDest.m_iWidth;
    unsigned char bHas_Remain_y = oDest.m_iHeight > iMid_Height * 2 && y == iMid_Height - 1 ? 1 : 0;
    unsigned char bEven = (y * 2 + 1 < oDest.m_iHeight);

    //�ظ������ȫ����ȡ�������첻��
    unsigned short* r0 = &oMid.Data[0][iGet_Border_y_GPU(y - 1, iMid_Height, BORDER_REFLECT101) * oDest.m_iWidth + x],
        * r1 = &oMid.Data[0][iGet_Border_y_GPU(y, iMid_Height, BORDER_REFLECT) * oDest.m_iWidth + x],
        * r2 = &oMid.Data[0][iGet_Border_y_GPU(y + 1, iMid_Height, BORDER_REFLECT) * oDest.m_iWidth + x];

    //int iPos_d = (y * 2) * oDest.m_iWidth + x;
    //for (int i = 0; i < oDest.m_iChannel_Count; i++, r0 += iSize, r1 += iSize, r2 += iSize)
    //{
    //    short iValue = oDest.m_pChannel[i][iPos_d] - ((*r0 + *r1 * 6 + *r2 + 32) >> 6);
    //    oDest.m_pChannel[i][iPos_d] = Clip3(-128, 127, iValue);
    //    if (bEven)
    //    {
    //        //d1�� =   [(r1 + r2)*4 + 32]>>6
    //        iValue = oDest.m_pChannel[i][iPos_d + oDest.m_iWidth] - ((((*r1 + *r2) << 2) + 32) >> 6);
    //        oDest.m_pChannel[i][iPos_d + oDest.m_iWidth] = Clip3(-128, 127, iValue);
    //        if (bHas_Remain_y)
    //        {
    //            iValue = oDest.m_pChannel[i][iPos_d + (oDest.m_iWidth << 1)] - oDest.m_pChannel[i][iPos_d];
    //            oDest.m_pChannel[i][iPos_d + (oDest.m_iWidth << 1)] = Clip3(-128, 127, iValue);
    //        }
    //    }
    //}

    unsigned char* pDest = &oDest.m_pChannel[0][(y * 2) * oDest.m_iWidth + x];
    int iDest_Size = oDest.m_iWidth * oDest.m_iHeight;
    int iValue = *pDest - ((*r0 + *r1 * 6 + *r2 + 32) >> 6);
    *pDest = Clip3(-128, 127, iValue);
    if (bEven)
    {
        iValue = pDest[oDest.m_iWidth] - ((((*r1 + *r2) << 2) + 32) >> 6);
        pDest[oDest.m_iWidth] = Clip3(-128, 127, iValue);
        if (bHas_Remain_y)
        {
            iValue = pDest[oDest.m_iWidth << 1] - (*pDest);
            pDest[oDest.m_iWidth << 1] = Clip3(-128, 127, iValue);
        }
    }

    if (oDest.m_iChannel_Count > 1)
    {
        pDest += iDest_Size, r0 += iSize, r1 += iSize, r2 += iSize;
        iValue = *pDest - ((*r0 + *r1 * 6 + *r2 + 32) >> 6);
        *pDest = Clip3(-128, 127, iValue);
        if (bEven)
        {
            iValue = pDest[oDest.m_iWidth] - ((((*r1 + *r2) << 2) + 32) >> 6);
            pDest[oDest.m_iWidth] = Clip3(-128, 127, iValue);
            if (bHas_Remain_y)
            {
                iValue = pDest[oDest.m_iWidth << 1] - (*pDest);
                pDest[oDest.m_iWidth << 1] = Clip3(-128, 127, iValue);
            }
        }

        if (oDest.m_iChannel_Count > 2)
        {
            pDest += iDest_Size, r0 += iSize, r1 += iSize, r2 += iSize;
            iValue = *pDest - ((*r0 + *r1 * 6 + *r2 + 32) >> 6);
            *pDest = Clip3(-128, 127, iValue);
            if (bEven)
            {
                iValue = pDest[oDest.m_iWidth] - ((((*r1 + *r2) << 2) + 32) >> 6);
                pDest[oDest.m_iWidth] = Clip3(-128, 127, iValue);
                if (bHas_Remain_y)
                {
                    iValue = pDest[oDest.m_iWidth << 1] - (*pDest);
                    pDest[oDest.m_iWidth << 1] = Clip3(-128, 127, iValue);
                }
            }
        }
    }

}
__global__ void _Pyr_Up_col_GPU(Data_Block<unsigned short*, 3>oMid, int iMid_Height, Image oDest)
{
    int iThread_ID = GET_THREAD_ID();
    int iSize = iMid_Height * oDest.m_iWidth;

    if (iThread_ID >= iSize)
        return;

    short x = iThread_ID % oDest.m_iWidth,
        y = iThread_ID / oDest.m_iWidth;
    unsigned char bHas_Remain_y = oDest.m_iHeight > iMid_Height * 2 && y == iMid_Height - 1 ? 1 : 0;
    unsigned char bEven = (y * 2 + 1 < oDest.m_iHeight);

    //�ظ������ȫ����ȡ�������첻��
    unsigned short* r0 = &oMid.Data[0][iGet_Border_y_GPU(y - 1, iMid_Height, BORDER_REFLECT101) * oDest.m_iWidth + x],
        * r1 = &oMid.Data[0][iGet_Border_y_GPU(y, iMid_Height, BORDER_REFLECT) * oDest.m_iWidth + x],
        * r2 = &oMid.Data[0][iGet_Border_y_GPU(y + 1, iMid_Height, BORDER_REFLECT) * oDest.m_iWidth + x];
    
    //int iPos_d = (y * 2) * oDest.m_iWidth + x;
    //for (int i = 0; i < oDest.m_iChannel_Count; i++, r0 += iSize, r1 += iSize, r2 += iSize)
    //{
    //    oDest.m_pChannel[i][iPos_d] = (*r0 + *r1 * 6 + *r2 + 32) >> 6;
    //    if (bEven)
    //    {
    //        //d1�� =   [(r1 + r2)*4 + 32]>>6
    //        oDest.m_pChannel[i][iPos_d + oDest.m_iWidth] = (((*r1 + *r2) << 2) + 32) >> 6;
    //        if (bHas_Remain_y)
    //            oDest.m_pChannel[i][iPos_d + (oDest.m_iWidth << 1)] = oDest.m_pChannel[i][iPos_d];
    //    }
    //}
    unsigned char* pDest = &oDest.m_pChannel[0][(y * 2) * oDest.m_iWidth + x];
    int iDest_Size = oDest.m_iWidth * oDest.m_iHeight;
    *pDest = (*r0 + *r1 * 6 + *r2 + 32) >> 6;
    if (bEven)
    {
        pDest[oDest.m_iWidth] = (((*r1 + *r2) << 2) + 32) >> 6;
        if (bHas_Remain_y)
            pDest[oDest.m_iWidth << 1] = *pDest;
    }

    if(oDest.m_iChannel_Count>1)
    {
        pDest += iDest_Size, r0 += iSize, r1 += iSize, r2 += iSize;
        *pDest = (*r0 + *r1 * 6 + *r2 + 32) >> 6;
        if (bEven)
        {
            pDest[oDest.m_iWidth] = (((*r1 + *r2) << 2) + 32) >> 6;
            if (bHas_Remain_y)
                pDest[oDest.m_iWidth << 1] = *pDest;
        }

        if (oDest.m_iChannel_Count > 2)
        {
            pDest += iDest_Size, r0 += iSize, r1 += iSize, r2 += iSize;
            *pDest = (*r0 + *r1 * 6 + *r2 + 32) >> 6;
            if (bEven)
            {
                pDest[oDest.m_iWidth] = (((*r1 + *r2) << 2) + 32) >> 6;
                if (bHas_Remain_y)
                    pDest[oDest.m_iWidth << 1] = *pDest;
            }
        }
    }
}

void Pyr_Up_GPU(Image oSource, Image oDest)
{
    Data_Block<unsigned short*, 3>oMid;
    int iSize = oSource.m_iHeight * oDest.m_iWidth;
    oMid.Data[0] = (unsigned short*)pMalloc_GPU(iSize * oDest.m_iChannel_Count * sizeof(unsigned short));
    oMid.Data[1] = oMid.Data[0] + iSize;
    oMid.Data[2] = oMid.Data[1] + iSize;

    dim3 oThread, oGrid;
    iSize = oSource.m_iWidth * oSource.m_iHeight;
    oThread.x = Min(512, iSize);
    oGrid.x = (iSize + oThread.x - 1) / oThread.x;

    /*Disp_Cuda_Error();
    unsigned long long tStart = iGet_Tick_Count();
    for(int i=0;i<10000;i++)*/
    _Pyr_Up_row_GPU << <oGrid, oThread >> > (oSource, oDest.m_iWidth, oMid);
    /*Disp_Cuda_Error();
    printf("%lld\n", iGet_Tick_Count() - tStart);*/

    //Disp_GPU(oMid.Data[0], oSource.m_iHeight, oDest.m_iWidth, "Mid");
    iSize = oDest.m_iWidth * oSource.m_iHeight;
    oThread.x = Min(512, iSize);
    oGrid.x = (iSize + oThread.x - 1) / oThread.x;

    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();
    //for(int i=0;i<10000;i++)
    //_Pyr_Up_col_GPU << <oGrid, oThread >> > (oMid, oSource.m_iHeight, oDest);
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);

    _Pyr_Up_col_Subtract_GPU << <oGrid, oThread >> > (oMid, oSource.m_iHeight, oDest);

    ////Disp_GPU(oDest.m_pChannel[0], oDest.m_iHeight, oDest.m_iWidth, "Dest");
    //bSave_Image_GPU("c:\\tmp\\3.bmp", oDest);
    //Compare_Image("c:\\tmp\\2.bmp", "c:\\tmp\\3.bmp");
    if (oMid.Data[0])
        Free_GPU(oMid.Data[0]);
    return;
}

__global__ void Gen_Gauss_Filter_GPU(int r, float fSigma, float Kernel[])
{
    int i, j, d = r + r + 1;
    float* pFilter = Kernel;
    float fValue, fSum = 0;

    for (i = 0; i < r; i++)
    {
        fValue = (float)(i - r) / fSigma;
        pFilter[i] = (float)exp(-0.5f * fValue * fValue);
        fSum += pFilter[i];
    }

    fValue = (float)(i - r) / fSigma;
    pFilter[i] = (float)exp(-0.5f * fValue * fValue);
    fSum = fSum * 2 + pFilter[i];

    fValue = 1.f / fSum;
    j = d - 1;
    for (i = 0; i < r; i++, j--)
        pFilter[j] = (pFilter[i] *= fValue);
    pFilter[r] *= fValue;
    return;
}
__global__ void _Copy_Make_Border_GPU(Image oSource, Image oDest, short iLeft, short iTop, short iRight, short iBottom, char iPix_Group_Size)
{//�е��ң���Ҫ����

    int iDest_Size = oDest.m_iWidth * oDest.m_iHeight;
    short x_s = threadIdx.x * iPix_Group_Size;
    short x_d = x_s + iLeft,
        y_d = blockIdx.x + iTop;

    {//�ȳ��м䲿��
        //int iPos_s = blockIdx.x * oSource.m_iWidth + x_s;
        //int iPos_d = y_d * oDest.m_iWidth + x_d;
        int iSource_Size = oSource.m_iWidth * oSource.m_iHeight;
        short iRemain_x = oSource.m_iWidth - x_s;
        unsigned char* pDest = &oDest.m_pChannel[0][y_d * oDest.m_iWidth + x_d],
            * pSource = &oSource.m_pChannel[0][blockIdx.x * oSource.m_iWidth + x_s];

        for (short j = 0; j < oDest.m_iChannel_Count; j++, pSource += iSource_Size, pDest += iDest_Size)
        {
            if (iRemain_x < iPix_Group_Size)
            {
                for (short i = 0; i < iRemain_x; i++)
                    pDest[i] = pSource[i];
            }
            else
                *(Pixel_4*)pDest = *(Pixel_4*)pSource;
        }

        //if (iRemain_x < iPix_Group_Size)
        //{//���һ��,���Ҳ�����Ҫ������
        //    /*for (short i = 0; i < iRemain_x; i++)
        //        for (short j = 0; j < oDest.m_iChannel_Count; j++)
        //            oDest.m_pChannel[j][iPos_d + i] = oSource.m_pChannel[j][iPos_s + i];*/
        //    for (short j = 0; j < oDest.m_iChannel_Count; j++)
        //        for (short i = 0; i < iRemain_x; i++)
        //            oDest.m_pChannel[j][iPos_d + i] = oSource.m_pChannel[j][iPos_s + i];
        //}else
        //{
        //    for (short j = 0; j < oDest.m_iChannel_Count; j++)
        //        *(Pixel_4*)&oDest.m_pChannel[j][iPos_d] = *(Pixel_4*)&oSource.m_pChannel[j][iPos_s];
        //}
    }

    __syncthreads();

    //���м䲿�ֳ�����߽�
    if (threadIdx.x < iLeft)
    {
        /* unsigned char* pDest = &oDest.m_pChannel[0][y_d * oDest.m_iWidth + threadIdx.x],
             * pSource = &oDest.m_pChannel[0][y_d * oDest.m_iWidth + iLeft * 2 - threadIdx.x - 1];
         for (int j = 0; j < oDest.m_iChannel_Count; j++,pSource+= iDest_Size,pDest+= iDest_Size)
             *pDest = *pSource;*/
        for (int j = 0; j < oDest.m_iChannel_Count; j++)
            oDest.m_pChannel[j][y_d * oDest.m_iWidth + threadIdx.x] = oDest.m_pChannel[j][y_d * oDest.m_iWidth + iLeft * 2 - threadIdx.x - 1];
    }

    if (threadIdx.x < iRight)
    {
        unsigned char* pDest = &oDest.m_pChannel[0][y_d * oDest.m_iWidth + oDest.m_iWidth - threadIdx.x - 1],
            * pSource = &oDest.m_pChannel[0][y_d * oDest.m_iWidth + oDest.m_iWidth - iRight * 2 + threadIdx.x];
        for (int j = 0; j < oDest.m_iChannel_Count; j++, pSource += iDest_Size, pDest += iDest_Size)
            *pDest = *pSource;
        //for (int j = 0; j < oDest.m_iChannel_Count; j++)
            //oDest.m_pChannel[j][ y_d * oDest.m_iWidth + oDest.m_iWidth - threadIdx.x - 1] = oDest.m_pChannel[j][y_d * oDest.m_iWidth  + oDest.m_iWidth - iRight*2 + threadIdx.x];
    }
    __syncthreads();

    short iWidth_Div_4 = oDest.m_iWidth >> 2;
    //���м�����ݳ�������ȥ
    if (blockIdx.x < iTop)
    {
        unsigned char* pDest = &oDest.m_pChannel[0][blockIdx.x * oDest.m_iWidth],
            * pSource = &oDest.m_pChannel[0][((iTop << 1) - blockIdx.x - 1) * oDest.m_iWidth];
        int iDist = oDest.m_iWidth - (iWidth_Div_4 << 2);

        for (short j = 0; j < oDest.m_iChannel_Count; j++, pSource += iDest_Size, pDest += iDest_Size)
        {
            for (short x = threadIdx.x; x < iWidth_Div_4; x += blockDim.x)
                *(Pixel_4*)&pDest[x * 4] = *(Pixel_4*)&pSource[x * 4];

            //for (short x = threadIdx.x; x < iWidth_Div_4; x += blockDim.x)
            //    *(Pixel_4*)&oDest.m_pChannel[j][blockIdx.x * oDest.m_iWidth + x * 4] = *(Pixel_4*)&oDest.m_pChannel[j][(iTop * 2 - blockIdx.x - 1) * oDest.m_iWidth + x * 4];

            //��β����
            if (threadIdx.x < iDist)
                pDest[oDest.m_iWidth - threadIdx.x - 1] = pDest[oDest.m_iWidth - iRight * 2 + threadIdx.x];
            //oDest.m_pChannel[j][blockIdx.x * oDest.m_iWidth + oDest.m_iWidth - threadIdx.x - 1] = oDest.m_pChannel[j][blockIdx.x * oDest.m_iWidth + oDest.m_iWidth - iRight * 2 + threadIdx.x];
        }
    }

    //���м����ݳ����±�ȥ
    if (blockIdx.x >= oSource.m_iHeight - iBottom)
    {
        int iDist_y = oSource.m_iHeight - 1 - blockIdx.x;
        int iDist = oDest.m_iWidth - (iWidth_Div_4 << 2);
        unsigned char* pDest = &oDest.m_pChannel[0][(oDest.m_iHeight - iBottom + iDist_y) * oDest.m_iWidth],
            * pSource = &oDest.m_pChannel[0][(oDest.m_iHeight - iBottom - iDist_y - 1) * oDest.m_iWidth];

        for (int j = 0; j < oDest.m_iChannel_Count; j++, pSource += iDest_Size, pDest += iDest_Size)
        {
            for (short x = threadIdx.x; x < iWidth_Div_4; x += blockDim.x)
            {
                *(Pixel_4*)&pDest[x * 4] = *(Pixel_4*)&pSource[x * 4];
                //*(Pixel_4*)&oDest.m_pChannel[j][(oDest.m_iHeight - iBottom + iDist_y) * oDest.m_iWidth + x * 4] =
                    //*(Pixel_4*)&oDest.m_pChannel[j][(oDest.m_iHeight - iBottom - iDist_y - 1) * oDest.m_iWidth + x * 4];
            }
            //��β����
            if (threadIdx.x < iDist)
                pDest[oDest.m_iWidth - threadIdx.x - 1] = pDest[oDest.m_iWidth - iRight * 2 + threadIdx.x];
            //oDest.m_pChannel[j][(oDest.m_iHeight - iBottom + iDist_y) * oDest.m_iWidth + oDest.m_iWidth - threadIdx.x - 1] = oDest.m_pChannel[j][(oDest.m_iHeight - iBottom + iDist_y) * oDest.m_iWidth + oDest.m_iWidth - iRight * 2 + threadIdx.x];
        }
    }
}

void Copy_Make_Border_GPU(Image oSource, Image oDest, short iLeft, short iTop, short iRight, short iBottom)
{//�����Ż����߿���
    dim3 oThread, oGrid;
    const int iPix_Group_Size = 4;
    oThread.x = (oSource.m_iWidth + iPix_Group_Size - 1) / iPix_Group_Size;
    oGrid.x = oSource.m_iHeight;
    //Disp_Cuda_Error();
    //unsigned long long tStart = iGet_Tick_Count();

    //���427������480
    //for (int i = 0; i < 10000; i++)
        _Copy_Make_Border_GPU << <oGrid, oThread >> > (oSource, oDest, iLeft, iTop, iRight, iBottom, iPix_Group_Size);
    //Disp_Cuda_Error();
    //printf("%lld\n", iGet_Tick_Count() - tStart);
    //bSave_Image_GPU("c:\\tmp\\3.bmp", oDest);
    //Compare_Image("c:\\tmp\\2.bmp", "c:\\tmp\\3.bmp");

    return;
}